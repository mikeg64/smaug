#include "hip/hip_runtime.h"
// includes, system
#include <stdio.h>
#include <assert.h>
 
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */

// Simple utility function to check for CUDA runtime errors
void checkCUDAError(const char* msg);
 
// Part3: implement the kernel




__global__ void max_parallel(double *cmax, double *temp,int ndimp, double maxac)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  extern __shared__ double partialResult[];

  int i;
   partialResult[tid]=0.0;
   if(iindex<ndimp)
              partialResult[tid]=temp[iindex];
  __syncthreads();

      //   if(temp[iindex]==maxac)
      //      printf("max here %d %d \n",tid,blockIdx.x);



     //if(tid==0)
    //     printf("sero %d\n",blockIdx.x);
for(unsigned int s=1; s < blockDim.x; s *= 2) {


        if ((tid % (2*s)) == 0) {
            if(partialResult[tid+s]>partialResult[tid])
                 partialResult[tid]=partialResult[tid + s];
        }
        __syncthreads();
    }

    __syncthreads();
    if(tid==0)
    {
      cmax[blockIdx.x]=partialResult[0];
      //temp[blockIdx.x]=partialResult[0];
     }
    __syncthreads();

}




/////////////////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////////////////
int main( int argc, char** argv) 
{
    // pointer for host memory and size
    int *h_a;
    double *h_c, *h_temp;
    double maxc=-1.0;
    double tmax=-1.0;
    int dimA = 256 * 1024; // 256K elements (1MB total)
    dimA=256*2048;
    dimA=2097152;
    // pointer for device memory
    int *d_b, *d_a;

    double *d_c, *d_temp;
 
    // define grid and block size
    int numThreadsPerBlock = 128;
 
    // Part 1: compute number of blocks needed based on 
    // array size and desired block size
    int numBlocks = dimA / numThreadsPerBlock;  
 
    srand (time(NULL));

    // allocate host and device memory
    size_t memSize = numBlocks * numThreadsPerBlock * sizeof(int);
    h_a = (int *) malloc(memSize);
    hipMalloc( (void **) &d_a, memSize );
    hipMalloc( (void **) &d_b, memSize );

     int smemSize = numThreadsPerBlock * sizeof(double);

    size_t dmemSize = numBlocks * numThreadsPerBlock * sizeof(double);
    h_c = (double *) malloc(dmemSize);
    h_temp = (double *) malloc(dmemSize);

    hipMalloc( (void **) &d_c, dmemSize );
    hipMalloc( (void **) &d_temp, dmemSize );

    int imax;

    int ccount=0;
    int j=0;
   // for( j=0; j<1000; j++)
   // {
   // tmax=-1;

    // Initialize input array on host
    for (int i = 0; i < dimA; ++i)
    {
        h_a[i] = i;
        h_c[i]=(rand()%100000000);

        if(h_c[i]>tmax)
        {
            tmax=h_c[i];
            imax=i;
        }
        //printf(" %g ",h_c[i]);
    }
    printf("\n\n\n %d %f %d\n", dimA, tmax, imax);
 
    // Copy host array to device array
    hipMemcpy( d_a, h_a, memSize, hipMemcpyHostToDevice );
    hipMemcpy( d_c, h_c, dmemSize, hipMemcpyHostToDevice );
    hipMemcpy( d_temp, h_c, dmemSize, hipMemcpyHostToDevice );
 
 
    // device to host copy
    hipMemcpy( h_a, d_b, memSize, hipMemcpyDeviceToHost );
 
    // Check for any CUDA errors
    checkCUDAError("memcpy");
 


	  for(int i=0;i<numBlocks;i++)
		       h_temp[i]=0;
	  hipMemcpy(d_temp, h_temp, numBlocks*sizeof(double), hipMemcpyHostToDevice);

	  max_parallel<<<numBlocks,numThreadsPerBlock,smemSize>>>(d_temp,d_c,dimA,tmax);
	  hipDeviceSynchronize();
	  hipMemcpy(h_temp, d_temp, numBlocks*sizeof(double), hipMemcpyDeviceToHost);

         for(int i=0;i<numBlocks;i++)
         {          		
                if(h_temp[i]>maxc) maxc=h_temp[i]; 
                //printf(" %f ",h_temp[i]);

         }

       if(maxc==tmax) ccount++;

         printf("\n\n\nnumblocks %d %d max=%f %f %d\n",j, numBlocks, maxc, tmax, ccount);



 //     }

  


       
 
    // check if kernel execution generated an error
    // Check for any CUDA errors
    checkCUDAError("kernel invocation");

















 
    // free device memory
    hipFree(d_a);
    hipFree(d_b);
 
    // free host memory
    free(h_a);
 
    // If the program makes it this far, then the results are 
    // correct and there are no run-time errors.  Good work!
    printf("Correct!\n");
 
    return 0;
}
void checkCUDAError(const char *msg)
{
    hipError_t err = hipGetLastError();
    if( hipSuccess != err) 
    {
        fprintf(stderr, "Cuda error: %s: %s.\n", msg, 
                                  hipGetErrorString( err) );
        exit(EXIT_FAILURE);
    }                         
}

