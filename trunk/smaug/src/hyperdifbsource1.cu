#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"
#include "../include/iobparams.h"
/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/smaugcukernels.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_hdb1.cuh"






__global__ void hyperdifbsource4_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb,real dt)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];

  int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

#ifdef USE_SAC_3D
  real rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1)+(p->dx[2])*(dim==2));
#else
  real rdx=(((p->dx[0])*(dim==0))+  (p->dx[1])*(dim==1)  );
#endif

int shift=order*NVAR*dimp;
   

     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
     if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
    if(i<((p->n[0])) && j<((p->n[1])))
     #endif

                         {
                              //                                                                                  - sign here same as vac maybe a +
                              wmod[fencode3_hdb1(p,iia,b1+ii0)+(ordero*NVAR*dimp)]=wmod[fencode3_hdb1(p,iia,b1+ii0)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdb1(p,iia,b1+ii0)]; 
                             wmod[fencode3_hdb1(p,iia,energy)+(ordero*NVAR*dimp)]=wmod[fencode3_hdb1(p,iia,energy)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdb1(p,iia,energy)]; 

                         }
              //  }	

  //__syncthreads();  
}




__global__ void hyperdifbsource3_parallel(struct params *p, real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb,real dt)
{
 
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

   real dy=p->dx[1];
  real dx=p->dx[0];

  int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

#ifdef USE_SAC_3D
  real rdx=(((wd[encode3_hdb1(p,i,j,k,delx1)])*(dim==0))+(wd[encode3_hdb1(p,i,j,k,delx2)])*(dim==1)+(wd[encode3_hdb1(p,i,j,k,delx3)])*(dim==2));
#else
  real rdx=(((wd[encode3_hdb1(p,i,j,k,delx1)])*(dim==0))+  (wd[encode3_hdb1(p,i,j,k,delx2)])*(dim==1)  );
#endif

int shift=order*NVAR*dimp;

   

     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
     if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
    if(i<((p->n[0])) && j<((p->n[1])))
     #endif
	{		               



dwn1[fencode3_hdb1(p,iia,b1+ii0)]=sb*(wtemp[fencode3_hdb1(p,iia,tmp5)]*wd[fencode3_hdb1(p,iia,hdnur)]-wtemp[fencode3_hdb1(p,iia,tmp4)]*wd[fencode3_hdb1(p,iia,hdnul)])/rdx;

dwn1[fencode3_hdb1(p,iia,energy)]=sb*(wtemp[fencode3_hdb1(p,iia,tmp3)]*wtemp[fencode3_hdb1(p,iia,tmp5)]*wd[fencode3_hdb1(p,iia,hdnur)]-wtemp[fencode3_hdb1(p,iia,tmp2)]*wtemp[fencode3_hdb1(p,iia,tmp4)]*wd[fencode3_hdb1(p,iia,hdnul)])/rdx;

//if(i==0 && j==139)
//           printf("b1 e %d %10.20g %10.20g\n",ii0,dwn1[fencode3_hdb1(p,iia,b1+ii0)],dwn1[fencode3_hdb1(p,iia,energy)]);



//    if(i==127 && j==2)
//           printf("tmpL R %10.20g %10.20g\n",wtemp[fencode3_hdb1(p,iia,tmp4)]*wd[fencode3_hdb1(p,iia,hdnul)],wtemp[fencode3_hdb1(p,iia,tmp5)]*wd[fencode3_hdb1(p,iia,hdnur)]);
//    if(i==127 && j==2)
//           printf("nuL R %10.20g %10.20g\n",wd[fencode3_hdb1(p,iia,hdnul)],wd[fencode3_hdb1(p,iia,hdnur)]);

   }

 //__syncthreads();


 
}






__global__ void hyperdifbsource2_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];

  int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  


int shift=order*NVAR*dimp;

   

     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
     //if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     if(i>0 && j >0 && k>0 && i<((p->n[0])-1) && j<((p->n[1])-1)  && k < ((p->n[2])-1))
     #else
    //if(i<((p->n[0])) && j<((p->n[1])))
    if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
     #endif
 // if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
	{		               

//wtemp[fencode3_hdb1(p,iia,tmp4)]=grad1l3n_hdb1(wtemp,wd,p,iia,tmp1,dim);
//wtemp[fencode3_hdb1(p,iia,tmp5)]=grad1r3n_hdb1(wtemp,wd,p,iia,tmp1,dim);


wtemp[fencode3_hdb1(p,iia,tmp4)]=grad1l3n_hdb1(wtemp,wd,p,iia,tmp1,dim);
wtemp[fencode3_hdb1(p,iia,tmp5)]=grad1r3n_hdb1(wtemp,wd,p,iia,tmp1,dim);

   }

 //__syncthreads();   

  //   if(i==127 && j==2)
  //         printf("L R %d %10.20g %10.20g\n",dim,wtemp[fencode3_hdb1(p,iia,tmp4)],wtemp[fencode3_hdb1(p,iia,tmp5)]);


   



}



__global__ void hyperdifbsource1_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];

  int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  


int shift=order*NVAR*dimp;
   

     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
     if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
    if(i<((p->n[0])) && j<((p->n[1])))
     #endif

  //init rhol and rhor

  {
    for(int f=tmp1; f<=tmp8; f++)	
        wtemp[fencode3_hdb1(p,iia,f)]=0.0;

   dwn1[fencode3_hdb1(p,iia,energy)]=0.0;
   dwn1[fencode3_hdb1(p,iia,b1+ii0)]=0.0;
  }


 //__syncthreads();



   

     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
     if(i>0 && j >0 && k>0 && i<((p->n[0])-1) && j<((p->n[1])-1) && k<((p->n[2])-1))
     #else
    if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
     #endif

  //if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {

     #ifdef USE_SAC_3D
       wtemp[fencode3_hdb1(p,iia,tmp2)]=(wmod[shift+fencode3_hdb1(p,iia,b1+jj)]+wmod[shift+encode3_hdb1(p,i-(dim==0),j-(dim==1),k-(dim==2),b1+jj)])/2;
       wtemp[fencode3_hdb1(p,iia,tmp3)]=(wmod[shift+fencode3_hdb1(p,iia,b1+jj)]+wmod[shift+encode3_hdb1(p,i+(dim==0),j+(dim==1),k+(dim==2),b1+jj)])/2;
     #else
       wtemp[fencode3_hdb1(p,iia,tmp2)]=(wmod[shift+fencode3_hdb1(p,iia,b1+jj)]+wmod[shift+encode3_hdb1(p,i-(dim==0),j-(dim==1),k,b1+jj)])/2;
       wtemp[fencode3_hdb1(p,iia,tmp3)]=(wmod[shift+fencode3_hdb1(p,iia,b1+jj)]+wmod[shift+encode3_hdb1(p,i+(dim==0),j+(dim==1),k,b1+jj)])/2;
     #endif
     wtemp[fencode3_hdb1(p,iia,tmp1)]=wmod[shift+fencode3_hdb1(p,iia,b1+field)];

     //if(i==127 && j==2)
     //      printf("tmp1 %d %10.20g\n",field,wmod[shift+fencode3_hdb1(p,iia,b1+field)]);

   }


//__syncthreads();






   

}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdb1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifbsource1(struct params **p,  struct params **d_p,   real **d_wmod, real **d_dwn1, real **d_wd, int order,int ordero, real **d_wtemp, int field, int dim, int jj, int ii0,int mm,real sb, real dt)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     hyperdifbsource1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb); 
     hipDeviceSynchronize();
     hyperdifbsource2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb); 
     hipDeviceSynchronize();
     hyperdifbsource3_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb,dt); 
     hipDeviceSynchronize();
     hyperdifbsource4_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb,dt); 
     hipDeviceSynchronize();

}







