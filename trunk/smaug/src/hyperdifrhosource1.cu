#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"
#include "../include/iobparams.h"
/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/smaugcukernels.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_hdr1.cuh"
__global__ void hyperdifrhosource2_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, real dt)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1,ii0;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  real rdx;

   int ip,jp;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;  
   
  

     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
	  rdx=(((wd[encode3_hdr1(p,i,j,k,delx1)])*(dim==0))+(wd[encode3_hdr1(p,i,j,k,delx2)])*(dim==1)+(wd[encode3_hdr1(p,i,j,k,delx3)])*(dim==2));
	#else
	  rdx=(((wd[encode3_hdr1(p,i,j,k,delx1)])*(dim==0))+(wd[encode3_hdr1(p,i,j,k,delx2)])*(dim==1));
	#endif



     #ifdef USE_SAC_3D
       if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif
  {
     



//dwn1[fencode3_hdr1(p,ii,field)]=( (wd[fencode3_hdr1(p,ii,hdnur)]+wd[fencode3_hdr1(p,ii,nushk1+dim)]) * wtemp[fencode3_hdr1(p,ii,tmp1)] - (wd[fencode3_hdr1(p,ii,hdnul)]+wd[fencode3_hdr1(p,ii,nushk1+dim)]) *wtemp[fencode3_hdr1(p,ii,tmp2)]            )/rdx;

                             // wmod[fencode3_hdr1(p,ii,field)+(ordero*NVAR*dimp)]=wmod[fencode3_hdr1(p,ii,field)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdr1(p,ii,field)]; 
wmod[fencode3_hdr1(p,ii,field)+(ordero*NVAR*dimp)]=wmod[fencode3_hdr1(p,ii,field)+(ordero*NVAR*dimp)]+dt*( (wd[fencode3_hdr1(p,ii,hdnur)]+wd[fencode3_hdr1(p,ii,nushk1+dim)]) * wtemp[fencode3_hdr1(p,ii,tmp1)] - (wd[fencode3_hdr1(p,ii,hdnul)]+wd[fencode3_hdr1(p,ii,nushk1+dim)]) *wtemp[fencode3_hdr1(p,ii,tmp2)]            )/rdx; 
  }

//__syncthreads();




 
}



__global__ void hyperdifrhosource1_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1,ii0;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;
  real rdx;

   int ip,jp;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;  

 

     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif
  //init rhol and rhor
  //if(i<((p->n[0])) && j<((p->n[1])))
  {
    //for(int f=tmp1; f<=tmprhor; f++)	
    //    wtemp[fencode_hdr1(p,i,j,f)]=0.0;
    dwn1[fencode3_hdr1(p,ii,field)]=0.0;
    wtemp[fencode3_hdr1(p,ii,tmp1)]=0.0;
    wtemp[fencode3_hdr1(p,ii,tmp2)]=0.0;
    //wtemp[fencode_hdr1(p,i,j,tmp3)]=0.0;
   }

 //__syncthreads();

/*     #ifdef USE_SAC_3D
	  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1)+(p->dx[2])*(dim==2));
	#else
	  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1));
	#endif   */

 

     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(i>0 && j >0 && k>0 && i<((p->n[0])-1) && j<((p->n[1])-1) && k<((p->n[2])-1))
     #else
       if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
     #endif

  
  //if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {
     

    wtemp[fencode3_hdr1(p,ii,tmp1)]=grad1r3n_hdr1(wmod+shift,wd,p,ii,rho,dim);
    wtemp[fencode3_hdr1(p,ii,tmp2)]=grad1l3n_hdr1(wmod+shift,wd,p,ii,rho,dim);
  }

//__syncthreads();




 
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdr1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifrhosource1(struct params **p, struct params **d_p,   real **d_wmod, real **d_dwn1, real **d_wd, int order, int ordero,real **d_wtemp, int field, int dim, real dt)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     hyperdifrhosource1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim);
     hipDeviceSynchronize();
    hyperdifrhosource2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,dt);
     hipDeviceSynchronize();


}







