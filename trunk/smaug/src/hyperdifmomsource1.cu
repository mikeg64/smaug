#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"
#include "../include/iobparams.h"
/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/smaugcukernels.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_hdm1.cuh"


__global__ void hyperdifmomsource3_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  real rdx;
   int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;








     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

#ifdef USE_SAC_3D
  rdx=(((wd[encode3_hdm1(p,i,j,k,delx1)])*(dim==0))+(wd[encode3_hdm1(p,i,j,k,delx2)])*(dim==1)+(wd[encode3_hdm1(p,i,j,k,delx3)])*(dim==2));
#else
  rdx=(((wd[encode3_hdm1(p,i,j,k,delx1)])*(dim==0))+  (wd[encode3_hdm1(p,i,j,k,delx2)])*(dim==1)  );
#endif


     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif

  //if(i<((p->n[0])) && j<((p->n[1])))
	{		               

;//dwn1[fencode3_hdm1(p,iia,energy)]=(wtemp[fencode3_hdm1(p,iia,tmp6)]*(wd[fencode3_hdm1(p,iia,hdnur)]+wd[fencode3_hdm1(p,iia,nushk1+dim)])*wtemp[fencode3_hdm1(p,iia,tmp8)]-wtemp[fencode3_hdm1(p,iia,tmp5)]*(wd[fencode3_hdm1(p,iia,hdnul)]+wd[fencode3_hdm1(p,iia,nushk1+dim)])*wtemp[fencode3_hdm1(p,iia,tmp7)])/(rdx)/2;

dwn1[fencode3_hdm1(p,iia,mom1+ii0)]=(wtemp[fencode3_hdm1(p,iia,tmp3)]*(wd[fencode3_hdm1(p,iia,hdnur)]+wd[fencode3_hdm1(p,iia,nushk1+dim)])*wtemp[fencode3_hdm1(p,iia,tmp8)]-wtemp[fencode3_hdm1(p,iia,tmp2)]*(wd[fencode3_hdm1(p,iia,hdnul)]+wd[fencode3_hdm1(p,iia,nushk1+dim)])*wtemp[fencode3_hdm1(p,iia,tmp7)])/(rdx)/2;

                              wmod[fencode3_hdm1(p,iia,mom1+ii0)+(ordero*NVAR*dimp)]=wmod[fencode3_hdm1(p,iia,mom1+ii0)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdm1(p,iia,mom1+ii0)];
                             //wmod[fencode3_hdm1(p,iia,mom1+ii0)+(ordero*NVAR*dimp)]=wmod[fencode3_hdm1(p,iia,mom1+ii0)+(ordero*NVAR*dimp)]+dt*((wtemp[fencode3_hdm1(p,iia,tmp3)]*(wd[fencode3_hdm1(p,iia,hdnur)]+wd[fencode3_hdm1(p,iia,nushk1+dim)])*wtemp[fencode3_hdm1(p,iia,tmp8)]-wtemp[fencode3_hdm1(p,iia,tmp2)]*(wd[fencode3_hdm1(p,iia,hdnul)]+wd[fencode3_hdm1(p,iia,nushk1+dim)])*wtemp[fencode3_hdm1(p,iia,tmp7)])/(rdx)/2);

   //del=wmod[fencode3_hdm1(p,iia,energy)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdm1(p,iia,energy)]; 
   //if(del<0.011 && del>0.009)
           //  wmod[fencode3_hdm1(p,iia,energy)+(ordero*NVAR*dimp)]=del; 
                            ;//wmod[fencode3_hdm1(p,iia,energy)+(ordero*NVAR*dimp)]=wmod[fencode3_hdm1(p,iia,energy)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdm1(p,iia,energy)]; 
                               
                             // wmod[fencode3_hdm1(p,iia,mom1+ii0)+(ordero*NVAR*dimp)]=wmod[fencode3_hdm1(p,iia,mom1+ii0)+(ordero*NVAR*dimp)]+dt*((wtemp[fencode3_hdm1(p,iia,tmp3)]*wd[fencode3_hdm1(p,iia,hdnur)]*wtemp[fencode3_hdm1(p,iia,tmp8)]-wtemp[fencode3_hdm1(p,iia,tmp2)]*wd[fencode3_hdm1(p,iia,hdnul)]*wtemp[fencode3_hdm1(p,iia,tmp7)])/(rdx)/2); 

                            // wmod[fencode3_hdm1(p,iia,energy)+(ordero*NVAR*dimp)]=wmod[fencode3_hdm1(p,iia,energy)+(ordero*NVAR*dimp)]+dt*((wtemp[fencode3_hdm1(p,iia,tmp6)]*wd[fencode3_hdm1(p,iia,hdnur)]*wtemp[fencode3_hdm1(p,iia,tmp8)]-wtemp[fencode3_hdm1(p,iia,tmp5)]*wd[fencode3_hdm1(p,iia,hdnul)]*wtemp[fencode3_hdm1(p,iia,tmp7)])/(rdx)/2); 


   }

 //__syncthreads();






   



  
}



__global__ void hyperdifmomsource3a_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  real rdx;
   int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;







     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif


#ifdef USE_SAC_3D
  rdx=(((wd[encode3_hdm1(p,i,j,k,delx1)])*(dim==0))+(wd[encode3_hdm1(p,i,j,k,delx2)])*(dim==1)+(wd[encode3_hdm1(p,i,j,k,delx3)])*(dim==2));
#else
  rdx=(((wd[encode3_hdm1(p,i,j,k,delx1)])*(dim==0))+  (wd[encode3_hdm1(p,i,j,k,delx2)])*(dim==1)  );
#endif

     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif

  //if(i<((p->n[0])) && j<((p->n[1])))
	{		               

dwn1[fencode3_hdm1(p,iia,energy)]=(wtemp[fencode3_hdm1(p,iia,tmp6)]*(wd[fencode3_hdm1(p,iia,hdnur)]+wd[fencode3_hdm1(p,iia,nushk1+dim)])*wtemp[fencode3_hdm1(p,iia,tmp8)]-wtemp[fencode3_hdm1(p,iia,tmp5)]*(wd[fencode3_hdm1(p,iia,hdnul)]+wd[fencode3_hdm1(p,iia,nushk1+dim)])*wtemp[fencode3_hdm1(p,iia,tmp7)])/(rdx)/2;

;//dwn1[fencode3_hdm1(p,iia,mom1+ii0)]=(wtemp[fencode3_hdm1(p,iia,tmp3)]*(wd[fencode3_hdm1(p,iia,hdnur)]+wd[fencode3_hdm1(p,iia,nushk1+dim)])*wtemp[fencode3_hdm1(p,iia,tmp8)]-wtemp[fencode3_hdm1(p,iia,tmp2)]*(wd[fencode3_hdm1(p,iia,hdnul)]+wd[fencode3_hdm1(p,iia,nushk1+dim)])*wtemp[fencode3_hdm1(p,iia,tmp7)])/(rdx)/2;

                             ;// wmod[fencode3_hdm1(p,iia,mom1+ii0)+(ordero*NVAR*dimp)]=wmod[fencode3_hdm1(p,iia,mom1+ii0)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdm1(p,iia,mom1+ii0)];

   //del=wmod[fencode3_hdm1(p,iia,energy)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdm1(p,iia,energy)]; 
   //if(del<0.011 && del>0.009)
           //  wmod[fencode3_hdm1(p,iia,energy)+(ordero*NVAR*dimp)]=del; 
                            wmod[fencode3_hdm1(p,iia,energy)+(ordero*NVAR*dimp)]=wmod[fencode3_hdm1(p,iia,energy)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdm1(p,iia,energy)]; 
                              // wmod[fencode3_hdm1(p,iia,energy)+(ordero*NVAR*dimp)]=wmod[fencode3_hdm1(p,iia,energy)+(ordero*NVAR*dimp)]+dt*((wtemp[fencode3_hdm1(p,iia,tmp6)]*(wd[fencode3_hdm1(p,iia,hdnur)]+wd[fencode3_hdm1(p,iia,nushk1+dim)])*wtemp[fencode3_hdm1(p,iia,tmp8)]-wtemp[fencode3_hdm1(p,iia,tmp5)]*(wd[fencode3_hdm1(p,iia,hdnul)]+wd[fencode3_hdm1(p,iia,nushk1+dim)])*wtemp[fencode3_hdm1(p,iia,tmp7)])/(rdx)/2); 
                             // wmod[fencode3_hdm1(p,iia,mom1+ii0)+(ordero*NVAR*dimp)]=wmod[fencode3_hdm1(p,iia,mom1+ii0)+(ordero*NVAR*dimp)]+dt*((wtemp[fencode3_hdm1(p,iia,tmp3)]*wd[fencode3_hdm1(p,iia,hdnur)]*wtemp[fencode3_hdm1(p,iia,tmp8)]-wtemp[fencode3_hdm1(p,iia,tmp2)]*wd[fencode3_hdm1(p,iia,hdnul)]*wtemp[fencode3_hdm1(p,iia,tmp7)])/(rdx)/2); 

                            // wmod[fencode3_hdm1(p,iia,energy)+(ordero*NVAR*dimp)]=wmod[fencode3_hdm1(p,iia,energy)+(ordero*NVAR*dimp)]+dt*((wtemp[fencode3_hdm1(p,iia,tmp6)]*wd[fencode3_hdm1(p,iia,hdnur)]*wtemp[fencode3_hdm1(p,iia,tmp8)]-wtemp[fencode3_hdm1(p,iia,tmp5)]*wd[fencode3_hdm1(p,iia,hdnul)]*wtemp[fencode3_hdm1(p,iia,tmp7)])/(rdx)/2); 


   }

 //__syncthreads();






   



  
}


__global__ void hyperdifmomsource2_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  real rdx;

   int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;

/*#ifdef USE_SAC_3D
  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1)+(p->dx[2])*(dim==2));
#else
  rdx=(((p->dx[0])*(dim==0))+  (p->dx[1])*(dim==1)  );
#endif*/

 

     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i>0 && j >0 && k>0 && i<((p->n[0])-1) && j<((p->n[1])-1)  && k<((p->n[2])-1))
     #else
       if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
     #endif
//if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {

     wtemp[fencode3_hdm1(p,iia,tmp8)]=grad1r3n_hdm1(wtemp,wd,p,iia,tmp4,dim);
     wtemp[fencode3_hdm1(p,iia,tmp7)]=grad1l3n_hdm1(wtemp,wd,p,iia,tmp4,dim);

   }


//__syncthreads();  //can remove?



  
}




__global__ void hyperdifmomsource1_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  real rdx;
   int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;

/*#ifdef USE_SAC_3D
  rdx=(((p->dx[0])*(dim==0))+(p->dx[1])*(dim==1)+(p->dx[2])*(dim==2));
#else
  rdx=(((p->dx[0])*(dim==0))+  (p->dx[1])*(dim==1)  );
#endif*/



     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
       if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif
  //init rhol and rhor
  //if(i<((p->n[0])) && j<((p->n[1])))
  {
    for(int f=tmp1; f<=tmp8; f++)	
        wtemp[fencode3_hdm1(p,iia,f)]=0.0;


dwn1[fencode3_hdm1(p,iia,energy)]=0.0;
dwn1[fencode3_hdm1(p,iia,mom1+ii0)]=0.0;
   }



 //__syncthreads();

//tmp2  rhor
//tmp3  rhol
//tmp1  mom+field/rho

//tmp4  rhoc


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
       if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif
//if( i<((p->n[0])) && j<((p->n[1])))
  {
#ifdef ADIABHYDRO
;
#else
    wtemp[fencode3_hdm1(p,iia,tmp4)]=wmod[(shift)+fencode3_hdm1(p,iia,mom1+field)]/(wmod[(shift)+fencode3_hdm1(p,iia,rho)]+wmod[(shift)+fencode3_hdm1(p,iia,rhob)]);
#endif

   }


//__syncthreads();





     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
       if(i>0 && j >0 && k>0 && i<((p->n[0])-1) && j<((p->n[1])-1)  && k<((p->n[2])-1))
     #else
       if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
     #endif
//if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {

     #ifdef USE_SAC_3D
       wtemp[fencode3_hdm1(p,iia,tmp2)]=(wmod[(shift)+fencode3_hdm1(p,iia,rho)]+wmod[(shift)+fencode3_hdm1(p,iia,rhob)]+wmod[(shift)+encode3_hdm1(p,i-(dim==0),j-(dim==1),k-(dim==2),rho)]+wmod[(shift)+encode3_hdm1(p,i-(dim==0),j-(dim==1),k-(dim==2),rhob)])/2;
       wtemp[fencode3_hdm1(p,iia,tmp3)]=(wmod[(shift)+fencode3_hdm1(p,iia,rho)]+wmod[(shift)+fencode3_hdm1(p,iia,rhob)]+wmod[(shift)+encode3_hdm1(p,i+(dim==0),j+(dim==1),k+(dim==2),rho)]+wmod[(shift)+encode3_hdm1(p,i+(dim==0),j+(dim==1),k+(dim==2),rhob)])/2;
     #endif

     #ifdef USE_SAC
       wtemp[fencode3_hdm1(p,iia,tmp2)]=(wmod[(shift)+fencode3_hdm1(p,iia,rho)]+wmod[(shift)+fencode3_hdm1(p,iia,rhob)]+wmod[(shift)+fencode_hdm1(p,i-(dim==0),j-(dim==1),rho)]+wmod[(shift)+fencode_hdm1(p,i-(dim==0),j-(dim==1),rhob)])/2;
       wtemp[fencode3_hdm1(p,iia,tmp3)]=(wmod[(shift)+fencode3_hdm1(p,iia,rho)]+wmod[(shift)+fencode3_hdm1(p,iia,rhob)]+wmod[(shift)+fencode_hdm1(p,i+(dim==0),j+(dim==1),rho)]+wmod[(shift)+fencode_hdm1(p,i+(dim==0),j+(dim==1),rhob)])/2;
     #endif


   }


//__syncthreads();



     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
       if(i>0 && j >0 && k>0 && i<((p->n[0])-1) && j<((p->n[1])-1)  && k<((p->n[2])-1))
     #else
       if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
     #endif
//if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {
     #ifdef USE_SAC_3D
     wtemp[fencode3_hdm1(p,iia,tmp5)]=(wmod[(shift)+fencode3_hdm1(p,iia,mom1+ii0)]+wmod[(shift)+encode3_hdm1(p,i-(dim==0),j-(dim==1),k-(dim==2),mom1+ii0)])/2;
     wtemp[fencode3_hdm1(p,iia,tmp6)]=(wmod[(shift)+fencode3_hdm1(p,iia,mom1+ii0)]+wmod[(shift)+encode3_hdm1(p,i+(dim==0),j+(dim==1),k+(dim==2),mom1+ii0)])/2;
     #else
     wtemp[fencode3_hdm1(p,iia,tmp5)]=(wmod[(shift)+fencode3_hdm1(p,iia,mom1+ii0)]+wmod[(shift)+fencode_hdm1(p,i-(dim==0),j-(dim==1),mom1+ii0)])/2;
     wtemp[fencode3_hdm1(p,iia,tmp6)]=(wmod[(shift)+fencode3_hdm1(p,iia,mom1+ii0)]+wmod[(shift)+fencode_hdm1(p,i+(dim==0),j+(dim==1),mom1+ii0)])/2;
     #endif
   }


//__syncthreads();



  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdm1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifmomsource1(struct params **p,  struct params **d_p,   real **d_wmod, real **d_dwn1, real **d_wd, int order, int ordero, real **d_wtemp, int field, int dim, int ii, int ii0, real dt)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;

     hyperdifmomsource1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();

     hyperdifmomsource2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();

     hyperdifmomsource3_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
     hyperdifmomsource3a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();

}







