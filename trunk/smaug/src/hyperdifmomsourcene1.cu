#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"
#include "../include/iobparams.h"
/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/smaugcukernels.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_hdmne1.cuh"

__global__ void hyperdifmomsourcene6_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
   int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;





     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
      if(i<((p->n[0])) && j<((p->n[1])))
     #endif

                        //if(i<((p->n[0])) && j<((p->n[1])))
                         {

                             wmod[fencode3_hdmne1(p,iia,mom1+ii0)+(ordero*NVAR*dimp)]=wmod[fencode3_hdmne1(p,iia,mom1+ii0)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdmne1(p,iia,mom1+ii0)];

 
                           //  wmod[fencode3_hdmne1(p,iia,energy)+(ordero*NVAR*dimp)]=wmod[fencode3_hdmne1(p,iia,energy)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdmne1(p,iia,energy)];
    //if(i==127 && j==252)
    //  p->test=dt*dwn1[fencode3_hdmne1(p,iia,mom1+ii0)];

                         }
              //  }	

  //__syncthreads();


  



}


__global__ void hyperdifmomsourcene6a_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
   int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;





     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
      if(i<((p->n[0])) && j<((p->n[1])))
     #endif

                        //if(i<((p->n[0])) && j<((p->n[1])))
                         {

                           //  wmod[fencode3_hdmne1(p,iia,mom1+ii0)+(ordero*NVAR*dimp)]=wmod[fencode3_hdmne1(p,iia,mom1+ii0)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdmne1(p,iia,mom1+ii0)];

 
                             wmod[fencode3_hdmne1(p,iia,energy)+(ordero*NVAR*dimp)]=wmod[fencode3_hdmne1(p,iia,energy)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdmne1(p,iia,energy)];

   // if(i==127 && j==252)
   //   p->test=dt*dwn1[fencode3_hdmne1(p,iia,energy)];

   // if(i==127 && j==252)
   //   p->test=wmod[fencode3_hdmne1(p,iia,energy)+(ordero*NVAR*dimp)];

                         }
              //  }	

  //__syncthreads();


  



}






__global__ void hyperdifmomsourcene5_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
   int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;




     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif

{
 
    dwn1[fencode3_hdmne1(p,iia,mom1+ii0)]=(grad13n_hdmne1(wtemp,wd,p,iia,tmp7,ii));


  }

 //__syncthreads();


  
  



}



__global__ void hyperdifmomsourcene5a_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
   int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;




 


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i>0 && j >0 && k>0 && i<((p->n[0])-1) && j<((p->n[1])-1) && k<((p->n[2])-1))
     #else
      if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
     #endif

     dwn1[fencode3_hdmne1(p,iia,energy)]=(grad13n_hdmne1(wtemp,wd,p,iia,tmp8,ii));
     //dwn1[fencode3_hdmne1(p,iia,energy)]=-2.9e-4;
    //if(i==127 && j==252)
    //  p->test=(p->dt)*dwn1[fencode3_hdmne1(p,iia,energy)];



 //__syncthreads();


  



}

__global__ void hyperdifmomsourcene4_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
   int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;




     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif
  //if(i<((p->n[0])) && j<((p->n[1])))
	{		               
     wtemp[fencode3_hdmne1(p,iia,tmp7)]=wtemp[fencode3_hdmne1(p,iia,tmp1)]*wtemp[fencode3_hdmne1(p,iia,tmp6)];

     wtemp[fencode3_hdmne1(p,iia,tmp8)]=wtemp[fencode3_hdmne1(p,iia,tmp6)]*wmod[(shift)+fencode3_hdmne1(p,iia,mom1+ii0)];

  //  if(i==127 && j==252)
  //    p->test=wtemp[fencode3_hdmne1(p,iia,tmp7)];



   }

 //__syncthreads();




  



}


__global__ void hyperdifmomsourcene3_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];

   int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;




     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif
  //if(i<((p->n[0])) && j<((p->n[1])))
  {

     //wtemp[fencode3_hdmne1(p,iia,tmp6)]=wtemp[fencode3_hdmne1(p,iia,tmp5)]*((wd[fencode3_hdmne1(p,iia,hdnur)]+wd[fencode3_hdmne1(p,iia,hdnul)]+2.0*wd[fencode3_hdmne1(p,iia,nushk1+dim)]))/4.0;
     //wtemp[fencode3_hdmne1(p,iia,tmp6)]=1.0;
     // wtemp[fencode3_hdmne1(p,iia,tmp6)]=-1.0e-41*((wd[fencode3_hdmne1(p,iia,hdnur)]+wd[fencode3_hdmne1(p,iia,hdnul)]+2.0*wd[fencode3_hdmne1(p,iia,nushk1+dim)]))/4.0;
    wtemp[fencode3_hdmne1(p,iia,tmp6)]=wtemp[fencode3_hdmne1(p,iia,tmp5)]*((wd[fencode3_hdmne1(p,iia,hdnur)]+wd[fencode3_hdmne1(p,iia,hdnul)]+2.0*wd[fencode3_hdmne1(p,iia,nushk1+dim)]))/4.0;


 //wtemp[fencode3_hdmne1(p,iia,tmp6)]=wtemp[fencode3_hdmne1(p,iia,tmp5)]*((1.4e-4))/4.0;
  //  if(i==127 && j==252)
  //    p->test=((wd[fencode3_hdmne1(p,iia,hdnur)]+wd[fencode3_hdmne1(p,iia,hdnul)]+2.0*wd[fencode3_hdmne1(p,iia,nushk1+dim)]))/4.0;
;//   if(i==127 && j==252)
;//      p->test=((wtemp[fencode3_hdmne1(p,iia,tmp5)]));


   }

//__syncthreads();

/*if(iindex==0)
{

    for(iia[0]=0;iia[0]<((p->n[0]));iia[0]++)
      for(iia[1]=0;iia[1]<((p->n[1]));iia[1]++)
        //(p->test)+=wtemp[fencode3_hdmne1(p,iia,tmp5)];
        if((wtemp[fencode3_hdmne1(p,iia,tmp5)])>(p->test))
               p->test=wtemp[fencode3_hdmne1(p,iia,tmp5)];

//   p->test/=((p->n[0])*(p->n[1]));


}*/




  



}

__global__ void hyperdifmomsourcene2_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];

   int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;




     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i>0 && j >0 && k>0 && i<((p->n[0])-1) && j<((p->n[1])-1)  && k<((p->n[2])-1))
     #else
       if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
     #endif
  //if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
      wtemp[fencode3_hdmne1(p,iia,tmp5)]=(grad13n_hdmne1(wtemp,wd,p,iia,tmp4,dim));
        // if(i==127 && j==252)
        //       p->test=wtemp[fencode3_hdmne1(p,iia,tmp5)];



//__syncthreads();

}


__global__ void hyperdifmomsourcene1_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, int ii, int ii0, real dt)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];



  

   int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;



  //init rhol and rhor

     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif
  //if(i<((p->n[0])) && j<((p->n[1])))
  {
    for(int f=tmp1; f<=tmp8; f++)	
        wtemp[fencode3_hdmne1(p,iia,f)]=0.0;

     dwn1[fencode3_hdmne1(p,iia,energy)]=0.0;
     dwn1[fencode3_hdmne1(p,iia,mom1+ii0)]=0.0;

   }

 //__syncthreads();



     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif
  //if(i<((p->n[0])) && j<((p->n[1])))
  {

     #ifdef ADIABHYDRO
;
    #else
     wtemp[fencode3_hdmne1(p,iia,tmp1)]=wmod[(shift)+fencode3_hdmne1(p,iia,rho)]+wmod[(shift)+fencode3_hdmne1(p,iia,rhob)];

     wtemp[fencode3_hdmne1(p,iia,tmp4)]=wmod[(shift)+fencode3_hdmne1(p,iia,mom1+field)]/(wmod[(shift)+fencode3_hdmne1(p,iia,rho)]+wmod[(shift)+fencode3_hdmne1(p,iia,rhob)]);
    #endif



   }

//__syncthreads();


}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdmne1ne(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifmomsourcene1(struct params **p, struct params **d_p, real **d_wmod, real **d_dwn1, real **d_wd, int order, int ordero, real **d_wtemp, int field, int dim, int ii, int ii0, real dt)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   //hipSetDevice(selectedDevice);
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     hyperdifmomsourcene1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
     hyperdifmomsourcene2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
     hyperdifmomsourcene3_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();





     hyperdifmomsourcene4_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
     hyperdifmomsourcene5_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
     hyperdifmomsourcene6_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();

     hyperdifmomsourcene5a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
     hyperdifmomsourcene6a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,ii,ii0,dt);
     hipDeviceSynchronize();
hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);
//printf("tmp2 %d %d %10.20g\n",ii,dim,(*p)->test);
//hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);
////printf("test %g\n",(*p)->test);

}







