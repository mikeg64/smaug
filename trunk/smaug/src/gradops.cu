#include "hip/hip_runtime.h"
/* differential operators and boundary condition*/


//These operators for the differential operators
//and the boundary condition routines and may be used by the kernel functions
//They are used as follows during make the field MODID is replaced by a unique identifier
//for the particular cuda source file 
//For example the file centdiff1.cu has identifier cd1
//so that dimproduct_MODID becomes dimproduct_cd1

//The make routine copies the resulting file to a new file called gradops_cd1.cuh
//This file is then included using the line #include "../include/gradops_cd1.cuh"
//in centdiff1.cu

//The routines in centdiff1.cu must call these routines with _MODID replaced by _cd1





__device__ __host__
int dimproduct_MODID (struct params *dp) {

  int tot=1;
  for(int i=0;i<NDIM;i++)
    tot*=dp->n[i];
  return tot; 
}






__device__ __host__
int fencode_MODID (struct params *dp,int ix, int iy, int field) {


    return ( (iy * ((dp)->n[0]) + ix)+(field*((dp)->n[0])*((dp)->n[1])));

}


__device__ __host__
int fencode3_MODID (struct params *dp,int *ii, int field) {


#ifdef USE_SAC_3D
   return (ii[2]*((dp)->n[0])*((dp)->n[1])  + ii[1] * ((dp)->n[0]) + ii[0]+(field*((dp)->n[0])*((dp)->n[1])*((dp)->n[2])));
#else
   return ( ii[1] * ((dp)->n[0]) + ii[0]+(field*((dp)->n[0])*((dp)->n[1])));
#endif

}

__device__ __host__
int encode3p1_MODID (struct params *dp,int ix, int iy, int iz, int field) {


  #ifdef USE_SAC_3D
    return ( (iz*(((dp)->n[0])+1)*(((dp)->n[1])+1)  + iy * (((dp)->n[0])+1) + ix)+(field*(((dp)->n[0])+1)*(((dp)->n[1])+1)*(((dp)->n[2])+1)));
  #else
    return ( (iy * (((dp)->n[0])+1) + ix)+(field*(((dp)->n[0])+1)*(((dp)->n[1])+1)));
  #endif
}




__device__ __host__
int encode3p2_MODID (struct params *dp,int ix, int iy, int iz, int field) {


  #ifdef USE_SAC_3D
    return ( (iz*(((dp)->n[0])+2)*(((dp)->n[1])+2)  + iy * (((dp)->n[0])+2) + ix)+(field*(((dp)->n[0])+2)*(((dp)->n[1])+2)*(((dp)->n[2])+2)));
  #else
    return ( (iy * (((dp)->n[0])+2) + ix)+(field*(((dp)->n[0])+2)*(((dp)->n[1])+2)));
  #endif
}

__device__ __host__
int fencode3p2_MODID (struct params *dp,int *ii, int field) {

  return(encode3p2_MODID(dp,ii[0],ii[1],ii[2],field));
}


__device__ __host__
int encode3_MODID (struct params *dp,int ix, int iy, int iz, int field) {


  #ifdef USE_SAC_3D
    return ( (iz*((dp)->n[0])*((dp)->n[1])  + iy * ((dp)->n[0]) + ix)+(field*((dp)->n[0])*((dp)->n[1])*((dp)->n[2])));
  #else
    return ( (iy * ((dp)->n[0]) + ix)+(field*((dp)->n[0])*((dp)->n[1])));
  #endif
}

__device__ __host__
int encodefixed13_MODID (struct params *dp,int ix, int iy, int iz, int field) {
  #ifdef USE_SAC_3D
    return ( (ix*((dp)->n[1])*((dp)->n[1])  + iy * ((dp)->n[2]) + iz)+(field*4*((dp)->n[1])*((dp)->n[2])));
  #else
    return ( (ix * ((dp)->n[1]) + iy)+(field*4*((dp)->n[1])));
  #endif
}

__device__ __host__
int encodefixed23_MODID (struct params *dp,int ix, int iy, int iz, int field) {
  #ifdef USE_SAC_3D
    return ( (iy*((dp)->n[0])*((dp)->n[2])  + ix * ((dp)->n[0]) + iz)+(4*field*((dp)->n[0])*((dp)->n[2])));
  #else
    return ( (  iy * ((dp)->n[0]) + ix)+(4*field*((dp)->n[0])));
  #endif
}

__device__ __host__
int encodefixed33_MODID (struct params *dp,int ix, int iy, int iz, int field) {
  #ifdef USE_SAC_3D
    return ( ( iz*((dp)->n[0])*((dp)->n[1])  + iy * ((dp)->n[0]) + ix)+(4*field*((dp)->n[0])*((dp)->n[1])));
  #endif
}








__device__ __host__
real grad3d_MODID(real *wmod,struct params *p,int *ii,int field,int dir)
{


 real grad=0;

 
 

 switch(dir)
 {
   case 0:
 
#ifdef USE_SAC_3D
  #ifdef USE_DORDER3
 if(ii[0]>2 && ii[0]<((p->n[0])-3) )
  grad=(  ( ((3*wmod[encode3_MODID(p,ii[0]+1,ii[1],ii[2],field)]-3*wmod[encode3_MODID(p,ii[0]-1,ii[1],ii[2],field)]+3.0*(wmod[encode3_MODID(p,ii[0]-2,ii[1],ii[2],field)]-wmod[encode3_MODID(p,ii[0]+2,ii[1],ii[2],field)])/5.0-(wmod[encode3_MODID(p,ii[0]-3,ii[1],ii[2],field)]-wmod[encode3_MODID(p,ii[0]+3,ii[1],ii[2],field)])/15.0)/2.0))/(2.0*(p->dx[0]))    );
 else 
  #endif
if(ii[0]>1 && ii[0]<((p->n[0])-2) )
 grad=(  ( ((8*wmod[encode3_MODID(p,ii[0]+1,ii[1],ii[2],field)]-8*wmod[encode3_MODID(p,ii[0]-1,ii[1],ii[2],field)]+wmod[encode3_MODID(p,ii[0]-2,ii[1],ii[2],field)]-wmod[encode3_MODID(p,ii[0]+2,ii[1],ii[2],field)])/6.0))/(2.0*(p->dx[0]))    );

   if((ii[0]==(p->n[0])-3) || (ii[0]==(p->n[0])-4)  && ii[1]>1   && ii[1]<(p->n[1])-2 && ii[2]>1   && ii[2]<(p->n[2])-2  )
       grad=0;
   else if(ii[0]==2 || ii[0]==3  && ii[1]>1   && ii[1]<(p->n[1])-2 && ii[2]>1   && ii[2]<(p->n[2])-2  )
       grad=0;
#else

  #ifdef USE_DORDER3
if(ii[0]>2 && ii[0]<((p->n[0])-3) )
 grad=(  ( ((3*wmod[encode3_MODID(p,ii[0]+1,ii[1],0,field)]-3*wmod[encode3_MODID(p,ii[0]-1,ii[1],0,field)]+3.0*(wmod[encode3_MODID(p,ii[0]-2,ii[1],0,field)]-wmod[encode3_MODID(p,ii[0]+2,ii[1],0,field)])/5.0-(wmod[encode3_MODID(p,ii[0]-3,ii[1],0,field)]-wmod[encode3_MODID(p,ii[0]+3,ii[1],0,field)])/15.0)/2.0))/(2.0*(p->dx[0]))    );
 else 
  #endif
if(ii[0]>1 && ii[0]<((p->n[0])-2) )
 grad=(  ( ((8*wmod[encode3_MODID(p,ii[0]+1,ii[1],0,field)]-8*wmod[encode3_MODID(p,ii[0]-1,ii[1],0,field)]+wmod[encode3_MODID(p,ii[0]-2,ii[1],0,field)]-wmod[encode3_MODID(p,ii[0]+2,ii[1],0,field)])/6.0))/(2.0*(p->dx[0]))    );

   if((ii[0]==(p->n[0])-3) || (ii[0]==(p->n[0])-4)  && ii[1]>1   && ii[1]<(p->n[1])-2  )
       grad=0;
   else if(ii[0]==2 || ii[0]==3  && ii[1]>1   && ii[1]<(p->n[1])-2  )
       grad=0;
#endif



   break;

   case 1:

#ifdef USE_SAC_3D

  #ifdef USE_DORDER3
 if(ii[1]>2 && ii[1]<((p->n[1])-3) )
  grad=(  ( ((3*wmod[encode3_MODID(p,ii[0],ii[1]+1,ii[2],field)]-3*wmod[encode3_MODID(p,ii[0],ii[1]-1,ii[2],field)]+3.0*(wmod[encode3_MODID(p,ii[0],ii[1]-2,ii[2],field)]-wmod[encode3_MODID(p,ii[0],ii[1]+2,ii[2],field)])/5.0-(wmod[encode3_MODID(p,ii[0],ii[1]-3,ii[2],field)]-wmod[encode3_MODID(p,ii[0],ii[1]+3,ii[2],field)])/15.0)/2.0))/(2.0*(p->dx[1]))    );
 else 
#endif
if( ii[1] >1 &&  ii[1]<((p->n[1])-2))
	grad=(  ( ((8*wmod[encode3_MODID(p,ii[0],ii[1]+1,ii[2],field)]-8*wmod[encode3_MODID(p,ii[0],ii[1]-1,ii[2],field)]+wmod[encode3_MODID(p,ii[0],ii[1]-2,ii[2],field)]-wmod[encode3_MODID(p,ii[0],ii[1]+2,ii[2],field)])/6.0))/(2.0*(p->dx[1]))    );

   if((ii[1]==(p->n[1])-3) || (ii[1]==(p->n[1])-4)  && ii[0]>1   && ii[0]<(p->n[0])-2  && ii[2]>1   && ii[2]<(p->n[2])-2  )
       grad=0;
   else if(ii[1]==2 || ii[1]==3  && ii[0]>1   && ii[0]<(p->n[0])-2  && ii[2]>1   && ii[2]<(p->n[2])-2  )
       grad=0;
#else

  #ifdef USE_DORDER3
if(ii[1]>2 && ii[1]<((p->n[1])-3) )
 grad=(  ( ((3*wmod[encode3_MODID(p,ii[0],ii[1]+1,0,field)]-3*wmod[encode3_MODID(p,ii[0],ii[1]-1,0,field)]+3.0*(wmod[encode3_MODID(p,ii[0],ii[1]-2,0,field)]-wmod[encode3_MODID(p,ii[0],ii[1]+2,0,field)])/5.0-(wmod[encode3_MODID(p,ii[0],ii[1]-3,0,field)]-wmod[encode3_MODID(p,ii[0],ii[1]+3,0,field)])/15.0)/2.0))/(2.0*(p->dx[1]))    );
else  
#endif
if( ii[1] >1 &&  ii[1]<((p->n[1])-2))
	grad=(  ( ((8*wmod[encode3_MODID(p,ii[0],ii[1]+1,0,field)]-8*wmod[encode3_MODID(p,ii[0],ii[1]-1,0,field)]+wmod[encode3_MODID(p,ii[0],ii[1]-2,0,field)]-wmod[encode3_MODID(p,ii[0],ii[1]+2,0,field)])/6.0))/(2.0*(p->dx[1]))    );

   if((ii[1]==(p->n[1])-3) || (ii[1]==(p->n[1])-4)  && ii[0]>1   && ii[0]<(p->n[0])-2  )
       grad=0;
   else if(ii[1]==2 || ii[1]==3  && ii[0]>1   && ii[0]<(p->n[0])-2  )
       grad=0;
#endif
   break;


   case 2:

#ifdef USE_SAC_3D
  #ifdef USE_DORDER3
 if(ii[2]>2 && ii[2]<((p->n[2])-3) )
  grad=(  ( ((3*wmod[encode3_MODID(p,ii[0],ii[1],ii[2]+1,field)]-3*wmod[encode3_MODID(p,ii[0],ii[1],ii[2]-1,field)]+3.0*(wmod[encode3_MODID(p,ii[0],ii[1],ii[2]-2,field)]-wmod[encode3_MODID(p,ii[0],ii[1],ii[2]+2,field)])/5.0-(wmod[encode3_MODID(p,ii[0],ii[1],ii[2]-3,field)]-wmod[encode3_MODID(p,ii[0],ii[1],ii[2]+3,field)])/15.0)/2.0))/(2.0*(p->dx[2]))    );
 else 
#endif
if( ii[2] >1 &&  ii[2]<((p->n[2])-2))
	grad=(  ( ((8*wmod[encode3_MODID(p,ii[0],ii[1],ii[2]+1,field)]-8*wmod[encode3_MODID(p,ii[0],ii[1],ii[2]-1,field)]+wmod[encode3_MODID(p,ii[0],ii[1],ii[2]-2,field)]-wmod[encode3_MODID(p,ii[0],ii[1],ii[2]+2,field)])/6.0))/(2.0*(p->dx[2]))    );

   if((ii[2]==(p->n[2])-3) || (ii[2]==(p->n[2])-4)  && ii[0]>1   && ii[0]<(p->n[0])-2 && ii[1]>1   && ii[1]<(p->n[1])-2  )
       grad=0;
   else if(ii[2]==2 || ii[2]==3  && ii[0]>1   && ii[0]<(p->n[0])-2 && ii[1]>1   && ii[1]<(p->n[1])-2  )
       grad=0;
#endif
   break;

}



 return grad;


}

__device__ __host__
real grad1l3_MODID(real *wmod,struct params *p,int *ii,int field,int dir)
{
 real grad=0;
   int i,j,k;
   i=ii[0];
   j=ii[1];
   k=0;
   #ifdef USE_SAC_3D
    k=ii[2];
   #endif


 if((dir == 0) && i>0 && i<((p->n[0])))
 {
    grad=(  ( wmod[encode3_MODID(p,i,j,k,field)]-wmod[encode3_MODID(p,i-1,j,k,field)]) /((p->dx[0]))    );

   #ifdef USE_SAC_3D
	   if((i==(p->n[0])-2) || (i==(p->n[0])-3)  && j>0   && j<(p->n[1])-1 && k>1   && k<(p->n[2])-1 )
	       grad=0;
	   else if(i==1 || i==2  && j>0   && j<(p->n[1])-1 && k>0   && k<(p->n[2])-1 )
	       grad=0;
   #else
	   if((i==(p->n[0])-2) || (i==(p->n[0])-3)  && j>0   && j<(p->n[1])-1  )
	       grad=0;
	   else if(i==1 || i==2  && j>0   && j<(p->n[1])-1  )
	       grad=0;
   #endif
 }
 else if((dir == 1)    && j>0 && j<((p->n[1])))
 {
    grad=(  ( wmod[encode3_MODID(p,i,j,k,field)]-wmod[encode3_MODID(p,i,j-1,k,field)])/((p->dx[1]))    );
   #ifdef USE_SAC_3D
	   if((j==(p->n[1])-2) || (j==(p->n[1])-3)  && i>0   && i<(p->n[0])-1 && k>0   && k<(p->n[2])-1 )
	       grad=0;
	   else if(j==1 || j==2  && i>0   && i<(p->n[0])-1 && k>0   && k<(p->n[2])-1 )
	       grad=0;
   #else
	   if((j==(p->n[1])-2) || (j==(p->n[1])-3)  && i>0   && i<(p->n[0])-1  )
	       grad=0;
	   else if(j==1 || j==2  && i>0   && i<(p->n[0])-1  )
	       grad=0;
   #endif


  }
   #ifdef USE_SAC_3D
 else if((dir == 2)    && k>0 && k<((p->n[2])))
 {
    grad=(  ( wmod[encode3_MODID(p,i,j,k,field)]-wmod[encode3_MODID(p,i,j,k-1,field)])/((p->dx[2]))    );

   if((k==(p->n[2])-2) || (k==(p->n[2])-3)  && i>0   && i<(p->n[0])-1  && j>0   && j<(p->n[1])-1 )
       grad=0;
   else if(k==1 || k==2  && i>0   && i<(p->n[0])-1  && j>0   && j<(p->n[1])-1 )
       grad=0;


  }
  #endif
 return grad;

}

__device__ __host__
real grad1r3_MODID(real *wmod,struct params *p,int *ii,int field,int dir)
{
  real grad=0;
   int i,j,k;
   i=ii[0];
   j=ii[1];
   k=0;
   #ifdef USE_SAC_3D
    k=ii[2];
   #endif


 if((dir == 0) && /*i>0 &&*/ i<((p->n[0])-1))
 {
    grad=(  ( wmod[encode3_MODID(p,i+1,j,k,field)]-wmod[encode3_MODID(p,i,j,k,field)]) /((p->dx[0]))    );

   #ifdef USE_SAC_3D
	   if((i==(p->n[0])-2) || (i==(p->n[0])-3)  && j>0   && j<(p->n[1])-1 && k>0   && k<(p->n[2])-1 )
	       grad=0;
	   else if(i==1 || i==2  && j>0   && j<(p->n[1])-1 && k>0   && k<(p->n[2])-1 )
	       grad=0;
   #else
	   if((i==(p->n[0])-2) || (i==(p->n[0])-3)  && j>0   && j<(p->n[1])-1  )
	       grad=0;
	   else if(i==1 || i==2  && j>0   && j<(p->n[1])-1  )
	       grad=0;
   #endif
 }
 else if((dir == 1)    /*&& j>0*/ && j<((p->n[1])-1))
 {
    grad=(  ( wmod[encode3_MODID(p,i,j+1,k,field)]-wmod[encode3_MODID(p,i,j,k,field)])/((p->dx[1]))    );
   #ifdef USE_SAC_3D
	   if((j==(p->n[1])-2) || (j==(p->n[1])-3)  && i>0   && i<(p->n[0])-1 && k>0   && k<(p->n[2])-1 )
	       grad=0;
	   else if(j==1 || j==2  && i>0   && i<(p->n[0])-1 && k>0   && k<(p->n[2])-1 )
	       grad=0;
   #else
	   if((j==(p->n[1])-2) || (j==(p->n[1])-3)  && i>0   && i<(p->n[0])-1  )
	       grad=0;
	   else if(j==1 || j==2  && i>0   && i<(p->n[0])-1  )
	       grad=0;
   #endif


  }
   #ifdef USE_SAC_3D
 else if((dir == 2)    /*&& k>0*/ && k<((p->n[2])-1))
 {
    grad=(  ( wmod[encode3_MODID(p,i,j,k+1,field)]-wmod[encode3_MODID(p,i,j,k,field)])/((p->dx[2]))    );

   if((k==(p->n[2])-2) || (k==(p->n[2])-3)  && i>0   && i<(p->n[0])-1  && j>0   && j<(p->n[1])-1 )
       grad=0;
   else if(k==1 || k==2  && i>0   && i<(p->n[0])-1  && j>0   && j<(p->n[1])-1 )
       grad=0;


  }
  #endif
 return grad;
}



__device__ __host__
real grad13_MODID(real *wmod,struct params *p,int *ii,int field,int dir)
{
  real grad=0;
   int i,j,k;
   i=ii[0];
   j=ii[1];
   k=0;
   #ifdef USE_SAC_3D
    k=ii[2];
   #endif


 if((dir == 0) && i>0 && i<((p->n[0])-1))
 {
    grad=(  ( wmod[encode3_MODID(p,i+1,j,k,field)]-wmod[encode3_MODID(p,i-1,j,k,field)]) /((p->dx[0]))/2.0    );

   #ifdef USE_SAC_3D
	   if((i==(p->n[0])-2) || (i==(p->n[0])-3)  && j>0   && j<(p->n[1])-1 && k>0   && k<(p->n[2])-1 )
	       grad=0;
	   else if(i==1 || i==2  && j>0   && j<(p->n[1])-1 && k>0   && k<(p->n[2])-1 )
	       grad=0;
   #else
	   if((i==(p->n[0])-2) || (i==(p->n[0])-3)  && j>0   && j<(p->n[1])-1  )
	       grad=0;
	   else if(i==1 || i==2  && j>0   && j<(p->n[1])-1  )
	       grad=0;
   #endif
 }
 else if((dir == 1)    && j>0 && j<((p->n[1])-1))
 {
    grad=(  ( wmod[encode3_MODID(p,i,j+1,k,field)]-wmod[encode3_MODID(p,i,j-1,k,field)])/((p->dx[1]))/2.0    );
   #ifdef USE_SAC_3D
	   if((j==(p->n[1])-2) || (j==(p->n[1])-3)  && i>0   && i<(p->n[0])-1 && k>0   && k<(p->n[2])-1 )
	       grad=0;
	   else if(j==1 || j==2  && i>0   && i<(p->n[0])-1 && k>0   && k<(p->n[2])-1 )
	       grad=0;
   #else
	   if((j==(p->n[1])-2) || (j==(p->n[1])-3)  && i>0   && i<(p->n[0])-1  )
	       grad=0;
	   else if(j==1 || j==2  && i>0   && i<(p->n[0])-1  )
	       grad=0;
   #endif


  }
   #ifdef USE_SAC_3D
 else if((dir == 2)    && k>0 && k<((p->n[2])-1))
 {
    grad=(  ( wmod[encode3_MODID(p,i,j,k+1,field)]-wmod[encode3_MODID(p,i,j,k-1,field)])/((p->dx[2]))/2.0    );

   if((k==(p->n[2])-2) || (k==(p->n[2])-3)  && i>0   && i<(p->n[0])-1  && j>0   && j<(p->n[1])-1 )
       grad=0;
   else if(k==1 || k==2  && i>0   && i<(p->n[0])-1  && j>1   && j<(p->n[1])-1 )
       grad=0;


  }
  #endif
 return grad;
}




/*****************************************************/




__device__ __host__
real grad3dn_MODID(real *wmod, real *wd,struct params *p,int *ii,int field,int dir)
{


 real grad=0;

 

 switch(dir)
 {
   case 0:
 
#ifdef USE_SAC_3D
  #ifdef USE_DORDER3
 if(ii[0]>2 && ii[0]<((p->n[0])-3) )
  grad=(  ( ((3*wmod[encode3_MODID(p,ii[0]+1,ii[1],ii[2],field)]-3*wmod[encode3_MODID(p,ii[0]-1,ii[1],ii[2],field)]+3.0*(wmod[encode3_MODID(p,ii[0]-2,ii[1],ii[2],field)]-wmod[encode3_MODID(p,ii[0]+2,ii[1],ii[2],field)])/5.0-(wmod[encode3_MODID(p,ii[0]-3,ii[1],ii[2],field)]-wmod[encode3_MODID(p,ii[0]+3,ii[1],ii[2],field)])/15.0)/2.0))/(2.0*(wd[fencode3_MODID(p,ii,delx1)]))    );

  #else
if(ii[0]>1 && ii[0]<((p->n[0])-2) )
 grad=(  ( ((8*wmod[encode3_MODID(p,ii[0]+1,ii[1],ii[2],field)]-8*wmod[encode3_MODID(p,ii[0]-1,ii[1],ii[2],field)]+wmod[encode3_MODID(p,ii[0]-2,ii[1],ii[2],field)]-wmod[encode3_MODID(p,ii[0]+2,ii[1],ii[2],field)])/6.0))/(2.0*(wd[fencode3_MODID(p,ii,delx1)]))    );
 #endif

#ifdef USE_MPI
if(p->boundtype[field][dir][0] !=1  )
  if(p->mpiupperb[dir]==1  )
#else
if(p->boundtype[field][dir][0] !=0  )
#endif
{

  ;//for OZT test using MPI use this directive further clarification needed
  #ifndef USE_MPI
   if((ii[0]==(p->n[0])-3) || (ii[0]==(p->n[0])-4)  && ii[1]>1   && ii[1]<(p->n[1])-2 && ii[2]>1   && ii[2]<(p->n[2])-2  )
       grad=0;
   else if(ii[0]==2 || ii[0]==3  && ii[1]>1   && ii[1]<(p->n[1])-2 && ii[2]>1   && ii[2]<(p->n[2])-2  )
       grad=0;
  #endif
}
#else  

  #ifdef USE_DORDER3
if(ii[0]>2 && ii[0]<((p->n[0])-3) )
 grad=(  ( ((3*wmod[encode3_MODID(p,ii[0]+1,ii[1],0,field)]-3*wmod[encode3_MODID(p,ii[0]-1,ii[1],0,field)]+3.0*(wmod[encode3_MODID(p,ii[0]-2,ii[1],0,field)]-wmod[encode3_MODID(p,ii[0]+2,ii[1],0,field)])/5.0-(wmod[encode3_MODID(p,ii[0]-3,ii[1],0,field)]-wmod[encode3_MODID(p,ii[0]+3,ii[1],0,field)])/15.0)/2.0))/(2.0*(wd[fencode3_MODID(p,ii,delx1)]))    );

  #else
if(ii[0]>1 && ii[0]<((p->n[0])-2) )
 grad=(  ( ((8*wmod[encode3_MODID(p,ii[0]+1,ii[1],0,field)]-8*wmod[encode3_MODID(p,ii[0]-1,ii[1],0,field)]+wmod[encode3_MODID(p,ii[0]-2,ii[1],0,field)]-wmod[encode3_MODID(p,ii[0]+2,ii[1],0,field)])/6.0))/(2.0*(wd[fencode3_MODID(p,ii,delx1)]))    );
 #endif
#ifdef USE_MPI
if(p->boundtype[field][dir][0] !=1  )
  if(p->mpiupperb[dir]==1  )
#else
if(p->boundtype[field][dir][0] !=0  )
#endif
{

  ;//for OZT test using MPI use this directive further clarification needed
  #ifndef USE_MPI
   if((ii[0]==(p->n[0])-3) || (ii[0]==(p->n[0])-4)  && ii[1]>1   && ii[1]<(p->n[1])-2  )
       grad=0;
   else if(ii[0]==2 || ii[0]==3  && ii[1]>1   && ii[1]<(p->n[1])-2  )
       grad=0;
  #endif
}
#endif



   break;

   case 1:

#ifdef USE_SAC_3D

  #ifdef USE_DORDER3
 if(ii[1]>2 && ii[1]<((p->n[1])-3) )
  grad=(  ( ((3*wmod[encode3_MODID(p,ii[0],ii[1]+1,ii[2],field)]-3*wmod[encode3_MODID(p,ii[0],ii[1]-1,ii[2],field)]+3.0*(wmod[encode3_MODID(p,ii[0],ii[1]-2,ii[2],field)]-wmod[encode3_MODID(p,ii[0],ii[1]+2,ii[2],field)])/5.0-(wmod[encode3_MODID(p,ii[0],ii[1]-3,ii[2],field)]-wmod[encode3_MODID(p,ii[0],ii[1]+3,ii[2],field)])/15.0)/2.0))/(2.0*(wd[fencode3_MODID(p,ii,delx2)]))    );

#else
if( ii[1] >1 &&  ii[1]<((p->n[1])-2))
	grad=(  ( ((8*wmod[encode3_MODID(p,ii[0],ii[1]+1,ii[2],field)]-8*wmod[encode3_MODID(p,ii[0],ii[1]-1,ii[2],field)]+wmod[encode3_MODID(p,ii[0],ii[1]-2,ii[2],field)]-wmod[encode3_MODID(p,ii[0],ii[1]+2,ii[2],field)])/6.0))/(2.0*(wd[fencode3_MODID(p,ii,delx2)]))    );
 #endif
#ifdef USE_MPI
if(p->boundtype[field][dir][0] !=1  )
  if(p->mpiupperb[dir]==1  )
#else
if(p->boundtype[field][dir][0] !=0  )
#endif
{
  ;//for OZT test using MPI use this directive further clarification needed
  #ifndef USE_MPI
   if((ii[1]==(p->n[1])-3) || (ii[1]==(p->n[1])-4)  && ii[0]>1   && ii[0]<(p->n[0])-2  && ii[2]>1   && ii[2]<(p->n[2])-2  )
       grad=0;
   else if(ii[1]==2 || ii[1]==3  && ii[0]>1   && ii[0]<(p->n[0])-2  && ii[2]>1   && ii[2]<(p->n[2])-2  )
       grad=0;
  #endif
}
#else

  #ifdef USE_DORDER3
if(ii[1]>2 && ii[1]<((p->n[1])-3) )
 grad=(  ( ((3*wmod[encode3_MODID(p,ii[0],ii[1]+1,0,field)]-3*wmod[encode3_MODID(p,ii[0],ii[1]-1,0,field)]+3.0*(wmod[encode3_MODID(p,ii[0],ii[1]-2,0,field)]-wmod[encode3_MODID(p,ii[0],ii[1]+2,0,field)])/5.0-(wmod[encode3_MODID(p,ii[0],ii[1]-3,0,field)]-wmod[encode3_MODID(p,ii[0],ii[1]+3,0,field)])/15.0)/2.0))/(2.0*(wd[fencode3_MODID(p,ii,delx2)]))    );

#endif
if( ii[1] >1 &&  ii[1]<((p->n[1])-2))
	grad=(  ( ((8*wmod[encode3_MODID(p,ii[0],ii[1]+1,0,field)]-8*wmod[encode3_MODID(p,ii[0],ii[1]-1,0,field)]+wmod[encode3_MODID(p,ii[0],ii[1]-2,0,field)]-wmod[encode3_MODID(p,ii[0],ii[1]+2,0,field)])/6.0))/(2.0*(wd[fencode3_MODID(p,ii,delx2)]))    );

#ifdef USE_MPI
if(p->boundtype[field][dir][0] !=1  )
  if(p->mpiupperb[dir]==1  )
#else
if(p->boundtype[field][dir][0] !=0  )
#endif
{

  ;//for OZT test using MPI use this directive further clarification needed
  #ifndef USE_MPI
   if((ii[1]==(p->n[1])-3) || (ii[1]==(p->n[1])-4)  && ii[0]>1   && ii[0]<(p->n[0])-2  )
       grad=0;
   else if(ii[1]==2 || ii[1]==3  && ii[0]>1   && ii[0]<(p->n[0])-2  )
       grad=0;
  #endif
}
#endif
   break;


   case 2:

#ifdef USE_SAC_3D
  #ifdef USE_DORDER3
 if(ii[2]>2 && ii[2]<((p->n[2])-3) )
  grad=(  ( ((3*wmod[encode3_MODID(p,ii[0],ii[1],ii[2]+1,field)]-3*wmod[encode3_MODID(p,ii[0],ii[1],ii[2]-1,field)]+3.0*(wmod[encode3_MODID(p,ii[0],ii[1],ii[2]-2,field)]-wmod[encode3_MODID(p,ii[0],ii[1],ii[2]+2,field)])/5.0-(wmod[encode3_MODID(p,ii[0],ii[1],ii[2]-3,field)]-wmod[encode3_MODID(p,ii[0],ii[1],ii[2]+3,field)])/15.0)/2.0))/(2.0*(wd[fencode3_MODID(p,ii,delx3)]))    );

#else
if( ii[2] >1 &&  ii[2]<((p->n[2])-2))
	grad=(  ( ((8*wmod[encode3_MODID(p,ii[0],ii[1],ii[2]+1,field)]-8*wmod[encode3_MODID(p,ii[0],ii[1],ii[2]-1,field)]+wmod[encode3_MODID(p,ii[0],ii[1],ii[2]-2,field)]-wmod[encode3_MODID(p,ii[0],ii[1],ii[2]+2,field)])/6.0))/(2.0*(wd[fencode3_MODID(p,ii,delx3)]))    );
#endif

#ifdef USE_MPI
if(p->boundtype[field][dir][0] !=1  )
  if(p->mpiupperb[dir]==1  )
#else
if(p->boundtype[field][dir][0] !=0  )
#endif
{

  ;//for OZT test using MPI use this directive further clarification needed
  #ifndef USE_MPI
   if((ii[2]==(p->n[2])-3) || (ii[2]==(p->n[2])-4)  && ii[0]>1   && ii[0]<(p->n[0])-2 && ii[1]>1   && ii[1]<(p->n[1])-2  )
       grad=0;
   else if(ii[2]==2 || ii[2]==3  && ii[0]>1   && ii[0]<(p->n[0])-2 && ii[1]>1   && ii[1]<(p->n[1])-2  )
       grad=0;
  #endif
}
#endif
   break;

}



 return grad;


}




__device__ __host__
real grad1l3n_MODID(real *wmod, real *wd,struct params *p,int *ii,int field,int dir)
{
 real grad=0;
   int i,j,k;
   i=ii[0];
   j=ii[1];
   k=0;
   #ifdef USE_SAC_3D
    k=ii[2];
   #endif


 if((dir == 0) && i>0 && i<((p->n[0])))
 {
    grad=(  ( wmod[encode3_MODID(p,i,j,k,field)]-wmod[encode3_MODID(p,i-1,j,k,field)]) /((wd[fencode3_MODID(p,ii,delx1)]))    );

   if(p->boundtype[field][dir][0] !=0)
	{
	   #ifdef USE_SAC_3D
		   if((i==(p->n[0])-2) || (i==(p->n[0])-3)  && j>0   && j<(p->n[1])-1 && k>1   && k<(p->n[2])-1 )
		       grad=0;
		   else if(i==1 || i==2  && j>0   && j<(p->n[1])-1 && k>0   && k<(p->n[2])-1 )
		       grad=0;
	   #else
		   if((i==(p->n[0])-2) || (i==(p->n[0])-3)  && j>0   && j<(p->n[1])-1  )
		       grad=0;
		   else if(i==1 || i==2  && j>0   && j<(p->n[1])-1  )
		       grad=0;
	   #endif
	}
 }
 else if((dir == 1)    && j>0 && j<((p->n[1])))
 {
    grad=(  ( wmod[encode3_MODID(p,i,j,k,field)]-wmod[encode3_MODID(p,i,j-1,k,field)])/((wd[fencode3_MODID(p,ii,delx2)]))    );

  if(p->boundtype[field][dir][0] !=0)
  {
   #ifdef USE_SAC_3D
	   if((j==(p->n[1])-2) || (j==(p->n[1])-3)  && i>0   && i<(p->n[0])-1 && k>0   && k<(p->n[2])-1 )
	       grad=0;
	   else if(j==1 || j==2  && i>0   && i<(p->n[0])-1 && k>0   && k<(p->n[2])-1 )
	       grad=0;
   #else
	   if((j==(p->n[1])-2) || (j==(p->n[1])-3)  && i>0   && i<(p->n[0])-1  )
	       grad=0;
	   else if(j==1 || j==2  && i>0   && i<(p->n[0])-1  )
	       grad=0;
   #endif
   }

  }
   #ifdef USE_SAC_3D
 else if((dir == 2)    && k>0 && k<((p->n[2])))
 {
    grad=(  ( wmod[encode3_MODID(p,i,j,k,field)]-wmod[encode3_MODID(p,i,j,k-1,field)])/((wd[fencode3_MODID(p,ii,delx3)]))    );

 if(p->boundtype[field][dir][0] !=0)
 {
   if((k==(p->n[2])-2) || (k==(p->n[2])-3)  && i>0   && i<(p->n[0])-1  && j>0   && j<(p->n[1])-1 )
       grad=0;
   else if(k==1 || k==2  && i>0   && i<(p->n[0])-1  && j>0   && j<(p->n[1])-1 )
       grad=0;
 }


  }
  #endif
 return grad;

}

__device__ __host__
real grad1r3n_MODID(real *wmod, real *wd,struct params *p,int *ii,int field,int dir)
{
  real grad=0;
   int i,j,k;
   i=ii[0];
   j=ii[1];
   k=0;
   #ifdef USE_SAC_3D
    k=ii[2];
   #endif


 if((dir == 0) && /*i>0 &&*/ i<((p->n[0])-1))
 {
    grad=(  ( wmod[encode3_MODID(p,i+1,j,k,field)]-wmod[encode3_MODID(p,i,j,k,field)]) /((wd[fencode3_MODID(p,ii,delx1)]))    );


   if(p->boundtype[field][dir][0] !=0)
   {
   #ifdef USE_SAC_3D
	   if((i==(p->n[0])-2) || (i==(p->n[0])-3)  && j>0   && j<(p->n[1])-1 && k>0   && k<(p->n[2])-1 )
	       grad=0;
	   else if(i==1 || i==2  && j>0   && j<(p->n[1])-1 && k>0   && k<(p->n[2])-1 )
	       grad=0;
   #else
	   if((i==(p->n[0])-2) || (i==(p->n[0])-3)  && j>0   && j<(p->n[1])-1  )
	       grad=0;
	   else if(i==1 || i==2  && j>0   && j<(p->n[1])-1  )
	       grad=0;
   #endif
   }
 }
 else if((dir == 1)    /*&& j>0*/ && j<((p->n[1])-1))
 {
    grad=(  ( wmod[encode3_MODID(p,i,j+1,k,field)]-wmod[encode3_MODID(p,i,j,k,field)])/((wd[fencode3_MODID(p,ii,delx2)]))    );


  if(p->boundtype[field][dir][0] !=0)
  {
   #ifdef USE_SAC_3D
	   if((j==(p->n[1])-2) || (j==(p->n[1])-3)  && i>0   && i<(p->n[0])-1 && k>0   && k<(p->n[2])-1 )
	       grad=0;
	   else if(j==1 || j==2  && i>0   && i<(p->n[0])-1 && k>0   && k<(p->n[2])-1 )
	       grad=0;
   #else
	   if((j==(p->n[1])-2) || (j==(p->n[1])-3)  && i>0   && i<(p->n[0])-1  )
	       grad=0;
	   else if(j==1 || j==2  && i>0   && i<(p->n[0])-1  )
	       grad=0;
   #endif
   }


  }
   #ifdef USE_SAC_3D
 else if((dir == 2)    /*&& k>0*/ && k<((p->n[2])-1))
 {
    grad=(  ( wmod[encode3_MODID(p,i,j,k+1,field)]-wmod[encode3_MODID(p,i,j,k,field)])/((wd[fencode3_MODID(p,ii,delx3)]))    );

if(p->boundtype[field][dir][0] !=0)
 {
   if((k==(p->n[2])-2) || (k==(p->n[2])-3)  && i>0   && i<(p->n[0])-1  && j>0   && j<(p->n[1])-1 )
       grad=0;
   else if(k==1 || k==2  && i>0   && i<(p->n[0])-1  && j>0   && j<(p->n[1])-1 )
       grad=0;
  }

  }
  #endif
 return grad;
}



__device__ __host__
real grad13n_MODID(real *wmod, real *wd,struct params *p,int *ii,int field,int dir)
{
  real grad=0;
   int i,j,k;
   i=ii[0];
   j=ii[1];
   k=0;
   #ifdef USE_SAC_3D
    k=ii[2];
   #endif


 if((dir == 0) && i>0 && i<((p->n[0])-1))
 {
    grad=(  ( wmod[encode3_MODID(p,i+1,j,k,field)]-wmod[encode3_MODID(p,i-1,j,k,field)]) /((wd[fencode3_MODID(p,ii,delx1)]))/2.0    );


	if(p->boundtype[field][dir][0] !=0)
	{
	   #ifdef USE_SAC_3D
		   if((i==(p->n[0])-2) || (i==(p->n[0])-3)  && j>0   && j<(p->n[1])-1 && k>0   && k<(p->n[2])-1 )
		       grad=0;
		   else if(i==1 || i==2  && j>0   && j<(p->n[1])-1 && k>0   && k<(p->n[2])-1 )
		       grad=0;
	   #else
		   if((i==(p->n[0])-2) || (i==(p->n[0])-3)  && j>0   && j<(p->n[1])-1  )
		       grad=0;
		   else if(i==1 || i==2  && j>0   && j<(p->n[1])-1  )
		       grad=0;
	   #endif
	}
 }
 else if((dir == 1)    && j>0 && j<((p->n[1])-1))
 {
    grad=(  ( wmod[encode3_MODID(p,i,j+1,k,field)]-wmod[encode3_MODID(p,i,j-1,k,field)])/((wd[fencode3_MODID(p,ii,delx2)]))/2.0    );

	if(p->boundtype[field][dir][0] !=0)
	{
	   #ifdef USE_SAC_3D
		   if((j==(p->n[1])-2) || (j==(p->n[1])-3)  && i>0   && i<(p->n[0])-1 && k>0   && k<(p->n[2])-1 )
		       grad=0;
		   else if(j==1 || j==2  && i>0   && i<(p->n[0])-1 && k>0   && k<(p->n[2])-1 )
		       grad=0;
	   #else
		   if((j==(p->n[1])-2) || (j==(p->n[1])-3)  && i>0   && i<(p->n[0])-1  )
		       grad=0;
		   else if(j==1 || j==2  && i>0   && i<(p->n[0])-1  )
		       grad=0;
	   #endif
	}

  }
   #ifdef USE_SAC_3D
 else if((dir == 2)    && k>0 && k<((p->n[2])-1))
 {
    grad=(  ( wmod[encode3_MODID(p,i,j,k+1,field)]-wmod[encode3_MODID(p,i,j,k-1,field)])/((wd[fencode3_MODID(p,ii,delx3)]))/2.0    );

	if(p->boundtype[field][dir][0] !=0)
	{
	   if((k==(p->n[2])-2) || (k==(p->n[2])-3)  && i>0   && i<(p->n[0])-1  && j>0   && j<(p->n[1])-1 )
	       grad=0;
	   else if(k==1 || k==2  && i>0   && i<(p->n[0])-1  && j>1   && j<(p->n[1])-1 )
	       grad=0;
	}

  }
  #endif
 return grad;
}








__device__ __host__
void bc3_cont_MODID(real *wt, struct params *p,int *ii, int f) {

   
int i,j,k;
i=ii[0];
j=ii[1];
k=0;
        #ifdef USE_SAC_3D
          k=ii[2];

                /*if(i<2 && j<2  && k<2)
                {
                 if(i==j==k )
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,2,2,2,f)];
                }
                else if(i<2 && j>((p->n[1])-3)  &&  k<2)
                {
                  if(i==(j-(p->n[1]))  && k==(j-(p->n[1])))                  
                     wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,2,(p->n[1])-3,2,f)];                     
                }
                else if(i>((p->n[0])-3) && j<2 && k<2)
                {
                  if(j==(i-(p->n[0]))  && k==(i-(p->n[0])))                  
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,((p->n[0])-3),2,2,f)];                  
                }
                else if(i>((p->n[0])-3) && j>((p->n[1])-3) && k<2)
                {
                  if(i==j  && k==(i-(p->n[0])) )                  
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,((p->n[0])-3),((p->n[1])-3),2,f)];                                                  
                }
                else if(i<2 && j<2  && k>((p->n[2])-3))
                {
                 if(i==j && k==(i-(p->n[0]))  )
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,2,2,(p->n[2])-3,f)];
                } 
                else if(i>((p->n[0])-3) && j<2  && k>((p->n[2])-3))
                {
                 if(i==k && j==(i-(p->n[0]))  )
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,(p->n[0])-3,2,(p->n[2])-3,f)];
                }
                else if(i<2 && j>((p->n[1])-3)  && k>((p->n[2])-3) )
                {
                 if(j==k && i==(j-(p->n[1]))  )
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,2,(p->n[1])-3,(p->n[2])-3,f)];
                } 
                else if(i>((p->n[0])-3) && j>((p->n[1])-3)  && k>((p->n[2])-3) )
                {
                 if(i==j==k  )
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,(p->n[0])-3,(p->n[1])-3,(p->n[2])-3,f)];
                }                     
                else*/ if(i==0 || i==1  && ((p->boundtype[f][0][0])==3))                
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,2,j,k,f)];              
                else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)) && ((p->boundtype[f][0][0])==3))                
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,(p->n[0])-3,j,k,f)];                            
                else if(j==0 || j==1 && ((p->boundtype[f][1][0])==3))                
                   wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,2,k,f)];                    
                else if((j==((p->n[1])-1)) || (j==((p->n[1])-2))  && ((p->boundtype[f][1][0])==3))                
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,(p->n[1])-3,k,f)];
                else if(k==0 || k==1  && ((p->boundtype[f][2][0])==3))                
                   wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,2,k,f)];                    
                else if((k==((p->n[2])-1)) || (k==((p->n[2])-2))  && ((p->boundtype[f][2][0])==3))                
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,(p->n[2])-3,f)];

        #else
             /*if(i<2 && j<2)
                {
                  if(i==j)
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,2,j,f)];
                  else                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,2,f)];                  
                }
                else if(i<2 && j>((p->n[1])-3))
                {
                  if(i==(j-(p->n[1])))                  
                     wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,2,j,f)];                     
                  else                  
                     wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,((p->n[1])-3),f)];                   
                }
                else if(i>((p->n[0])-3) && j<2)
                {
                  if((i-(p->n[0]))==j)                  
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(i-3),j,f)];
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,((p->n[0])-3),j,f)];                  
                  else                  
                   // wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,j+2,f)];
                   wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,2,f)];                        
                }
                else if(i>((p->n[0])-3) && j>((p->n[1])-3))
                {
                  if(i==j)                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(i-3),j,f)];                   
                  else                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(j-3),f)];                  
                }                       
                else*/ if(i==0 || i==1 && ((p->boundtype[f][0][0])==3))                
   
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,2,j,f)];              
                else if((i==((p->n[0])-1)) || (i==((p->n[0])-2))  && ((p->boundtype[f][0][0])==3))                
                  //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(i-3),j,f)];    
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-3,j,f)];                            
                else if(j==0 || j==1  && ((p->boundtype[f][1][0])==3))                
                  //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,j+2,f)]; 
                   wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,2,f)];                    
                else if((j==((p->n[1])-1)) || (j==((p->n[1])-2))  && ((p->boundtype[f][1][0])==3))                
                  //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(j-3),f)];
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-3,f)];
                
         #endif



}

__device__ __host__
void bc3_cont_dir_MODID(real *wt, struct params *p,int *ii, int f, int dir) {

   
int i,j,k;
i=ii[0];
j=ii[1];
k=0;
        #ifdef USE_SAC_3D
          k=ii[2];

               /* if(i<2 && j<2  && k<2)
                {
                 if(i==j==k )
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,2,2,2,f)];
                }
                else if(i<2 && j>((p->n[1])-3)  &&  k<2)
                {
                  if(i==(j-(p->n[1]))  && k==(j-(p->n[1])))                  
                     wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,2,(p->n[1])-3,2,f)];                     
                }
                else if(i>((p->n[0])-3) && j<2 && k<2)
                {
                  if(j==(i-(p->n[0]))  && k==(i-(p->n[0])))                  
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,((p->n[0])-3),2,2,f)];                  
                }
                else if(i>((p->n[0])-3) && j>((p->n[1])-3) && k<2)
                {
                  if(i==j  && k==(i-(p->n[0])) )                  
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,((p->n[0])-3),((p->n[1])-3),2,f)];                                                  
                }
                else if(i<2 && j<2  && k>((p->n[2])-3) )
                {
                 if(i==j && k==(i-(p->n[0]))  )
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,2,2,(p->n[2])-3,f)];
                } 
                else if(i>((p->n[0])-3) && j<2  && k>((p->n[2])-3)  )
                {
                 if(i==k && j==(i-(p->n[0]))  )
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,(p->n[0])-3,2,(p->n[2])-3,f)];
                }
                else if(i<2 && j>((p->n[1])-3)  && k>((p->n[2])-3) )
                {
                 if(j==k && i==(j-(p->n[1]))  )
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,2,(p->n[1])-3,(p->n[2])-3,f)];
                } 
                else if(i>((p->n[0])-3) && j>((p->n[1])-3)  && k>((p->n[2])-3) )
                {
                 if(i==j==k  )
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,(p->n[0])-3,(p->n[1])-3,(p->n[2])-3,f)];
                }                     
                else*/ if((i==0 || i==1)  && dir==0 && ((p->boundtype[f][dir][0])==3))                
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,2,j,k,f)];              
                else if(((i==((p->n[0])-1)) || (i==((p->n[0])-2)))  && dir==0 && ((p->boundtype[f][dir][0])==3))                
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,(p->n[0])-3,j,k,f)];                            
                else if((j==0 || j==1)  && dir==1 && ((p->boundtype[f][dir][0])==3))                
                   wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,2,k,f)];                    
                else if(((j==((p->n[1])-1)) || (j==((p->n[1])-2)))  && dir==1  && ((p->boundtype[f][dir][0])==3))                
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,(p->n[1])-3,k,f)];
                else if(k==0 || k==1  && dir==2  && ((p->boundtype[f][dir][0])==3))                
                   wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,2,k,f)];                    
                else if((k==((p->n[2])-1)) || (k==((p->n[2])-2))  && dir==2  && ((p->boundtype[f][dir][0])==3))                
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,(p->n[2])-3,f)];

        #else
             /*if(i<2 && j<2)
                {
                  if(i==j)
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,2,j,f)];
                  else                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,2,f)];                  
                }
                else if(i<2 && j>((p->n[1])-3))
                {
                  if(i==(j-(p->n[1])))                  
                     wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,2,j,f)];                     
                  else                  
                     wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,((p->n[1])-3),f)];                   
                }
                else if(i>((p->n[0])-3) && j<2)
                {
                  if((i-(p->n[0]))==j)                  
                    //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(i-3),j,f)];
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,((p->n[0])-3),j,f)];                  
                  else                  
                   // wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,j+2,f)];
                   wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,2,f)];                        
                }
                else if(i>((p->n[0])-3) && j>((p->n[1])-3))
                {
                  if(i==j)                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(i-3),j,f)];                   
                  else                  
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(j-3),f)];                  
                }                       
                else */

                if((i==0 || i==1)  && dir==0 && ((p->boundtype[f][dir][0])==3))                
                  //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i+2,j,f)];   
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,2,j,f)];              
                else if(((i==((p->n[0])-1)) || (i==((p->n[0])-2)))  && dir==0  && ((p->boundtype[f][dir][0])==3))                
                  //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(i-2),j,f)];    
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-3,j,f)];                            
                else if((j==0 || j==1)  && dir==1  && ((p->boundtype[f][dir][0])==3))                
                  //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,j+2,f)]; 
                   wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,2,f)];                    
                else if(((j==((p->n[1])-1)) || (j==((p->n[1])-2)))  && dir==1  && ((p->boundtype[f][dir][0])==3))                
                  //wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(j-2),f)];
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-3,f)];
                
         #endif



}


__device__ __host__
void bc3_cont_cd4_MODID(real *wt, struct params *p,int *ii, int f) {


int i,j,k;
i=ii[0];
j=ii[1];
k=0;
        #ifdef USE_SAC_3D
          k=ii[2];
            if((p->boundtype[f][0][0])==4)
                if(i==0)              
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,4,j,k,f)];
                else if(i==1)                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,3,j,k,f)];
                else if( i==((p->n[0])-1))               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,(p->n[0])-5,j,k,f)];
                else if (i==((p->n[0])-2))                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,(p->n[0])-4,j,k,f)];
               

            if((p->boundtype[f][1][0])==4)
                if(j==0)               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,4,k,f)];
                else if(j==1)                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,3,k,f)];
                else if (j== ((p->n[1])-1))               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,(p->n[1])-5,k,f)];
               else if (j== ((p->n[1])-2))                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,(p->n[1])-4,k,f)];



            if((p->boundtype[f][2][0])==4)
                if(k==0)               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,4,f)];
                else if(k==1)                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,3,f)];
                else if (k== ((p->n[2])-1))               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,(p->n[2])-5,f)];
               else if (k== ((p->n[2])-2))                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,(p->n[2])-4,f)];
        #else
        if((p->boundtype[f][0][0])==4)   
                if(i==0)              
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,3,j,f)];
                else if(i==1)                
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,2,j,f)];
                else if( i==((p->n[0])-1))               
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-4,j,f)];
                else if (i==((p->n[0])-2))                
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-3,j,f)];
               

            if((p->boundtype[f][1][0])==4)
                if(j==0)               
                  // wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,4,f)];
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,3,f)];
                else if(j==1)                
                  //  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,3,f)];
                   wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,2,f)];
                else if (j== ((p->n[1])-1))               
                  //  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-5,f)];
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-4,f)];
               else if (j== ((p->n[1])-2))                
                  //  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-4,f)];
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-3,f)];
         #endif

}



__device__ __host__
void bc3_cont_cd4_dir_MODID(real *wt, struct params *p,int *ii, int f, int dir) {


int i,j,k;
i=ii[0];
j=ii[1];
k=0;
        #ifdef USE_SAC_3D

          k=ii[2];
                      if((p->boundtype[f][dir][0])==4)
                      {
                if((i==0 || i==1) && dir==0)              
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,4-i,j,k,f)];             
                else if((( i==((p->n[0])-1)   ))  && dir==0)               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i-4,j,k,f)];
                else if(((  i==((p->n[0])-2) ))  && dir==0)               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i-2,j,k,f)];
              

                if((j==0 || j==1) && dir==1)              
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,4-j,k,f)];             
                else if((( j==((p->n[1])-1)   ))  && dir==1)               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j-4,k,f)];
                else if(((  j==((p->n[1])-2) ))  && dir==1)               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j-2,k,f)];


                if((k==0 || k==1) && dir==2)              
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,4-k,f)];             
                else if((( k==((p->n[2])-1)   ))  && dir==2)               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,k-4,f)];
                else if(((  k==((p->n[2])-2) ))  && dir==2)               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,k-2,f)];

                    }
               
        #else

                          if((p->boundtype[f][dir][0])==4)
                          {
                if((i==0 || i==1) && dir==0)              
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,4-i,j,k,f)];             
                else if((( i==((p->n[0])-1)   ))  && dir==0)               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i-4,j,k,f)];
                else if(((  i==((p->n[0])-2) ))  && dir==0)               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i-2,j,k,f)];
              

                if((j==0 || j==1) && dir==1)              
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,4-j,k,f)];             
                else if((( j==((p->n[1])-1)   ))  && dir==1)               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j-4,k,f)];
                else if(((  j==((p->n[1])-2) ))  && dir==1)               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j-2,k,f)];
                    }
         #endif

}




__device__ __host__
void bc3_setfixed_dir_MODID(real *wt, struct params *p,struct bparams *bp,int *ii, int f,int dir) {


int i,j,k;
i=ii[0];
j=ii[1];
k=0;


        #ifdef USE_SAC_3D
          k=ii[2];
        #endif

          if((p->boundtype[f][dir][0])==5)   
                if(i==0 || i==1  && dir==0)                
                  bp->fixed1[encodefixed13_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,k,f)];                
                else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)) && dir==0 )                
                  bp->fixed1[encodefixed13_MODID(p,1+(p->n[0])-i,j,k,f)]=wt[encode3_MODID(p,i,j,k,f)];                
                else if(j==0 || j==1  && dir==1 )                
                  bp->fixed2[encodefixed23_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,k,f)];                
                else if((j==((p->n[1])-1)) || (j==((p->n[1])-2))  && dir==1)                
                  bp->fixed2[encodefixed23_MODID(p,i,1+(p->n[1])-j,k,f)]=wt[encode3_MODID(p,i,j,k,f)];
           #ifdef USE_SAC_3D
                else if(k==0 || k==1 && dir==2)                
                  bp->fixed3[encodefixed33_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,k,f)];                
                else if((k==((p->n[2])-1)) || (k==((p->n[2])-2))  && dir==2)                
                  bp->fixed3[encodefixed33_MODID(p,i,j,1+(p->n[2])-k,f)]=wt[encode3_MODID(p,i,j,k,f)];
           #endif

}


__device__ __host__
void bc3_fixed_dir_MODID(real *wt, struct params *p, struct bparams *bp,int *ii, int f,int dir) {


int i,j,k;
i=ii[0];
j=ii[1];
k=0;


        #ifdef USE_SAC_3D
          k=ii[2];
        #endif

     if((p->boundtype[f][dir][0])==5)         
                if(i==0 || i==1  && dir==0)                
                  wt[encode3_MODID(p,i,j,k,f)]=bp->fixed1[encodefixed13_MODID(p,i,j,k,f)];                
                else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)) && dir==0 )                
                  wt[encode3_MODID(p,i,j,k,f)]=bp->fixed1[encodefixed13_MODID(p,1+(p->n[0])-i,j,k,f)];                
                else if(j==0 || j==1  && dir==1 )                
                  wt[encode3_MODID(p,i,j,k,f)]=bp->fixed2[encodefixed23_MODID(p,i,j,k,f)];                
                else if((j==((p->n[1])-1)) || (j==((p->n[1])-2))  && dir==1)                
                  wt[encode3_MODID(p,i,j,k,f)]=bp->fixed2[encodefixed23_MODID(p,i,1+(p->n[1])-j,k,f)];
           #ifdef USE_SAC_3D
                else if(k==0 || k==1 && dir==2)                
                  wt[encode3_MODID(p,i,j,k,f)]=bp->fixed3[encodefixed33_MODID(p,i,j,k,f)];                
                else if((k==((p->n[2])-1)) || (k==((p->n[2])-2))  && dir==2)                
                  wt[encode3_MODID(p,i,j,k,f)]=bp->fixed3[encodefixed33_MODID(p,i,j,1+(p->n[2])-k,f)];
           #endif
               




}


__device__ __host__
void bc3_periodic1_dir_MODID(real *wt, struct params *p,int *ii, int f,int dir) {

int i,j,k;
i=ii[0];
j=ii[1];
k=0;
        #ifdef USE_SAC_3D
          k=ii[2];
          if((p->boundtype[f][dir][0])==0)   
                if((i==0 || i==1) && dir==0)                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,(p->n[0])-4+i,j,k,f)];
             
                else if(((i==((p->n[0])-1)) || (i==((p->n[0])-2))) && dir==0)                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,4-(p->n[0])+i,j,k,f)];

          if((p->boundtype[f][dir][0])==0)   
                if((j==0 || j==1) && dir==1)                
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,(p->n[1])-4+j,k,f)];

                else if(((j==((p->n[1])-1)) || (j==((p->n[1])-2))) && dir==1)                 
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,4-(p->n[1])+j,k,f)];

           if((p->boundtype[f][dir][0])==0)   
                if((k==0 || k==1) && dir==2)                
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,(p->n[2])-4+j,f)];

                else if(((k==((p->n[2])-1)) || (k==((p->n[2])-2))) && dir==2)                 
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,4-(p->n[2])+k,f)];
       #else


  	wt[encode3_MODID(p,i,j,k,f)]=(((i==0 || i==1  || i==((p->n[0])-1) || i==((p->n[0])-2)) && dir==0)?((i==0 || i==1) && dir==0)*wt[encode3_MODID(p,(p->n[0])-4+i,j,k,f)]+(((i==((p->n[0])-1)) || (i==((p->n[0])-2))) && dir==0)*wt[encode3_MODID(p,4-(p->n[0])+i,j,k,f)]:wt[encode3_MODID(p,i,j,k,f)]);
          // if((p->boundtype[f][dir][0])==0)   
            /*    if((i==0 || i==1) && dir==0   )                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,(p->n[0])-4+i,j,k,f)];

                else if(((i==((p->n[0])-1)) || (i==((p->n[0])-2))) && dir==0   )                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,4-(p->n[0])+i,j,k,f)];*/

          // if((p->boundtype[f][dir][0])==0)   
           /*     if((j==0 || j==1) && dir==1  )                
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,(p->n[1])-4+j,k,f)];

                else if(((j==((p->n[1])-1)) || (j==((p->n[1])-2))) && dir==1  )                 
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,4-(p->n[1])+j,k,f)];*/

       #endif


}


__device__ __host__
void bc3_symm_dir_MODID(real *wt, struct params *p,int *ii, int f,int dir) {

int i,j,k;
i=ii[0];
j=ii[1];
k=0;
        #ifdef USE_SAC_3D
          k=ii[2];
                    if((p->boundtype[f][dir][0])==6)   
                if(i==0  && dir==0)              
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,4,j,k,f)];
                else if(i==1 && dir==0)                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,3,j,k,f)];
                else if( i==((p->n[0])-1)  && dir==0)               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,(p->n[0])-5,j,k,f)];
                else if (i==((p->n[0])-2)  && dir==0)                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,(p->n[0])-4,j,k,f)];
               

          if((p->boundtype[f][dir][0])==6)   
                if(j==0  && dir==1)               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,4,k,f)];
                else if(j==1  && dir==1)                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,3,k,f)];
                else if (j== ((p->n[1])-1)  && dir==1)               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,(p->n[1])-5,k,f)];
               else if (j== ((p->n[1])-2)  && dir==1)                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,(p->n[1])-4,k,f)];



          if((p->boundtype[f][dir][0])==6)   
                if(k==0 && dir==2)               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,4,f)];
                else if(k==1 && dir==2)                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,3,f)];
                else if (k== ((p->n[2])-1) && dir==2)               
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,(p->n[2])-5,f)];
               else if (k== ((p->n[2])-2) && dir==2)                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,(p->n[2])-4,f)];
        #else
        if((p->boundtype[f][dir][0])==6)   
                if(i==0  && dir==0)              
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,3,j,f)];
                else if(i==1  && dir==0)                
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,2,j,f)];
                else if( i==((p->n[0])-1)  && dir==0)               
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-4,j,f)];
                else if (i==((p->n[0])-2)  && dir==0)                
                    wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,(p->n[0])-3,j,f)];
               

          if((p->boundtype[f][dir][0])==6)   
                if(j==0  && dir==1)               
                  // wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,4,f)];
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,3,f)];
                else if(j==1  && dir==1)                
                  //  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,3,f)];
                   wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,2,f)];
                else if (j== ((p->n[1])-1)  && dir==1)               
                  //  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-5,f)];
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-4,f)];
               else if (j== ((p->n[1])-2)  && dir==1)                
                  //  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-4,f)];
                  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-3,f)];
         #endif


}


__device__ __host__
void bc3_asymm_dir_MODID(real *wt, struct params *p,int *ii, int f,int dir) {

int i,j,k;
i=ii[0];
j=ii[1];
k=0;
        #ifdef USE_SAC_3D
          k=ii[2];
          if((p->boundtype[f][dir][0])==7)   
                if(i==0  && dir==0)              
                    wt[encode3_MODID(p,i,j,k,f)]=-wt[encode3_MODID(p,4,j,k,f)];
                else if(i==1 && dir==0)                
                    wt[encode3_MODID(p,i,j,k,f)]=-wt[encode3_MODID(p,3,j,k,f)];
                else if( i==((p->n[0])-1)  && dir==0)               
                    wt[encode3_MODID(p,i,j,k,f)]=-wt[encode3_MODID(p,(p->n[0])-5,j,k,f)];
                else if (i==((p->n[0])-2)  && dir==0)                
                    wt[encode3_MODID(p,i,j,k,f)]=-wt[encode3_MODID(p,(p->n[0])-4,j,k,f)];
               

          if((p->boundtype[f][dir][0])==7)   
                if(j==0  && dir==1)               
                    wt[encode3_MODID(p,i,j,k,f)]=-wt[encode3_MODID(p,i,4,k,f)];
                else if(j==1  && dir==1)                
                    wt[encode3_MODID(p,i,j,k,f)]=-wt[encode3_MODID(p,i,3,k,f)];
                else if (j== ((p->n[1])-1)  && dir==1)               
                    wt[encode3_MODID(p,i,j,k,f)]=-wt[encode3_MODID(p,i,(p->n[1])-5,k,f)];
               else if (j== ((p->n[1])-2)  && dir==1)                
                    wt[encode3_MODID(p,i,j,k,f)]=-wt[encode3_MODID(p,i,(p->n[1])-4,k,f)];



          if((p->boundtype[f][dir][0])==7)   
                if(k==0 && dir==2)               
                    wt[encode3_MODID(p,i,j,k,f)]=-wt[encode3_MODID(p,i,j,4,f)];
                else if(k==1 && dir==2)                
                    wt[encode3_MODID(p,i,j,k,f)]=-wt[encode3_MODID(p,i,j,3,f)];
                else if (k== ((p->n[2])-1) && dir==2)               
                    wt[encode3_MODID(p,i,j,k,f)]=-wt[encode3_MODID(p,i,j,(p->n[2])-5,f)];
               else if (k== ((p->n[2])-2) && dir==2)                
                    wt[encode3_MODID(p,i,j,k,f)]=-wt[encode3_MODID(p,i,j,(p->n[2])-4,f)];
        #else
           if((p->boundtype[f][dir][0])==7)     
                if(i==0  && dir==0)              
                    wt[fencode_MODID(p,i,j,f)]=-wt[fencode_MODID(p,3,j,f)];
                else if(i==1  && dir==0)                
                    wt[fencode_MODID(p,i,j,f)]=-wt[fencode_MODID(p,2,j,f)];
                else if( i==((p->n[0])-1)  && dir==0)               
                    wt[fencode_MODID(p,i,j,f)]=-wt[fencode_MODID(p,(p->n[0])-4,j,f)];
                else if (i==((p->n[0])-2)  && dir==0)                
                    wt[fencode_MODID(p,i,j,f)]=-wt[fencode_MODID(p,(p->n[0])-3,j,f)];
               

          if((p->boundtype[f][dir][0])==7)   
                if(j==0  && dir==1)               
                  // wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,4,f)];
                  wt[fencode_MODID(p,i,j,f)]=-wt[fencode_MODID(p,i,3,f)];
                else if(j==1  && dir==1)                
                  //  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,3,f)];
                   wt[fencode_MODID(p,i,j,f)]=-wt[fencode_MODID(p,i,2,f)];
                else if (j== ((p->n[1])-1)  && dir==1)               
                  //  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-5,f)];
                  wt[fencode_MODID(p,i,j,f)]=-wt[fencode_MODID(p,i,(p->n[1])-4,f)];
               else if (j== ((p->n[1])-2)  && dir==1)                
                  //  wt[fencode_MODID(p,i,j,f)]=wt[fencode_MODID(p,i,(p->n[1])-4,f)];
                  wt[fencode_MODID(p,i,j,f)]=-wt[fencode_MODID(p,i,(p->n[1])-3,f)];
         #endif


}



__device__ __host__
void bc3_periodic1_MODID(real *wt, struct params *p,int *ii, int f) {

int i,j,k;
i=ii[0];
j=ii[1];
k=0;
        #ifdef USE_SAC_3D
          k=ii[2];
          if((p->boundtype[f][0][0])==0)   
                if(i==0 || i==1 )                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,(p->n[0])-4+i,j,k,f)];
             
                else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)))                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,4-(p->n[0])+i,j,k,f)];

          if((p->boundtype[f][1][0])==0)
                if(j==0 || j==1 )                
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,(p->n[1])-4+j,k,f)];

                else if((j==((p->n[1])-1)) || (j==((p->n[1])-2)) )                 
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,4-(p->n[1])+j,k,f)];
                  
          if((p->boundtype[f][2][0])==0)
                if(k==0 || k==1 )                
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,(p->n[2])-4+j,f)];

                else if((k==((p->n[2])-1)) || (k==((p->n[2])-2)) )                 
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,4-(p->n[2])+k,f)];
       #else
          if((p->boundtype[f][0][0])==0)
                if(i==0 || i==1 )                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,(p->n[0])-4+i,j,k,f)];

                else if((i==((p->n[0])-1)) || (i==((p->n[0])-2)))                
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,4-(p->n[0])+i,j,k,f)];

           if((p->boundtype[f][0][0])==0)
                if(j==0 || j==1 )                
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,(p->n[1])-4+j,k,f)];

                else if((j==((p->n[1])-1)) || (j==((p->n[1])-2)) )                 
                  wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,4-(p->n[1])+j,k,f)];

       #endif


}


__device__ __host__
void bc3_periodic2_MODID(real *wt, struct params *p,int *ii, int f) {

int i,j,k;
i=ii[0];
j=ii[1];
k=0;
        #ifdef USE_SAC_3D
          k=ii[2];
          
                if(i<2 && j<2  && k<2)
                {
                 if(i==j==k )
                   if((p->boundtype[f][1][0])==0)
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,(p->n[1])-4+j,k,f)];

                }
                else if(i<2 && j>((p->n[1])-3)  &&  k<2)
                {
                  if(i==(j-(p->n[1]))  && k==(j-(p->n[1])))
                  if((p->boundtype[f][1][0])==0)                  
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,(p->n[0])-4+i,j,k,f)];                                     
             
                }
                else if(i>((p->n[0])-3) && j<2 && k<2)
                {
                     if((p->boundtype[f][0][0])==0)
                  if(j==(i-(p->n[0]))  && k==(i-(p->n[0])))                  
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,4-(p->n[0])+i,j,k,f)];                                    
           
                }
                else if(i>((p->n[0])-3) && j>((p->n[1])-3) && k<2)
                {
                     if((p->boundtype[f][1][0])==0)
                  if(i==j  && k==(i-(p->n[0])))   
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,4-(p->n[1])+j,k,f)];                                    
                                 
                                            
                }
                else if(i<2 && j<2  && k>((p->n[2])-3))
                {
                     if((p->boundtype[f][2][0])==0)
                 if(i==j && i==(k-(p->n[2]))  )                 
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,j,(p->n[2])-4+k,f)];                                     

                } 
                else if(i>((p->n[0])-3) && j<2  && k>((p->n[2])-3))
                {
                     if((p->boundtype[f][0][0])==2)
                 if(i==k && j==(i-(p->n[0]))  )
                     wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,4-(p->n[0])+i,j,4-(p->n[2])+k,f)];   
                }
                else if(i<2 && j>((p->n[1])-3)  && k>((p->n[2])-3) )
                {
                     if((p->boundtype[f][2][0])==0)
                 if(j==k && i==(j-(p->n[1]))  )
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,4-(p->n[1])+j,4-(p->n[2])+k,f)]; 
                } 
                else if(i>((p->n[0])-3) && j>((p->n[1])-3)  && k>((p->n[2])-3) )
                {
                     if((p->boundtype[f][2][0])==0)
                 if(i==j==k  )
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,4-(p->n[0])+i,4-(p->n[1])+j,4-(p->n[2])+k,f)]; 
                }   

        #else

               if(i<2 && j<2)
                {
                      if((p->boundtype[f][0][0])==1)
                  if(i==j)
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,(p->n[1])-4+j,k,f)];
                  else              
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,(p->n[0])-4+i,j,k,f)];                                    
                }
                else if(i<2 && j>((p->n[1])-3))
                {
                     if((p->boundtype[f][0][0])==0)
                  if(i==(j-(p->n[1])))                  
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,(p->n[0])-4+i,j,k,f)];                                     
                  else                  
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,4-(p->n[1])+j,k,f)];                                     
                }
                else if(i>((p->n[0])-3) && j<2)
                {
                     if((p->boundtype[f][1][0])==0)
                  if((i-(p->n[0]))==j)                  
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,4-(p->n[0])+i,j,k,f)];                                    
                  else                  
                   wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,(p->n[1])-4+j,k,f)];                                    
                }
                else if(i>((p->n[0])-3) && j>((p->n[1])-3))
                {
                     if((p->boundtype[f][1][0])==0)
                  if(i==j)                  
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,i,4-(p->n[1])+j,k,f)];                                    
                  else                  
                    wt[encode3_MODID(p,i,j,k,f)]=wt[encode3_MODID(p,4-(p->n[0])+i,j,k,f)];                                    
                }                       
                 
       #endif         




}


__device__ __host__
real sacdabs_MODID(real val) {
   return(fabs(val));
}
