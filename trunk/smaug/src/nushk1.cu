#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/smaugcukernels.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_nshk.cuh"


__global__ void zeropadmaxdtvisc_parallel(struct params *p,   real *wmod, real *wd, int order, int dir, real *temp, int ndimp)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
 
  //if(iindex<ndimp)
  //    temp[iindex]=0.0;

  unsigned int tid = threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;
//        extern __shared__ real sdata[];
 
  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif

    
//int numBlocks = (dimp+tnumThreadsPerBlock-1) / tnumThreadsPerBlock;
  //real temp[dimp];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
   //sdata[tid]=0.0;
   // if(iindex<1024)
    //  temp[iindex]=0.0;

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif
    //int s=1;
    temp[iindex]=0.0;
     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
             temp[iindex]=(   (p->maxviscoef)+wd[encode3_nshk(p,ii[0],ii[1],ii[2],nushk1+dir)]     )/((wd[encode3_nshk(p,ii[0],ii[1],ii[2],delx1+dir)])*(wd[encode3_nshk(p,ii[0],ii[1],ii[2],delx1+dir)]));//temp[iindex]/(wd[fencode3_cdf(p,ii,delx1+dir)]);

//p->dtdiffvisc=0.25/((maxtmpdt)/((wd[encode3_nshk(p,ii[0],ii[1],ii[2],delx1+dim)])*(wd[encode3_nshk(p,ii[0],ii[1],ii[2],delx1+dim)])));
}


__global__ void newreduction0computemaxdtvisc_parallel(real *cmax, real *temp,int ndimp)
{
  //real *cmax, real *temp, int ndimp

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  extern __shared__ double partialResult[];

  int i;
   partialResult[tid]=0.0;
   if(iindex<ndimp)
              partialResult[tid]=temp[iindex];
  __syncthreads();


for(unsigned int s=1; s < blockDim.x; s *= 2) {
        if ((tid % (2*s)) == 0) {
            if(partialResult[tid+s]>partialResult[tid])
                 partialResult[tid]=partialResult[tid + s];
        }
        __syncthreads();
    }

    __syncthreads();
    if(tid==0)
    {
      cmax[blockIdx.x]=partialResult[0];
      temp[blockIdx.x]=partialResult[0];
     }
    __syncthreads();


}



__global__ void myreduction0computemaxdtvisc_parallel(struct params *p,   real *wmod, real *wd, int order, int dir, real *temp,int ndimp,int s)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;
//        extern __shared__ real sdata[];
 
  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif

int tnumThreadsPerBlock = 128;
    
int numBlocks = (dimp+tnumThreadsPerBlock-1) / tnumThreadsPerBlock;
  //real temp[dimp];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
   //sdata[tid]=0.0;
   // if(iindex<1024)
    //  temp[iindex]=0.0;

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif
    //int s=1;


   
    //while(((s*=2)<=((ndimp/2)-1)) && ((iindex+s)<ndimp)) {
    if((iindex+s)<ndimp)
            if(temp[iindex+s]>temp[iindex])
                 temp[iindex]=temp[iindex + s];
            
       // }

       //  __syncthreads();
    

   // __syncthreads();

   if(iindex==0  && (p->dtdiffvisc<temp[0]))
      p->dtdiffvisc=0.25/temp[0];




 
}



__global__ void getdtvisc_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int dim)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  real dtdiffvisc,tmpdt,maxtmpdt;
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  real maxt=0,max3=0, max1=0;
  
   int ip,jp;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  



int shift=order*NVAR*dimp;

   
if(iindex==0)
{
  maxtmpdt=0.0;
  
 // #ifdef USE_SHOCKVISC
    for(ii[0]=0;ii[0]<((p->n[0]));ii[0]++)
      for(ii[1]=0;ii[1]<((p->n[1]));ii[1]++)
     #ifdef USE_SAC_3D
        for(ii[2]=0;ii[2]<((p->n[2]));ii[2]++)
     #endif
 // #endif
	{              
           ;//  tmpdt=(p->maxviscoef)+wd[encode3_nshk(p,ii[0],ii[1],ii[2],nushk1+dim)];

              if(tmpdt>maxtmpdt)
                    maxtmpdt=tmpdt;
              p->dtdiffvisc=0.25/((maxtmpdt)/((wd[encode3_nshk(p,ii[0],ii[1],ii[2],delx1+dim)])*(wd[encode3_nshk(p,ii[0],ii[1],ii[2],delx1+dim)])));
	}

    //p->dtdiffvisc=0.25/((maxtmpdt)/((p->dx[dim])*(p->dx[dim])));                 
//wd[encode3_nshk(p,i,j,k,nushk1+dim)]
   


 //  }
}
 __syncthreads();



 
}





__global__ void nushk2_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int dim)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  real maxt=0,max3=0, max1=0;
  
   int ip,jp;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  


int shift=order*NVAR*dimp;

real cshk=0.5;




   //tmp1  tmp_nuI
 
//compute d3r and d1r
   //tmp2  d3r
    //tmp3 d1r


     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]>1 && ii[1]>1 && ii[2]>1 && ii[0]<p->n[0] && ii[1]<p->n[1]  && ii[2]<p->n[2])
     #else
       if(ii[0]>1 && ii[1]>1 && ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
 
   //if(i>1 && j>1 && i<((p->n[0])) && j<((p->n[1])))       
   { 

  ;//  wd[encode3_nshk(p,i,j,k,nushk1+dim)]=cshk*(p->dx[dim])*(p->dx[dim])*wtemp[encode3_nshk(p,i,j,k,tmp3)];
  ;//  if(wtemp[encode3_nshk(p,i,j,k,tmp3)]>=0.0)
  ;//      wd[encode3_nshk(p,i,j,k,nushk1+dim)]=0.0;
  ;//  wd[encode3_nshk(p,i,j,k,nushk1+dim)]=fabs(wd[encode3_nshk(p,i,j,k,nushk1+dim)]);
   }

   __syncthreads();




}



__global__ void nushk1a_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int dim)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  const int blockdim=blockDim.x;
  const int SZWT=blockdim;
  const int SZWM=blockdim*NVAR;
  int tid=threadIdx.x;
  int i,j,iv;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  real maxt=0,max3=0, max1=0;
  
   int ip,jp;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  



int shift=order*NVAR*dimp;
  __shared__ real wts[512];
  __shared__ real wms[512];





     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
    //set viscosities
   //if( i<((p->n[0])) && j<((p->n[1])))
   {
     #ifdef USE_SAC_3D
     wtemp[encode3_nshk(p,i,j,k,tmp3)]+=grad13_nshk(wtemp,p,ii,tmp1,dim);
     #else
     wtemp[encode3p2_nshk(p,i,j,0,tmp3)]+=grad13_nshk(wtemp,p,ii,tmp1,dim);
     #endif

   }

   
   __syncthreads();


 
}


__global__ void nushk1_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int dim)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  const int blockdim=blockDim.x;
  const int SZWT=blockdim;
  const int SZWM=blockdim*NVAR;
  int tid=threadIdx.x;
  real maxt=0,max3=0, max1=0;
  int i,j,iv;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];


  
   int ip,jp;



  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  



int bfac1,bfac2,bfac3;
//int bfac1=(field==rho || field>mom2)+(field>rho && field<energy);
//int bfac2= (field==rho || field>mom2);
//int bfac3=(field>rho && field<energy);
int shift=order*NVAR*dimp;
  __shared__ real wts[512];
  __shared__ real wms[512];




//init temp1 and temp2 to zero 
//the compute element initialising n[0] or n[1] element must do +1 and +2
//this is because we fit the problem geometrically to nixnj elements 


     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
    //set viscosities
   //if(i<((p->n[0])) && j<((p->n[1])))
   {


        for(int f=tmp1; f<=tmp8; f++)
                 wtemp[fencode3_nshk(p,ii,f)]=0;


   }



  

   __syncthreads();


 

     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
    //set viscosities
   //if(i<((p->n[0])) && j<((p->n[1])))
   {

        //for(iv=0;iv<NVAR;iv++)
        //               wms[tid+iv*blockdim]=wmod[fencode_nshk(p,i,j,iv)+shift];
        //wts[tid]=wtemp[fencode_nshk(p,i,j,tmp6)];
        //temp value for viscosity

       //tmp6  tmpnu


		wtemp[fencode3_nshk(p,ii,tmp1)]=wmod[fencode3_nshk(p,ii,mom1+dim)+shift]/(((wmod[fencode3_nshk(p,ii,rho)+shift] +wmod[fencode3_nshk(p,ii,rhob)+shift])));
      ;//   wd[fencode3_nshk(p,ii,nushk1+dim)]=0;       
        //wtemp2[encode3_nshk(p,i+1,j+1,k,tmpnui)]=wtemp[fencode3_nshk(p,ii,tmp6)];






        }
        //wtemp2[encode3_nshk(p,i+1,j+1,k+1,tmpnui)]=wtemp[fencode3_nshk(p,ii,tmp6)];





   


   __syncthreads();




}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_nshk(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cunushk1(struct params **p,  struct params **d_p,   real **d_wmod,  real **d_wd, int order, real **d_wtemp, real **d_wtemp1, real **d_wtemp2)
{

  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

// dim3 dimBlock(dimblock, 1);
 
 //   dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;

    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

     for(int dir=0;dir<NDIM;dir++)
     {
         nushk1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, dir);
         hipDeviceSynchronize();
         nushk1a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, dir);
         hipDeviceSynchronize();
     }
     
     for(int dir=0;dir<NDIM;dir++)
     {
         nushk2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, dir);
         hipDeviceSynchronize();
     }


}

int cugetdtvisc1(struct params **p,  struct params **d_p,   real **d_wmod,  real **wd, real **d_wd, int order, real **d_wtemp, real **d_wtemp1, real **d_wtemp2)
{

  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

  double *d_cmax;
  double *d_bmax;
  int i;


     real fn,fractn,in;
  int ndimp;
////hipSetDevice(selectedDevice);
   int nit=100;
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 


    fn=log(dimp)/log(2.0);
    fractn=modf(fn,&in);
    
    if(fractn>0)
    {
       fn+=1;
       ndimp=(int)pow(2,fn);
     }
     else
       ndimp=dimp;

       int NTPB=512;
  int numBlocks = (ndimp+NTPB-1) / NTPB;

  int smemSize = NTPB * sizeof(double);
 double *h_cmax = (double*)malloc(numBlocks*sizeof(double));

  hipMalloc((void**)&d_cmax, numBlocks*sizeof(double)); 
  hipMalloc((void**)&d_bmax, numBlocks*sizeof(double)); 




// dim3 dimBlock(dimblock, 1);
 
 //   dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 //  int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;
   
           /*for(int dim=0; dim<=(NDIM-1); dim++)
        {
        dtdiffvisc=0.25/(p->maxviscoef/((p->dx[dim])*(p->dx[dim])));
        if(dtdiffvisc>1.0e-8 && (p->dt)>dtdiffvisc )
                                      p->dt=dtdiffvisc;
        }*/

    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

 (*p)->dtdiffvisc=0.0000000001;


     for(int dir=0;dir<NDIM;dir++)
     {


     zeropadmaxdtvisc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir, *d_wtemp,ndimp);
      hipDeviceSynchronize();
	hipMemcpy(*wd, *d_wd, NDERV*dimp*sizeof(real), hipMemcpyDeviceToHost);
	//hipMemcpy(*d_wtemp, ((*wd)+(hdnur*dimp)), dimp*sizeof(real), hipMemcpyHostToDevice);

  for(i=0;i<numBlocks;i++)
               h_cmax[i]=0;
  hipMemcpy(d_bmax, h_cmax, numBlocks*sizeof(double), hipMemcpyHostToDevice);

  newreduction0computemaxdtvisc_parallel<<<numBlocks,NTPB,smemSize>>>(d_bmax,*d_wtemp,ndimp);
  hipDeviceSynchronize();
  hipMemcpy(h_cmax, d_bmax, numBlocks*sizeof(double), hipMemcpyDeviceToHost);

  int oldnumBlocks,newnumBlocks;
  newnumBlocks=numBlocks;

  /*while(newnumBlocks>1)
  {
        for(i=0;i<numBlocks;i++)
                h_cmax[i]=0;
        hipMemcpy(d_bmax, h_cmax, numBlocks*sizeof(double), hipMemcpyHostToDevice);

       oldnumBlocks=newnumBlocks;
  	newnumBlocks = (newnumBlocks+NTPB-1) / NTPB;

  	newreduction0computemaxdtvisc_parallel<<<newnumBlocks,NTPB,smemSize>>>(d_bmax,*d_wtemp,oldnumBlocks);
       hipDeviceSynchronize();
       hipMemcpy(h_cmax, d_bmax, newnumBlocks*sizeof(double), hipMemcpyDeviceToHost);*/

  /*for (i=0; i<numBlocks; i++)
    {
      fprintf(stdout,"cmax# %d %f\n",i, h_cmax[i]);
    }
       fprintf(stdout,"\n");*/


 // }
  for( i=0;i<numBlocks;i++)          		
                if(h_cmax[i]>((*p)->dtdiffvisc)) ((*p)->dtdiffvisc)=h_cmax[i];
  ((*p)->dtdiffvisc)=0.25/((*p)->dtdiffvisc);
;//if(h_cmax[0]>((*p)->dtdiffvisc))
        ;//  (*p)->dtdiffvisc=h_cmax[0];






	/*int s=1;
	while(((s*=2)<=((ndimp/2)-1)) ) 
	{
	   myreduction0computemaxdtvisc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir, *d_wtemp,ndimp,s);
	   hipDeviceSynchronize();
	}*/





         //getdtvisc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, dir);
         hipDeviceSynchronize();
         //hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);
         
         ;//if( ((*p)->dtdiffvisc)>1.0e-8 && ((*p)->dt)>((*p)->dtdiffvisc ))
         ;//                             (*p)->dt=(*p)->dtdiffvisc;
         hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
         
     }
     

   free(h_cmax);
  hipFree(d_bmax);
  hipFree(d_cmax);


}








