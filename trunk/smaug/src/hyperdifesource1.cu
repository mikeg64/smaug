#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"
#include "../include/iobparams.h"
/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/smaugcukernels.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_hde1.cuh"



__global__ void hyperdifesource4_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim, real dt)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1,ii0;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  //real dt=p->dt;
  real rdx;
  real dy=p->dx[1];
  real dx=p->dx[0];

   int ip,jp;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;


real del;


     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

#ifdef USE_SAC_3D
  rdx=(((wd[encode3_hde1(p,i,j,k,delx1)])*(dim==0))+(wd[encode3_hde1(p,i,j,k,delx2)])*(dim==1)+(wd[encode3_hde1(p,i,j,k,delx3)])*(dim==2));
#else
  rdx=(((wd[encode3_hde1(p,i,j,k,delx1)])*(dim==0))+  (wd[encode3_hde1(p,i,j,k,delx2)])*(dim==1)  );
#endif
     #ifdef USE_SAC_3D
       if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif
  {


//dwn1[fencode3_hde1(p,ii,field)]=( wtemp[fencode3_hde1(p,ii,hdnur)] *wtemp[fencode3_hde1(p,ii,tmp3)] - wtemp[fencode3_hde1(p,ii,hdnul)] *wtemp[fencode3_hde1(p,ii,tmp2)])/rdx;

   // wmod[fencode3_hde1(p,ii,field)+(ordero*NVAR*dimp)]=wmod[fencode3_hde1(p,ii,field)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hde1(p,ii,field)]; 
   //del=wmod[fencode3_hde1(p,ii,field)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hde1(p,ii,field)]; 
  // if(del<0.011 && del>0.009)
   //          wmod[fencode3_hde1(p,ii,field)+(ordero*NVAR*dimp)]=del;

   wmod[fencode3_hde1(p,ii,field)+(ordero*NVAR*dimp)]=wmod[fencode3_hde1(p,ii,field)+(ordero*NVAR*dimp)]+dt*( (wd[fencode3_hde1(p,ii,hdnur)]+wd[fencode3_hde1(p,ii,nushk1+dim)]) *wtemp[fencode3_hde1(p,ii,tmp3)] - (wd[fencode3_hde1(p,ii,hdnul)]+wd[fencode3_hde1(p,ii,nushk1+dim)]) *wtemp[fencode3_hde1(p,ii,tmp2)])/rdx;

  }

//__syncthreads();


/*if(iindex==0)
{
  p->hdmean=0.0;
  p->hdmax=0;

    for(ii[0]=0;ii[0]<((p->n[0]));ii[0]++)
      for(ii[1]=0;ii[1]<((p->n[1]));ii[1]++)
     #ifdef USE_SAC_3D
        for(ii[2]=0;ii[2]<((p->n[2]));ii[2]++)
     #endif
	{ 

             if((wtemp[encode3_hde1(p,ii[0],ii[1],0,tmp2)])>(p->hdmax))
                    p->hdmax=(wtemp[encode3_hde1(p,ii[0],ii[1],0,tmp2)]);
              p->hdmean=(p->hdmean)+wtemp[encode3_hde1(p,ii[0],ii[1],0,tmp2)];
	}
       p->hdmean=(p->hdmean)/(((p->n[0]))*((p->n[1])));

}
 //__syncthreads();*/


 
}


__global__ void hyperdifesource3_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1,ii0;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real rdx;
  real dy=p->dx[1];
  real dx=p->dx[0];

   int ip,jp;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;


#ifdef USE_SAC_3D
  rdx=(((wd[encode3_hde1(p,i,j,k,delx1)])*(dim==0))+(wd[encode3_hde1(p,i,j,k,delx2)])*(dim==1)+(wd[encode3_hde1(p,i,j,k,delx3)])*(dim==2));
#else
  rdx=(((wd[encode3_hde1(p,i,j,k,delx1)])*(dim==0))+  (wd[encode3_hde1(p,i,j,k,delx2)])*(dim==1)  );
#endif





     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif  
  {


 
dwn1[fencode3_hde1(p,ii,field)]=( (wd[fencode3_hde1(p,ii,hdnur)]+wd[fencode3_hde1(p,ii,nushk1+dim)]) *wtemp[fencode3_hde1(p,ii,tmp3)] - (wd[fencode3_hde1(p,ii,hdnul)]+wd[fencode3_hde1(p,ii,nushk1+dim)]) *wtemp[fencode3_hde1(p,ii,tmp2)])/rdx;
   


  }

//__syncthreads();



   
/*   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif  
                         //if(i<((p->n[0])) && j<((p->n[1])))
                         {
                              //                                                                                  - sign here same as vac maybe a +
                              wmod[fencode3_hde1(p,ii,field)+(ordero*NVAR*dimp)]=wmod[fencode3_hde1(p,ii,field)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hde1(p,ii,field)]; 

                         }
              //  }	
}
  //__syncthreads();*/



 
}

__global__ void hyperdifesource2_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1,ii0;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real rdx;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

   int ip,jp;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;


     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(i>0 && j >0 && k>0 && i<((p->n[0])-1) && j<((p->n[1])-1)   && k<((p->n[2])-1))
     #else
       if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
     #endif  
  //if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {
	wtemp[fencode3_hde1(p,ii,tmp2)]= grad1l3n_hde1(wtemp,wd,p,ii,tmp1,dim) ;
	wtemp[fencode3_hde1(p,ii,tmp3)]= grad1r3n_hde1(wtemp,wd,p,ii,tmp1,dim) ;
	//wtemp[fencode3_hde1(p,ii,tmp2)]= -0.0007 ;
	//wtemp[fencode3_hde1(p,ii,tmp3)]= -0.00005 ;
	//wtemp[fencode3_hde1(p,ii,tmp2)]= (  ( wtemp[encode3_hde1(p,i,j,k,rho)]-wtemp[encode3_hde1(p,i-(dim==0),j-(dim==1),k,rho)]) /((p->dx[0]))    ) ;
	//wtemp[fencode3_hde1(p,ii,tmp3)]= (  ( wtemp[encode3_hde1(p,i+(dim==0),j+(dim==1),k,rho)]-wtemp[encode3_hde1(p,i,j,k,rho)]) /((p->dx[0]))    ) ;

  }

//__syncthreads();




 
}



__global__ void hyperdifesource1a_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1,ii0;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real rdx;
  real dy=p->dx[1];
  real dx=p->dx[0];
   int ip,jp;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;


   

     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif 
//if(i<((p->n[0])) && j<((p->n[1])))
  {

#ifdef USE_SAC
     wtemp[fencode3_hde1(p,ii,tmp1)]=/*100**/((wmod[shift+fencode3_hde1(p,ii,energy)]-0.5*(


(wmod[shift+fencode3_hde1(p,ii,b1)]*wmod[shift+fencode3_hde1(p,ii,b1)]+wmod[shift+fencode3_hde1(p,ii,b2)]*wmod[shift+fencode3_hde1(p,ii,b2)])

+((wmod[shift+fencode3_hde1(p,ii,mom1)]*wmod[shift+fencode3_hde1(p,ii,mom1)]+wmod[shift+fencode3_hde1(p,ii,mom2)]*wmod[shift+fencode3_hde1(p,ii,mom2)])/(wmod[shift+fencode3_hde1(p,ii,rho)]+wmod[shift+fencode3_hde1(p,ii,rhob)])))));
#endif
#ifdef USE_SAC_3D
     wtemp[fencode3_hde1(p,ii,tmp1)]=wmod[shift+fencode3_hde1(p,ii,energy)]-0.5*((wmod[shift+fencode3_hde1(p,ii,b1)]*wmod[shift+fencode3_hde1(p,ii,b1)]+wmod[shift+fencode3_hde1(p,ii,b2)]*wmod[shift+fencode3_hde1(p,ii,b2)]+wmod[shift+fencode3_hde1(p,ii,b3)]*wmod[shift+fencode3_hde1(p,ii,b3)])

+((wmod[shift+fencode3_hde1(p,ii,mom1)]*wmod[shift+fencode3_hde1(p,ii,mom1)]+wmod[shift+fencode3_hde1(p,ii,mom2)]*wmod[shift+fencode3_hde1(p,ii,mom2)]+wmod[shift+fencode3_hde1(p,ii,mom3)]*wmod[shift+fencode3_hde1(p,ii,mom3)])/(wmod[shift+fencode3_hde1(p,ii,rho)]+wmod[shift+fencode3_hde1(p,ii,rhob)]))
);

#endif
 


  }

//__syncthreads();




 
}





__global__ void hyperdifesource1_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real *wtemp, int field, int dim)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int ii1,ii0;
  real fip,fim1;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real rdx;
  real dy=p->dx[1];
  real dx=p->dx[0];
   int ip,jp;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;


     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif  
//init rhol and rhor
  //if(i<((p->n[0])) && j<((p->n[1])))
  {
    for(int f=tmp1; f<=tmp8; f++)	
        wtemp[fencode3_hde1(p,ii,f)]=0.0;
    dwn1[fencode3_hde1(p,ii,field)]=0.0;
   }

 //__syncthreads();

 




 
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hde1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifesource1(struct params **p,  struct params **d_p,   real **d_wmod, real **d_dwn1, real **d_wd, int order,int ordero, real **d_wtemp, int field, int dim,real dt)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   //hipSetDevice(selectedDevice);
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     hyperdifesource1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim);
      hipDeviceSynchronize();

     hyperdifesource1a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim);
      hipDeviceSynchronize();



     hyperdifesource2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim);
      hipDeviceSynchronize();

     //hyperdifesource3_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim);
    //  hipDeviceSynchronize();

     hyperdifesource4_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,dt);
      hipDeviceSynchronize();

    /*hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);
    printf("dim hdmean hdmax %d %8.8g %8.8g \n",dim, (*p)->hdmean, (*p)->hdmax);*/
}







