#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/smaugcukernels.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_advect.cuh"
#include "../include/dervfields_advect.cuh"



__global__ void advect_parallel(struct params *p,  real *w, real *wnew, real *wmod, 
    real *dwn1, real *wd, int order)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;
  int order1;
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];
  int ix[NDIM];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;


  

   j=iindex/ni;
   //i=iindex-j*(iindex/ni);
   i=iindex-(j*ni);
   ix[0]=i;
   ix[1]=j;
   if(order==0 || order==1)
     dt=(p->dt)/2.0;
   if(order==2 )
     dt=(p->dt);
   if(order==3 )
     dt=(p->dt)/6.0;

   if(order==3)
     order1=0;
   else
     order1=order+1;

  //order1=1;
//dt=(p->dt);
  if(p->rkon != 1)
  {
    dt=(p->dt);
    order1=0;
  }


  //advance the solution for one of the advect steps
  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{ 
   
		for(int f=rho; f<=b3; f++)           
 			//wmod[fencode_advect(p,i,j,f)]=((w[fencode_advect(p,i+1,j,f)]+w[fencode_advect(p,i-1,j,f)]+w[fencode_advect(p,i,j+1,f)]+w[fencode_advect(p,i,j-1,f)])/4.0)+dt*dwn1[(NVAR*ni*nj*(order-1))+fencode_advect(p,i,j,f)];
                   wmod[NVAR*(ni*nj*(order1))+fencode_advect(p,i,j,f)]=wmod[NVAR*(ni*nj*(order))+fencode_advect(p,i,j,f)]+dt*dwn1[fencode_advect(p,i,j,f)];
	}

 __syncthreads();



  if((p->rkon == 1) && i<((p->n[0])) && j<((p->n[1])))
	{ 
   
		for(int f=rho; f<=b3; f++) 
                {   
                       if(order==1)
			   wmod[fencode_advect(p,i,j,f)]=wmod[NVAR*(ni*nj*(order1))+fencode_advect(p,i,j,f)];	
                        /*else if(order==1) 
			   wmod[NVAR*(ni*nj*(2))+fencode_advect(p,i,j,f)]=w[fencode_advect(p,i,j,f)];*/	
                        if(order==2)  
			   wmod[NVAR*(ni*nj*(1))+fencode_advect(p,i,j,f)]=(wmod[NVAR*(ni*nj*(1))+fencode_advect(p,i,j,f)]	+2*wmod[NVAR*(ni*nj*(2))+fencode_advect(p,i,j,f)]    + wmod[NVAR*(ni*nj*(3))+fencode_advect(p,i,j,f)]- 4*w[fencode_advect(p,i,j,f)])/3;

                        else if(order==3)      
 			//wmod[fencode_advect(p,i,j,f)]=((w[fencode_advect(p,i+1,j,f)]+w[fencode_advect(p,i-1,j,f)]+w[fencode_advect(p,i,j+1,f)]+w[fencode_advect(p,i,j-1,f)])/4.0)+dt*dwn1[(NVAR*ni*nj*(order-1))+fencode_advect(p,i,j,f)];
                   wmod[fencode_advect(p,i,j,f)]=wmod[NVAR*(ni*nj)+fencode_advect(p,i,j,f)]+wmod[fencode_advect(p,i,j,f)];
                }
	}

 __syncthreads();



  if((order==3) && (p->rkon == 1) && i<((p->n[0])) && j<((p->n[1])))   
		for(int f=rho; f<=b3; f++) 
                   wmod[fencode_advect(p,i,j,f)]=wmod[NVAR*(ni*nj)+fencode_advect(p,i,j,f)]+wmod[fencode_advect(p,i,j,f)];



 __syncthreads();

if(i<((p->n[0])) && j<((p->n[1])))
	{		
               //for(int f=rho; f<=b3; f++)
               //{               
               //   wmod[fencode_advect(p,i,j,f)]=w[fencode_advect(p,i,j,f)];
               //   wnew[fencode_advect(p,i,j,f)]=0.0;
               //}
               for(int f=current1; f<=f3; f++)
                  wd[fencode_advect(p,i,j,f)]=0; 
        }
               __syncthreads();


 /* if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{		               
               computej_advect(wmod+((p->n[0])*(p->n[1])*order1),wd,p,i,j);
               computepk_advect(wmod+((p->n[0])*(p->n[1])*order1),wd,p,i,j);
               computept_advect(wmod+((p->n[0])*(p->n[1])*order1),wd,p,i,j);

               computebdotv_advect(wmod+((p->n[0])*(p->n[1])*order1),wd,p,i,j);
               computedivb_advect(wmod+((p->n[0])*(p->n[1])*order1),wd,p,i,j);
         }
              __syncthreads();
  if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               computec_advect(wmod+((p->n[0])*(p->n[1])*order1),wd,p,i,j);
        }
              __syncthreads();*/


  
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_advect(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cuadvect(struct params **p, real **w, real **wnew, struct params **d_p, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd, int order)
{


//printf("calling propagate solution\n");

    //dim3 dimBlock(blocksize, blocksize);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimproduct_advect(*p)+numThreadsPerBlock-1) / numThreadsPerBlock;

//__global__ void prop_parallel(struct params *p, real *b, real *w, real *wnew, real *wmod, 
  //  real *dwn1, real *dwn2, real *dwn3, real *dwn4, real *wd)
     //init_parallel(struct params *p, real *b, real *u, real *v, real *h)
     advect_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w,*d_wnew, *d_wmod, *d_dwn1,  *d_wd, order);
     //prop_parallel<<<dimGrid,dimBlock>>>(*d_p,*d_b,*d_u,*d_v,*d_h);
	    //printf("called prop\n"); 
     hipDeviceSynchronize();
     //boundary_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called boundary\n");  
     //hipDeviceSynchronize();
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_b,*d_w,*d_wnew);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
 

  //  hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);

     //following used for testing to check current soundspeeds etc
     //hipMemcpy(*w, *d_wd, 7*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}






