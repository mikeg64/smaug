#include "hip/hip_runtime.h"



//bach3d
__device__ __host__
void init_user_MODID (real *w, struct params *p,int *ii) {
                    
	real p1,p2,rho0,rho2,v1,v2,v3,T1,T2, xc,yc,zc,r0;
	real Ly, e0,c0;
        real x,y,z;


	Ly=9.46d15;

	e0=1.e48;

	c0=8.95e13;

	p1=1.e0;
	rho0=2.e-22;

	v1=0.e0;
	v2=0.e0;
	v3=0.e0;

	xc=0.0e0;
	yc=0.0e0;
	zc=0.0e0;

	  int i,j,k;
	  i=ii[0];
	  j=ii[1];
	  k=ii[2];

          x=i*(p->dx[0]);
          y=i*(p->dx[1]);
          z=i*(p->dx[2]);
	#ifdef USE_SAC_3D



                    w[fencode3_i(p,ii,rhob)]=0.0;
                    w[fencode3_i(p,ii,energy)]=0.0;
		    w[fencode3_i(p,ii,rhob)]=rho0+c0/((x-xc)*(x-xc)+(y-yc)*(y-yc)+(z-zc)*(z-zc));

                    rgamm1=1.0/((p->gamma)-1);
                    w[fencode3_i(p,ii,energyb)]=rgamm1*pow(rho0,(p->gamma));
		    
		    w[fencode3_i(p,ii,b1)]=0;
		    w[fencode3_i(p,ii,b2)]=0;
		    w[fencode3_i(p,ii,b3)]=0;


		    w[fencode3_i(p,ii,mom3)]=v3;
		    w[fencode3_i(p,ii,mom2)]=v2;
		    w[fencode3_i(p,ii,mom1)]=v1;

                   e1=(0.5*rgamm1*(1-((p->gamma)-1))*(w[fencode3_i(p,ii,b1)]*w[fencode3_i(p,ii,b1)]+w[fencode3_i(p,ii,b2)]*w[fencode3_i(p,ii,b2)]+w[fencode3_i(p,ii,b3)]*w[fencode3_i(p,ii,b3)]));
                    w[fencode3_i(p,ii,energyb)]=w[fencode3_i(p,ii,energyb)]-e1;


                    w[fencode3_i(p,ii,energy)]=w[fencode3_i(p,ii,energyb)];
                    w[fencode3_i(p,ii,energyb)]=0.0;


                     if(i==16 && jj==16  && k==11)
                        w[fencode3_i(p,ii,energy)]=e0/pow(p->dx[0],3.0);
  
			// w(40,28,e_)=e0/(x(1,3,2)-x(1,2,2))**3.d0
			//  w(80,92,e_)=e0/(x(1,3,2)-x(1,2,2))**3.d0  

		    w[fencode3_i(p,ii,bg1)]=w[fencode3_i(p,ii,b1)];
		    w[fencode3_i(p,ii,bg2)]=w[fencode3_i(p,ii,b2)];
		    w[fencode3_i(p,ii,bg3)]=w[fencode3_i(p,ii,b3)];

		    w[fencode3_i(p,ii,b1)]=0;
		    w[fencode3_i(p,ii,b2)]=0;
		    w[fencode3_i(p,ii,b3)]=0;




       #endif





}



