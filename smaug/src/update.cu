#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/smaugcukernels.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_u.cuh"


__device__ __host__
int updatestate (struct params *p, struct state *s, real *w ,int *ii, int field) {

  int status=0;
                      // atomicExch(&(p->cmax),(wd[fencode3_pre(p,ii,soundspeed)]));
                    switch(field)
                    {
                      case rho:
                    	s->rho=s->rho+(w[fencode3_u(p,ii,field)]);
		      break;
                      case mom1:
                    	s->m1=s->m1+(w[fencode3_u(p,ii,field)]);
		      break;
                      case mom2:
                    	s->m2=s->m2+(w[fencode3_u(p,ii,field)]);
		      break;
                      /*case mom3:
                    	s->m3=s->m3+(w[fencode3_u(p,ii,field)]);
		      break;*/
                      case energy:
                    	s->e=s->e+(w[fencode3_u(p,ii,field)]);
		      break;
                      case b1:
                    	s->b1=s->b1+(w[fencode3_u(p,ii,field)]);
		      break;
                      case b2:
                    	s->b2=s->b2+(w[fencode3_u(p,ii,field)]);
		      break;
                      /*case b3:
                    	s->b3=s->b3+(w[fencode3_u(p,ii,field)]);
		      break;*/
                    };
  return status;
}



__global__ void update_parallel(struct params *p, struct state *s, real *w, real *wmod)
{

   int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,f;
  int index,k;
  __shared__ int ntot;

  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  //real g=p->g;
  real *u,  *v,  *h;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;
  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     



//int shift=order*NVAR*dimp;

  h=w+dimp*rho;
  u=w+dimp*mom1;
  v=w+dimp*mom2;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
           for( f=rho; f<=b3; f++)
     #else
           for( f=rho; f<=b2; f++)
     #endif
             {  
         #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif           
	{
            
                  w[fencode3_u(p,iia,f)]=wmod[fencode3_u(p,iia,f)];
                          //   if(p->ipe==0    && f==rho)
                          //      printf("wmod,w %d %d %lg %lg\n",iia[0],iia[1],wmod[fencode3_u(p,iia,f)],w[fencode3_u(p,iia,f)]);

	}


}

__syncthreads(); 







  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_u(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}
int cuupdate(struct params **p, real **w, real **wmod,real **wtemp2, struct state **state,struct params **d_p, real **d_w, real **d_wmod, real ** d_wtemp2, struct state **d_state, int step)
//int cuupdate(struct params **p, real **w, real **wmod, real **wd, real **temp2, struct state **state,
//             struct params **d_p, real **d_w, real **d_wmod, real **d_wtemp2, struct state **d_state, int step)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
    dim3 dimBlock(dimblock, 1);
 
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;
  // hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyHostToDevice);
hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

//no longer necessary as w field no longer used
//just do a memcpy at end of this call
 //    update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_state,*d_w,*d_wmod);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();
//following comments removed from if def pragmas  if
//using MPI and copying all cell data to host (how slow!?)
//#ifdef USE_MPI

//#else
    if((step%((*p)->cfgsavefrequency))==0)
//#endif
    {

//following commentes removed from section if
//using MPI and copying all cell data to host (how slow!?)
/*#ifdef USE_MPI
    hipMemcpy(*wmod, *d_w, NVAR*dimp*sizeof(real), hipMemcpyDeviceToHost);
    #ifdef USE_SAC_3D  
           hipMemcpy(*wtemp2, *d_wtemp2,NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2)* (((*p)->n[2])+2)*sizeof(real), hipMemcpyDeviceToHost);
    #else
       hipMemcpy(*wtemp2, *d_wtemp2,NTEMP2*(((*p)->n[0])+2)* (((*p)->n[1])+2)*sizeof(real), hipMemcpyDeviceToHost);
    #endif

#endif */ 


 #ifdef USE_GPUD

	#ifdef USE_SAC_3D
	   int ndimp=((*p)->n[0])*((*p)->n[1])*((*p)->n[2]);
        #else
	   int ndimp= ((*p)->n[0])*((*p)->n[1]);
	#endif      

     real      *wt=(real *)calloc(ndimp*NVAR,sizeof(real));
 

     int shift,oshift;
     int ok1,oj1,oi1;
     int oni,onj,onk;
     int i1,j1,k1;
     int ni,nj,nk;
     real *wa=*w;

 
     oni=((*p)->n[0])*((*p)->pnpe[0]);
     onj=((*p)->n[1])*((*p)->pnpe[1]);
     ni=((*p)->n[0]);
     nj=((*p)->n[1]);

     #ifdef USE_SAC_3D
     	onk=((*p)->n[2])*((*p)->pnpe[2]);
        nk=((*p)->n[2]);
     #endif

    hipMemcpy(wt, *d_w, NVAR*ndimp*sizeof(real), hipMemcpyDeviceToHost);



     for(int ivar=0; ivar<NVAR; ivar++)
     {

		#ifdef USE_SAC_3D
		   for(k1=0; k1<nk; k1++)
		#endif
        for(j1=0; j1<nj; j1++)
        for(i1=0; i1<ni; i1++)
        {
                oi1=i1+((*p)->pipe[0]*ni);
                oj1=j1+((*p)->pipe[1]*nj);  
		#ifdef USE_SAC_3D
                         shift=(k1*ni*nj+j1*ni+i1);
                         ok1=k1+((*p)->pipe[2]*nk);

                         oshift=(ok1*oni*onj+oj1*oni+oi1);
		#else
			 shift=(j1*ni+i1);
                         oshift=(oj1*oni+oi1);
                #endif
                 //if(i1==0 && j1==0)
                 //if(ivar==0 && ((*p)->ipe)==0 && step==5)
                 // printf("called update %d %d %d %lg %lg\n",ivar,shift,oshift+oni*onj*ivar,wa[oshift+oni*onj*ivar],wt[shift+ivar*ndimp]);//, wa[oshift+oni*onj*ivar]);//,wt[shift]);
                  
                   
              wa[oshift+oni*onj*ivar]=wt[shift+ivar*ndimp];
                                              
        }
     }

       printf("here1\n");   
          free(wt);
         // free(wdt);
#else

    //hipMemcpy(*w, *d_w, NVAR*dimp*sizeof(real), hipMemcpyDeviceToHost);
    hipMemcpy(*wmod, *d_wmod, NVAR*dimp*sizeof(real), hipMemcpyDeviceToHost);

#endif




   // hipMemcpy(*w, *d_w, NVAR*dimp*sizeof(real), hipMemcpyDeviceToHost);

    //hipMemcpy(*wnew, *d_wd, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);

   hipMemcpy(*state, *d_state, sizeof(struct state), hipMemcpyDeviceToHost);
    }

//hipMemcpy(*wnew, *d_wnew, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_u, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors("copy data from device");


 


}


int cuupdatehostwd(struct params **p, real **wd, real **wmod,real **wtemp2, struct state **state,struct params **d_p, real **d_wd, real **d_wmod, real ** d_wtemp2, struct state **d_state, int step)
//int cuupdate(struct params **p, real **w, real **wmod, real **wd, real **temp2, struct state **state,
//             struct params **d_p, real **d_w, real **d_wmod, real **d_wtemp2, struct state **d_state, int step)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
    dim3 dimBlock(dimblock, 1);
 
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;
  // hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyHostToDevice);
hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_state,*d_w,*d_wmod);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();



 #ifdef USE_GPUD

	#ifdef USE_SAC_3D
	   int ndimp=((*p)->n[0])*((*p)->n[1])*((*p)->n[2]);
        #else
	   int ndimp= ((*p)->n[0])*((*p)->n[1]);
	#endif      

     real      *wt=(real *)calloc(ndimp*NDERV,sizeof(real));
 

     int shift,oshift;
     int ok1,oj1,oi1;
     int oni,onj,onk;
     int i1,j1,k1;
     int ni,nj,nk;
     real *wa=*wd;

 
     oni=((*p)->n[0])*((*p)->pnpe[0]);
     onj=((*p)->n[1])*((*p)->pnpe[1]);
     ni=((*p)->n[0]);
     nj=((*p)->n[1]);

     #ifdef USE_SAC_3D
     	onk=((*p)->n[2])*((*p)->pnpe[2]);
        nk=((*p)->n[2]);
     #endif

    hipMemcpy(wt, *d_wd, NDERV*ndimp*sizeof(real), hipMemcpyDeviceToHost);



     for(int ivar=0; ivar<NDERV; ivar++)
     {

		#ifdef USE_SAC_3D
		   for(k1=0; k1<nk; k1++)
		#endif
        for(j1=0; j1<nj; j1++)
        for(i1=0; i1<ni; i1++)
        {
                oi1=i1+((*p)->pipe[0]*ni);
                oj1=j1+((*p)->pipe[1]*nj);  
		#ifdef USE_SAC_3D
                         shift=(k1*ni*nj+j1*ni+i1);
                         ok1=k1+((*p)->pipe[2]*nk);

                         oshift=(ok1*oni*onj+oj1*oni+oi1);
		#else
			 shift=(j1*ni+i1);
                         oshift=(oj1*oni+oi1);
                #endif
                 //if(i1==0 && j1==0)
                 //if(ivar==0 && ((*p)->ipe)==0 && step==5)
                 // printf("called update %d %d %d %lg %lg\n",ivar,shift,oshift+oni*onj*ivar,wa[oshift+oni*onj*ivar],wt[shift+ivar*ndimp]);//, wa[oshift+oni*onj*ivar]);//,wt[shift]);
                  
                   
              wa[oshift+oni*onj*ivar]=wt[shift+ivar*ndimp];
                                              
        }
     }

       printf("here1\n");   
          free(wt);
         // free(wdt);
#else

 hipMemcpy(*wd, *d_wd, NDERV*dimp*sizeof(real), hipMemcpyDeviceToHost);

/*real *wad=*wd;
int iii[3];
iii[2]=0;
printf("update host wd %d\n",(*p)->ipe);
if(((*p)->ipe)==3) 
        for(iii[0]=0; iii[0]<((*p)->n[0]); iii[0]++)
          for(iii[1]=0; iii[1]<((*p)->n[1]); iii[1]++)
             {
               //if(iii[0]==0)
               printf("delx 0 %d %d %16.20f  %16.20f\n",iii[0],iii[1],wad[(fencode3_u(*p,iii,pos1))],wad[(fencode3_u(*p,iii,pos2))]);
             //printf("delx 0 %d %d %d %d\n",iii[0],iii[1],(fencode3_u(*p,iii,pos1)),(fencode3_u(*p,iii,pos2)));
              }*/




   






#endif


  //checkErrors("copy data from device");


 


}



int cuupdatedevicewd(struct params **p, real **wd, real **wmod,real **wtemp2, struct state **state,struct params **d_p, real **d_wd, real **d_wmod, real ** d_wtemp2, struct state **d_state, int step)
//int cuupdate(struct params **p, real **w, real **wmod, real **wd, real **temp2, struct state **state,
//             struct params **d_p, real **d_w, real **d_wmod, real **d_wtemp2, struct state **d_state, int step)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
    dim3 dimBlock(dimblock, 1);
 
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;
  // hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyHostToDevice);
hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
     //update_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_state,*d_w,*d_wmod);
	    //printf("called update\n"); 
   // hipDeviceSynchronize();



 #ifdef USE_GPUD

	#ifdef USE_SAC_3D
	   int ndimp=((*p)->n[0])*((*p)->n[1])*((*p)->n[2]);
        #else
	   int ndimp= ((*p)->n[0])*((*p)->n[1]);
	#endif      

     real      *wt=(real *)calloc(ndimp*NDERV,sizeof(real));
 

     int shift,oshift;
     int ok1,oj1,oi1;
     int oni,onj,onk;
     int i1,j1,k1;
     int ni,nj,nk;
     real *wa=*wd;

 
     oni=((*p)->n[0])*((*p)->pnpe[0]);
     onj=((*p)->n[1])*((*p)->pnpe[1]);
     ni=((*p)->n[0]);
     nj=((*p)->n[1]);

    hipMemcpy(*d_wd,wt, NDERV*ndimp*sizeof(real), hipMemcpyHostToDevice);

     #ifdef USE_SAC_3D
     	onk=((*p)->n[2])*((*p)->pnpe[2]);
        nk=((*p)->n[2]);
     #endif

    



     for(int ivar=0; ivar<NDERV; ivar++)
     {

		#ifdef USE_SAC_3D
		   for(k1=0; k1<nk; k1++)
		#endif
        for(j1=0; j1<nj; j1++)
        for(i1=0; i1<ni; i1++)
        {
                oi1=i1+((*p)->pipe[0]*ni);
                oj1=j1+((*p)->pipe[1]*nj);  
		#ifdef USE_SAC_3D
                         shift=(k1*ni*nj+j1*ni+i1);
                         ok1=k1+((*p)->pipe[2]*nk);

                         oshift=(ok1*oni*onj+oj1*oni+oi1);
		#else
			 shift=(j1*ni+i1);
                         oshift=(oj1*oni+oi1);
                #endif
                 //if(i1==0 && j1==0)
                 //if(ivar==0 && ((*p)->ipe)==0 && step==5)
                 // printf("called update %d %d %d %lg %lg\n",ivar,shift,oshift+oni*onj*ivar,wa[oshift+oni*onj*ivar],wt[shift+ivar*ndimp]);//, wa[oshift+oni*onj*ivar]);//,wt[shift]);
                  
                   
              wa[oshift+oni*onj*ivar]=wt[shift+ivar*ndimp];
                                              
        }
     }

       printf("here1\n");   
          free(wt);
         // free(wdt);
#else

    hipMemcpy(*d_wd, *wd, NDERV*dimp*sizeof(real), hipMemcpyHostToDevice);

#endif


  //checkErrors("copy data from device");


 


}




int cufinish(struct params **p, real **w, real **wnew, struct state **state, struct params **d_p,struct bparams **d_bp, real **d_w, real **d_wnew, real **d_wmod, real **d_dwn1, real **d_wd, struct state **d_state, real **d_wtemp, real **d_wtemp1, real **d_wtemp2)
{
  

 //hipMemcpy(*w, *d_w, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_u, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  checkErrors_u("copy data from device");


  hipFree(*d_p);
  hipFree(*d_bp);
//  hipFree(*d_state);

//  hipFree(*d_w);
  hipFree(*d_wnew);
 // hipFree(*d_u);

  hipFree(*d_wmod);
  hipFree(*d_dwn1);
  hipFree(*d_wd);
  hipFree(*d_wtemp);
  hipFree(*d_wtemp1);
  hipFree(*d_wtemp2);
  




}

  #ifdef USE_MPI

int cufinishmgpu(struct params **p,real **w, real **wmod, real **temp2, real **gmpivisc0, real **gmpivisc1, real **gmpivisc2,   real **gmpiw0, real **gmpiwmod0,   real **gmpiw1, real **gmpiwmod1,   real **gmpiw2, real **gmpiwmod2, struct params **d_p,   real **d_w, real **d_wmod,real **d_wtemp2,    real **d_gmpivisc0,    real **d_gmpivisc1,    real **d_gmpivisc2,   real **d_gmpiw0, real **d_gmpiwmod0,   real **d_gmpiw1, real **d_gmpiwmod1,   real **d_gmpiw2, real **d_gmpiwmod2)
{
  

 //hipMemcpy(*w, *d_w, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*wnew, *d_wnew, 8*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
//hipMemcpy(*b, *d_u, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

  //checkErrors_u("copy data from device");


  hipFree(*d_gmpiw0);
  hipFree(*d_gmpiwmod0);

  hipFree(*d_gmpiw1);
  hipFree(*d_gmpiwmod1);
#ifdef USE_SAC_3D
  hipFree(*d_gmpiw2);
  hipFree(*d_gmpiwmod2);
  hipFree(*d_gmpivisc2);
#endif
  hipFree(*d_gmpivisc0);
  hipFree(*d_gmpivisc1);

  //free(*gmpiw0);
  //free(*gmpiwmod0);

 // free(*gmpiw1);
 // free(*gmpiwmod1);
#ifdef USE_SAC_3D
  free(*gmpiw2);
  free(*gmpiwmod2);
free(*gmpivisc2);
#endif

  free(*gmpivisc0);
free(*gmpivisc1);
  //free(*temp2);
}
#endif
