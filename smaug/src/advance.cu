#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/iotypes.h"
#include "../include/iobparams.h"
/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/smaugcukernels.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_adv.cuh"
#include "../include/dervfields_adv.cuh"


__global__ void advance_parallel(struct params *p, real *wmod, real *w,  int order)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;

  int index,i,j,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[0];
  real dx=p->dx[1];

    int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     

int shift=order*NVAR*dimp;




     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif
	{		               
 
               float big=9999.0;
               for(int f=rho; f<NVAR; f++)
               {
                  
                   
                  if((p->rkon)==1)
                  {
                    switch(order)
                     {
                        case 0:
                       wmod[fencode3_adv(p,iia,f)+(2*dimp*NVAR)]=wmod[fencode3_adv(p,iia,f)];

                         break;
                        case 1:
                       wmod[fencode3_adv(p,iia,f)+(3*dimp*NVAR)]=wmod[fencode3_adv(p,iia,f)];
 
                         break;
                        case 2:
                       wmod[fencode3_adv(p,iia,f)+(dimp*NVAR)]=(wmod[fencode3_adv(p,iia,f)+(dimp*NVAR)]+2.0*wmod[fencode3_adv(p,iia,f)+(2*dimp*NVAR)]+wmod[fencode3_adv(p,iia,f)+(3*dimp*NVAR)]-4.0*wmod[fencode3_adv(p,iia,f)])/3;


                         break;
                        case 3:

                        wmod[fencode3_adv(p,iia,f)]=wmod[fencode3_adv(p,iia,f)]+wmod[fencode3_adv(p,iia,f)+(dimp*NVAR)];

                         break;

                     }
                   }
                  else
                  {
                  //if((dwn1[fencode3_adv(p,iia,f)]<(big/100)) && ( dwn1[fencode3_adv(p,iia,f)]>(-big/100)) )
                  //  if( j!=2)
                       //wmod[fencode3_adv(p,iia,f)]=wmod[fencode3_adv(p,iia,f)+(order*(p->n[0])*(p->n[1])*NVAR)];
                      wmod[fencode3_adv(p,iia,f)]=wmod[fencode3_adv(p,iia,f)+(dimp*NVAR)];
                   //lax-friedrichs
                  //wmod[fencode3_adv(p,iia,f)]=((w[fencode3_adv(p,i+1,j,f)]+w[fencode3_adv(p,i-1,j,f)]+w[fencode3_adv(p,iia+1,f)]+w[fencode3_adv(p,iia-1,f)])/4.0)+(dt)*(dwn1[fencode3_adv(p,iia,f)]);
                   }
                  
                  /* if(isnan(wmod[fencode3_adv(p,iia,f)])) wmod[fencode3_adv(p,iia,f)]=w[fencode3_adv(p,iia,f)];
                   if(wmod[fencode3_adv(p,iia,f)]>big)
                           wmod[fencode3_adv(p,iia,f)]=w[fencode3_adv(p,iia,f)];
                   if(wmod[fencode3_adv(p,iia,f)]<-big)
                           wmod[fencode3_adv(p,iia,f)]=w[fencode3_adv(p,iia,f)];

                     if(f==rho)
                            if(wmod[fencode3_adv(p,iia,f)]<0)
                               wmod[fencode3_adv(p,iia,f)]=1.00;*/
               }



	}

 __syncthreads();





  
}
/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_adv(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}






int cuadvance(struct params **p, struct params **d_p,  real **d_wmod, real **d_w,  int order)
{

 dim3 dimBlock(dimblock, 1);
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;

     advance_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_wmod, *d_w, order);
     hipDeviceSynchronize();
}



