#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/smaugcukernels.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_hdv1.cuh"

__device__ __host__
void bc_hyperdif(real *wt, struct params *p,int *ii, int f,int dir) {

   int i=ii[0];
   int j=ii[1];
   int k=0;
 #ifdef USE_SAC_3D
	k=ii[2];
 #endif

int is=1;
 #ifdef USE_SAC
   if(  (dir == 0) && (i==(p->n[0])-1)   && j>=0   && j<(p->n[1])           )
   {
      //for(int is=0;is<3-2*(j<((p->n[1])-1));is++)   
         wt[encode3p2_hdv1(p,i+2,j+is,k,f)]=wt[encode3p2_hdv1(p,(p->n[0])-5,j+is,k,f)];
         
   }
   else if((dir == 1) && (j==(p->n[1])-1)    && i>0   && i<((p->n[0]))  )
    //for(int is=0;is<3-2*(j<((p->n[1])-1));is++)
       wt[encode3p2_hdv1(p,i+is,j+2,k,f)]=wt[encode3p2_hdv1(p,i+is,(p->n[1])-5,k,f)];
  else if((dir == 0) && (i==0)    && j>0   && j<((p->n[1]))   )
    //for(int is=0;is<3-2*(j<((p->n[1])-1));is++)
       wt[encode3p2_hdv1(p,0,j+is,k,f)]=wt[encode3p2_hdv1(p,6,j+is,k,f)];
   else if((dir == 1) && (j==0)    && i>0   && i<((p->n[0]))   )
    //for(int is=0;is<3-2*(j<((p->n[1])-1));is++)
       wt[encode3p2_hdv1(p,i+is,0,k,f)]=wt[encode3p2_hdv1(p,i+is,6,k,f)];

#endif
 #ifdef USE_SAC_3D
   if(  (dir == 0) && (i==(p->n[0])-1)   && j>0   && j<(p->n[1])      && k>0   && k<(p->n[2])     )
         wt[encode3p2_hdv1(p,i+2,j,k,f)]=wt[encode3p2_hdv1(p,(p->n[0])-5,j,k,f)];
   else if((dir == 1) && (j==(p->n[1])-1)    && i>0   && i<((p->n[0])) && k>0   && k<((p->n[2]))  )
       wt[encode3p2_hdv1(p,i,j+2,k,f)]=wt[encode3p2_hdv1(p,i,(p->n[1])-5,k,f)];
   else if((dir == 2) && (k==(p->n[2])-1)    && i>0   && i<((p->n[0])) && j>0   && j<((p->n[1]))  )
       wt[encode3p2_hdv1(p,i,j,k+2,f)]=wt[encode3p2_hdv1(p,i,j,(p->n[2])-5,f)];
  else if((dir == 0) && (i==0)    && j>0   && j<((p->n[1])) && k>0   && k<((p->n[2]))  )
       wt[encode3p2_hdv1(p,0,j,k,f)]=wt[encode3p2_hdv1(p,6,j,k,f)];
   else if((dir == 1) && (j==0)    && i>0   && i<((p->n[0]))  && k>0   && k<((p->n[2]))  )
       wt[encode3p2_hdv1(p,i,0,k,f)]=wt[encode3p2_hdv1(p,i,6,k,f)];
   else if((dir == 2) && (k==0)    && i>0   && i<((p->n[0])) && j>0   && j<((p->n[1]))   )
       wt[encode3p2_hdv1(p,i,j,0,f)]=wt[encode3p2_hdv1(p,i,j,6,f)];
#endif




 
}


/*__device__ __host__
void bc_periodic1_temp2(real *wt, struct params *p,int i, int j, int f) {

                if(i==1 )                
                    wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,6,j,f)];
                else if((i==((p->n[0]))) )                
                    wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i-4,j,f)];
                else if(j==1  )                
                  wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i,6,f)];
                else if((j==((p->n[1]))) )                
                  wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i,j-4,f)];
}*/

/*__device__ __host__
void bc_periodic2_temp2(real *wt, struct params *p,int i, int j, int f) {


               if(i<1 && j<1)
                {
                  if(i==j)
                    //wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,(p->n[0])-3+i,j,f)];
                    wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i,6,f)];
                  else                  
                    //wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i,(p->n[1])-3+j,f)];
                    wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,6,j,f)];                                    
                }
                else if(i<1 && j>((p->n[1])-1))
                {
                  if(i==(j-(p->n[1])-1))                  
                    //wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,(p->n[0])-3+i,4-(p->n[1])+j,f)];
                    wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,6,j,f)];                                     
                  else                  
                    wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i,j-6,f)];                                     
                }
                else if(i>((p->n[0])-1) && j<1)
                {
                  if((i-(p->n[0])+1)==j)                  
                    wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i-5,j,f)];                                    
                  else                  
                   wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i,4,f)];                                    
                }
                else if(i>((p->n[0])-1) && j>((p->n[1])-1))
                {
                  if(i==j)                  
                    wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i,j-5,f)];                                    
                  else                  
                    wt[fencode_hdv1(p,i,j,f)]=wt[fencode_hdv1(p,i-5,j,f)];                                    
                }                       
                 
                




}*/


__global__ void hyperdifvisc5_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim,int hand)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  real maxt=0,max3=0, max1=0;
  
   int ip,jp,ipg,jpg;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  


int shift=order*NVAR*dimp;

   
if(iindex==0)
{
  p->hdmean=0.0;
  p->hdmax=0;
 //  for(ipg=0;ipg<(p->npgp[0]);ipg++)
 //  for(jpg=0;jpg<(p->npgp[1]);jpg++)
  // {

  //   i=ip*(p->npgp[0])+ipg;
 //    j=jp*(p->npgp[1])+jpg;
   //if( i<((p->n[0])) && j<((p->n[1])))
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
    //p->cmax=0.0;
    for(ii[0]=1;ii[0]<((p->n[0])-1);ii[0]++)
      for(ii[1]=1;ii[1]<((p->n[1])-1);ii[1]++)
     #ifdef USE_SAC_3D
        for(ii[2]=1;ii[2]<((p->n[2])-1);ii[2]++)
     #endif
	{ 
              // computecmax3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);

             
                    // atomicExch(&(p->cmax),(wd[fencode3_MODID(p,ii,soundspeed)]));
               #ifdef USE_SAC_3D
                if(wd[encode3_hdv1(p,ii[0],ii[1],ii[2],hdnur+hand)]>(p->maxviscoef))
                    p->maxviscoef=(wd[encode3_hdv1(p,ii[0],ii[1],ii[2],hdnur+hand)]);
               #else
                 if(wd[encode3_hdv1(p,ii[0],ii[1],0,hdnur+hand)]>(p->maxviscoef))
                    p->maxviscoef=(wd[encode3_hdv1(p,ii[0],ii[1],0,hdnur+hand)]);
               #endif

              if(wd[encode3_hdv1(p,ii[0],ii[1],0,hdnur+hand)]>(p->hdmax))
                    p->hdmax=(wd[encode3_hdv1(p,ii[0],ii[1],0,hdnur+hand)]);

              p->hdmean=(p->hdmean)+wd[encode3_hdv1(p,ii[0],ii[1],0,hdnur+hand)];

	}
p->hdmean=(p->hdmean)/(((p->n[0])-2)*((p->n[1]))-2);
 //  }
}
 __syncthreads();



 
}







__global__ void hyperdifvisc4_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim,int hand)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  real maxt=0,max3=0, max1=0;
  
   int ip,jp,ipg,jpg;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  


int shift=order*NVAR*dimp;


   //tmp1  tmp_nuI
   //tmp2  d3r
    //tmp3 d1r
//tmp4    md3r
//tmp5    md1r
//tmp6    d3l
//tmp7    d1l
//tmp8    md3l
//tmp9    md1l







 //  p->maxviscoef=0;
//  p->cmax=1.0;

    //finally update nur and nul
//tmp4    md3r
//tmp5    md1r
   
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(i>1 && i<((p->n[0])-2) && j>1 && j<((p->n[1])-2) && k>1 && k<((p->n[2])-2))
     #else
       if(i>1 && i<((p->n[0])-2) && j>1 && j<((p->n[1])-2))
     #endif
   //if(i>1 && i<((p->n[0])-2) && j>1 && j<((p->n[1])-2))
   {
     //wd[encode3_hdv1(p,i,j,hdnur+hand)]=wtemp2[encode3_hdv1(p,i+1,j+1,tmpnui)];
     if(wtemp[encode3_hdv1(p,i,j,k,tmp5)]>0)
{
//p->cmax=1.0;
     #ifdef USE_SAC_3D
	wd[encode3_hdv1(p,i,j,k,hdnur+hand)]=((dim==0)*(p->dx[0])+(dim==1)*(p->dx[1])+(dim==2)*(p->dx[2]))*(p->cmax)*(p->chyp[field])*wtemp[encode3_hdv1(p,i,j,k,tmp4)]/wtemp[encode3_hdv1(p,i,j,k,tmp5)];
     #else
	wd[encode3_hdv1(p,i,j,k,hdnur+hand)]=((dim==0)*(p->dx[0])+(dim==1)*(p->dx[1]))*(p->cmax)*(p->chyp[field])*wtemp[encode3_hdv1(p,i,j,k,tmp4)]/wtemp[encode3_hdv1(p,i,j,k,tmp5)];
     #endif
       // wd[encode3_hdv1(p,i,j,k,hdnur+hand)]=1.0e-2; 
          //wd[encode3_hdv1(p,i,j,hdnur+hand)]=wtemp[encode3_hdv1(p,i,j,tmp4)];
	//wd[encode3_hdv1(p,i,j,hdnul+hand)]=0.01;
}
     else
        wd[encode3_hdv1(p,i,j,k,hdnur+hand)]=0;


//        wd[encode3_hdv1(p,i,j,k,hdnur+hand)]=1.0e-2; 


   }
}
 __syncthreads();



 
}





__global__ void hyperdifvisc3_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim,int hand)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int is,js,ks;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  real maxt1=0,max3=0, maxt2=0;
  
   int ip,jp,ipg,jpg;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  


int shift=order*NVAR*dimp;


   //tmp1  tmp_nuI
   //tmp2  d3r
    //tmp3 d1r
//tmp4    md3r
//tmp5    md1r
//tmp6    d3l
//tmp7    d1l
//tmp8    md3l
//tmp9    md1l





  //compute md3r and md1r
//tmp4    md3r
//tmp5    md1r
  //js=0;
 // is=0;
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       //if(ii[0]>1 && ii[1]>1 && ii[2]>1 && ii[0]<p->n[0] && ii[1]<p->n[1]  && ii[2]<p->n[2])
       if(i>1 && j>1 && k>1 && i<((p->n[0])-2) && j<((p->n[1])-2)   && k<((p->n[2]))-2)
     #else
       //if(ii[0]>1 && ii[1]>1 && ii[0]<p->n[0] && ii[1]<p->n[1])
       if(i>1 && j>1 && i<((p->n[0])-2) && j<((p->n[1])-2))
     #endif

 // if( i>1 && j>1 && i<((p->n[0])-2) && j<((p->n[1])-2))            
   {
         maxt1=0;

     #ifdef USE_SAC_3D
         for(is=-(dim==0); is<=(dim==0); is++)
                for(js=-(dim==1); js<=(dim==1); js++)
                   for(ks=-(dim==2); ks<=(dim==2); ks++)
                {
                   if(wtemp1[encode3p1_hdv1(p,i+1+is,j+1+js,k+1+ks,d3)]>maxt1)
                         maxt1=wtemp1[encode3p1_hdv1(p,i+1+is,j+1+js,k+1+ks,d3)];

                }
	#else
         for(is=-(dim==0); is<=(dim==0); is++)
                for(js=-(dim==1); js<=(dim==1); js++)
                {
                   if(wtemp1[encode3p1_hdv1(p,i+1+is,j+1+js,k,d3)]>maxt1)
                         maxt1=wtemp1[encode3p1_hdv1(p,i+1+is,j+1+js,k,d3)];

                }
	#endif
          wtemp[encode3_hdv1(p,i,j,k,tmp4)]=maxt1;

         maxt2=0;

     #ifdef USE_SAC_3D
         for(is=-2*(dim==0); is<=2*(dim==0); is++)
                for(js=-2*(dim==1); js<=2*(dim==1); js++)
                   for(ks=-(dim==2); ks<=(dim==2); ks++)
                {
                   if(wtemp1[encode3p1_hdv1(p,i+1+is,j+1+js,k+1+ks,d1)]>maxt2)
                        maxt2=wtemp1[encode3p1_hdv1(p,i+1+is,j+1+js,k+1+ks,d1)];

                }
	#else
         for(is=-2*(dim==0); is<=2*(dim==0); is++)
                for(js=-2*(dim==1); js<=2*(dim==1); js++)
                {
                   if(wtemp1[encode3p1_hdv1(p,i+1+is,j+1+js,k,d1)]>maxt2)
                        maxt2=wtemp1[encode3p1_hdv1(p,i+1+is,j+1+js,k,d1)];

                }
	#endif
          wtemp[encode3_hdv1(p,i,j,k,tmp5)]=maxt2;
   }
}
   __syncthreads();







 
}




__global__ void hyperdifvisc2_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim,int hand)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  real maxt=0,max3=0, max1=0;
  
   int ip,jp,ipg,jpg;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  


int shift=order*NVAR*dimp;






   //tmp1  tmp_nuI
 
//compute d3r and d1r
   //tmp2  d3r
    //tmp3 d1r

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]>1 && ii[1]>1 && ii[2]>1 && ii[0]<p->n[0] && ii[1]<p->n[1]  && ii[2]<p->n[2])
     #else
       if(ii[0]>1 && ii[1]>1 && ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
 
   //if(i>1 && j>1 && i<((p->n[0])) && j<((p->n[1])))       
   { 
     if(hand==0)
     {
	#ifdef USE_SAC_3D
		   wtemp1[encode3p1_hdv1(p,i,j,k,d3)]=fabs(3.0*(wtemp2[encode3p2_hdv1(p,i+(dim==0),j+(dim==1),k+(dim==2),tmpnui)] - wtemp2[encode3p2_hdv1(p,i,j,k,tmpnui)] ) - (wtemp2[encode3p2_hdv1(p,i+2*(dim==0),j+2*(dim==1),k+2*(dim==2),tmpnui)] - wtemp2[encode3p2_hdv1(p,i-(dim==0),j-(dim==1),k-(dim==2),tmpnui)]    ));
	#else
		   wtemp1[encode3p1_hdv1(p,i,j,k,d3)]=fabs(3.0*(wtemp2[encode3p2_hdv1(p,i+(dim==0),j+(dim==1),k,tmpnui)] - wtemp2[encode3p2_hdv1(p,i,j,k,tmpnui)] ) - (wtemp2[encode3p2_hdv1(p,i+2*(dim==0),j+2*(dim==1),k,tmpnui)] - wtemp2[encode3p2_hdv1(p,i-(dim==0),j-(dim==1),k,tmpnui)]    ));
	#endif
     }
     else
     {
	#ifdef USE_SAC_3D
		   wtemp1[encode3p1_hdv1(p,i,j,k,d3)]=fabs(3.0*(wtemp2[encode3p2_hdv1(p,i,j,k,tmpnui)] - wtemp2[encode3p2_hdv1(p,i-(dim==0),j-(dim==1),k-(dim==2),tmpnui)]) - (wtemp2[encode3p2_hdv1(p,i+(dim==0),j+(dim==1),k+(dim==2),tmpnui)] - wtemp2[encode3p2_hdv1(p,i-2*(dim==0),j-2*(dim==1),k-2*(dim==2),tmpnui)]    ));
		   //wtemp1[encode3_hdv1(p,i,j,k,d3)]=fabs(3.0*(wtemp2[encode3_hdv1(p,i,j,k,tmpnui)] - wtemp2[encode3_hdv1(p,i-(dim==0),j-(dim==1),k-(dim==2),tmpnui)]) - (wtemp2[encode3_hdv1(p,i+(dim==0),j+(dim==1),k+(dim==2),tmpnui)] - wtemp2[encode3_hdv1(p,i-2*(dim==0),j-2*(dim==1),k-2*(dim==2),tmpnui)]    ));
	#else
		   wtemp1[encode3p1_hdv1(p,i,j,k,d3)]=fabs(3.0*(wtemp2[encode3p2_hdv1(p,i,j,k,tmpnui)] - wtemp2[encode3p2_hdv1(p,i-(dim==0),j-(dim==1),k,tmpnui)] ) - (wtemp2[encode3p2_hdv1(p,i+(dim==0),j+(dim==1),k,tmpnui)] - wtemp2[encode3p2_hdv1(p,i-2*(dim==0),j-2*(dim==1),k,tmpnui)]    ));
	#endif
     }
   }
}
   __syncthreads();







   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       //if(ii[0]>1 && ii[1]>1 && ii[2]>1 && ii[0]<p->n[0] && ii[1]<p->n[1]  && ii[2]<p->n[2])
       if(i>0 && j>0 && k>0 && i<=((p->n[0])) && j<=((p->n[1]))   && k<=((p->n[2])))
     #else
       //if(ii[0]>1 && ii[1]>1 && ii[0]<p->n[0] && ii[1]<p->n[1])
       if(i>0 && j>0 && i<=((p->n[0])) && j<=((p->n[1])))
     #endif

   //if(i>0 && j>0 && i<=((p->n[0])) && j<=((p->n[1])))            
   { 
     if(hand==0)
     {
     #ifdef USE_SAC_3D
           wtemp1[encode3p1_hdv1(p,i,j,k,d1)]=fabs((wtemp2[encode3p2_hdv1(p,i+(dim==0),j+(dim==1),k+(dim==2),tmpnui)] - wtemp2[encode3p2_hdv1(p,i,j,k,tmpnui)] ));
     #else
           wtemp1[encode3p1_hdv1(p,i,j,k,d1)]=fabs((wtemp2[encode3p2_hdv1(p,i+(dim==0),j+(dim==1),k,tmpnui)] - wtemp2[encode3p2_hdv1(p,i,j,k,tmpnui)] ));
     #endif
     }
     else
     {
     #ifdef USE_SAC_3D
           wtemp1[encode3_hdv1(p,i,j,k,d1)]=fabs((wtemp2[encode3p2_hdv1(p,i,j,k,tmpnui)] - wtemp2[encode3p2_hdv1(p,i-(dim==0),j-(dim==1),k-(dim==2),tmpnui)] ));
     #else
           wtemp1[encode3p1_hdv1(p,i,j,k,d1)]=fabs(wtemp2[encode3p2_hdv1(p,i,j,k,tmpnui)]-(wtemp2[encode3p2_hdv1(p,i-(dim==0),j-(dim==1),k,tmpnui)]   ));
     #endif
     }
   }
}
   __syncthreads();



}



__global__ void hyperdifvisc1a_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim,int hand)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  const int blockdim=blockDim.x;
  const int SZWT=blockdim;
  const int SZWM=blockdim*NVAR;
  int tid=threadIdx.x;
  int i,j,iv;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  real maxt=0,max3=0, max1=0;
  
   int ip,jp,ipg,jpg;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  


int shift=order*NVAR*dimp;
  __shared__ real wts[512];
  __shared__ real wms[512];



   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
    //set viscosities
   //if( i<((p->n[0])) && j<((p->n[1])))
   {
     #ifdef USE_SAC_3D
     wtemp2[encode3p2_hdv1(p,i+1,j+1,k+1,tmpnui)]=wtemp[encode3_hdv1(p,i,j,k,tmp6)];
     #else
     wtemp2[encode3p2_hdv1(p,i+1,j+1,0,tmpnui)]=wtemp[encode3_hdv1(p,i,j,0,tmp6)];
     #endif

   }

   }
   __syncthreads();



   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<(p->n[1]) && ii[2]<(p->n[2]))
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
    //set viscosities
   //if(i<((p->n[0])) && j<((p->n[1])))
   {
	
        bc_hyperdif(wtemp2, p,ii, tmpnui,dim);

   }


    }
   __syncthreads();





 
}


__global__ void hyperdifvisc1_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim,int hand)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  const int blockdim=blockDim.x;
  const int SZWT=blockdim;
  const int SZWM=blockdim*NVAR;
  int tid=threadIdx.x;
  real maxt=0,max3=0, max1=0;
  int i,j,iv;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];


  
   int ip,jp,ipg,jpg;



  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  


int bfac1,bfac2,bfac3;
//int bfac1=(field==rho || field>mom2)+(field>rho && field<energy);
//int bfac2= (field==rho || field>mom2);
//int bfac3=(field>rho && field<energy);
int shift=order*NVAR*dimp;
  __shared__ real wts[512];
  __shared__ real wms[512];




//init temp1 and temp2 to zero 
//the compute element initialising n[0] or n[1] element must do +1 and +2
//this is because we fit the problem geometrically to nixnj elements 
   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
    //set viscosities
   //if(i<((p->n[0])) && j<((p->n[1])))
   {


        for(int f=tmp1; f<=tmp8; f++)
                 wtemp[fencode3_hdv1(p,ii,f)]=0;

        for(int f=d1; f<=d3; f++)
     #ifdef USE_SAC_3D
                 wtemp1[encode3p1_hdv1(p,ii[0],ii[1],ii[2],f)]=0;
                 wtemp2[encode3p2_hdv1(p,ii[0],ii[1],ii[2],tmpnui)]=0;
     #else
                 wtemp1[encode3p1_hdv1(p,ii[0],ii[1],k,f)]=0;
                 wtemp2[encode3p2_hdv1(p,ii[0],ii[1],k,tmpnui)]=0;
     #endif

      if(i==((p->n[0])-1))
      {
        for(int f=d1; f<=d3; f++)
                 wtemp1[encode3p1_hdv1(p,ii[0]+1,ii[1],k,f)]=0;
        wtemp2[encode3p2_hdv1(p,i+1,j,k,tmpnui)]=0;
        wtemp2[encode3p2_hdv1(p,i+2,j,k,tmpnui)]=0;
      }
      if(j==((p->n[1])-1))
      {
          for(int f=d1; f<=d3; f++)
                 wtemp1[encode3p1_hdv1(p,i,j+1,k,f)]=0;
          wtemp2[encode3p2_hdv1(p,i,j+1,k,tmpnui)]=0;
          wtemp2[encode3p2_hdv1(p,i,j+2,k,tmpnui)]=0;
      }

     #ifdef USE_SAC_3D
      if(k==((p->n[2])-1))
      {
          for(int f=d1; f<=d3; f++)
                 wtemp1[encode3p1_hdv1(p,i,j,k+1,f)]=0;
          wtemp2[encode3p2_hdv1(p,i,j,k+1,tmpnui)]=0;
          wtemp2[encode3p2_hdv1(p,i,j,k+2,tmpnui)]=0;
      }

     #endif
      if(j==((p->n[1])-1)  && i==((p->n[0])-1))
      {
          for(int f=d1; f<=d3; f++)
                 wtemp1[encode3p1_hdv1(p,i+1,j+1,k,f)]=0;



          for(int di=0; di<2; di++)
             for(int dj=0; dj<2; dj++)
                   wtemp2[encode3p2_hdv1(p,i+1+di,j+1+dj,k,tmpnui)]=0;
               

      }
     #ifdef USE_SAC_3D
      if(i==((p->n[0])-1)  && k==((p->n[2])-1))
      {
          for(int f=d1; f<=d3; f++)
                 wtemp1[encode3p1_hdv1(p,i+1,j,k+1,f)]=0;
          for(int di=0; di<2; di++)
             for(int dk=0; dk<2; dk++)
                   wtemp2[encode3p2_hdv1(p,i+1+di,j,k+1+dk,tmpnui)]=0;


      }
      #endif
     #ifdef USE_SAC_3D
      if(j==((p->n[1])-1)  && k==((p->n[2])-1))
      {
          for(int f=d1; f<=d3; f++)
                 wtemp1[encode3p1_hdv1(p,i+1,j+1,k,f)]=0;

          for(int dk=0; dk<2; dk++)
             for(int dj=0; dj<2; dj++)
                   wtemp2[encode3p2_hdv1(p,i,j+1+dj,k+1+dk,tmpnui)]=0;


      }
      #endif

     #ifdef USE_SAC_3D
      if(i==((p->n[0])-1) && j==((p->n[1])-1)  && k==((p->n[2])-1))
      {
          for(int f=d1; f<=d3; f++)
                 wtemp1[encode3p1_hdv1(p,i+1,j+1,k+1,f)]=0;
       
          for(int dk=0; dk<2; dk++)
             for(int dj=0; dj<2; dj++)
               for(int di=0; di<2; di++)
                   wtemp2[encode3p2_hdv1(p,i+1+di,j+1+dj,k+1+dk,tmpnui)]=0;


      }
      #endif

   }



  }

   __syncthreads();


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
    //set viscosities
   //if(i<((p->n[0])) && j<((p->n[1])))
   {

        //for(iv=0;iv<NVAR;iv++)
        //               wms[tid+iv*blockdim]=wmod[fencode_hdv1(p,i,j,iv)+shift];
        //wts[tid]=wtemp[fencode_hdv1(p,i,j,tmp6)];
        //temp value for viscosity

       //tmp6  tmpnu
#ifdef USE_SAC
        if(field==energy)
        wtemp[fencode3_hdv1(p,ii,tmp6)]=wmod[fencode3_hdv1(p,ii,energy)+shift]-0.5*((wmod[fencode3_hdv1(p,ii,b1)+shift]*wmod[fencode3_hdv1(p,ii,b1)+shift]+wmod[fencode3_hdv1(p,ii,b2)+shift]*wmod[fencode3_hdv1(p,ii,b2)+shift])+(wmod[fencode3_hdv1(p,ii,mom1)+shift]*wmod[fencode3_hdv1(p,ii,mom1)+shift]+wmod[fencode3_hdv1(p,ii,mom2)+shift]*wmod[fencode3_hdv1(p,ii,mom2)+shift])/(wmod[fencode3_hdv1(p,ii,rho)+shift]+wmod[fencode3_hdv1(p,ii,rhob)+shift] ));
        else
        {
           wtemp[fencode3_hdv1(p,ii,tmp6)]=wmod[fencode3_hdv1(p,ii,field)+shift];
	   if((field ==mom1 || field == mom2))
		wtemp[fencode3_hdv1(p,ii,tmp6)]=wmod[fencode3_hdv1(p,ii,field)+shift]/(((wmod[fencode3_hdv1(p,ii,rho)+shift] +wmod[fencode3_hdv1(p,ii,rhob)+shift])));
        }
        //wtemp2[encode3_hdv1(p,i+1,j+1,k,tmpnui)]=wtemp[fencode3_hdv1(p,ii,tmp6)];



#endif

#ifdef USE_SAC_3D
       if(field==energy)
        wtemp[fencode3_hdv1(p,ii,tmp6)]=wmod[fencode3_hdv1(p,ii,energy)+shift]-0.5*((wmod[fencode3_hdv1(p,ii,b1)+shift]*wmod[fencode3_hdv1(p,ii,b1)+shift]+wmod[fencode3_hdv1(p,ii,b2)+shift]*wmod[fencode3_hdv1(p,ii,b2)+shift]+wmod[fencode3_hdv1(p,ii,b3)+shift]*wmod[fencode3_hdv1(p,ii,b3)+shift])
+(wmod[fencode3_hdv1(p,ii,mom1)+shift]*wmod[fencode3_hdv1(p,ii,mom1)+shift]+wmod[fencode3_hdv1(p,ii,mom2)+shift]*wmod[fencode3_hdv1(p,ii,mom2)+shift]+wmod[fencode3_hdv1(p,ii,mom3)+shift]*wmod[fencode3_hdv1(p,ii,mom3)+shift])/(wmod[fencode3_hdv1(p,ii,rho)+shift]+wmod[fencode3_hdv1(p,ii,rhob)+shift] ));       
       else
       {
          wtemp[fencode3_hdv1(p,ii,tmp6)]=wmod[fencode3_hdv1(p,ii,field)+shift];
	if((field ==mom1 || field == mom2 || field == mom3))
		wtemp[fencode3_hdv1(p,ii,tmp6)]=wmod[fencode3_hdv1(p,ii,field)+shift]/(((wmod[fencode3_hdv1(p,ii,rho)+shift] +wmod[fencode3_hdv1(p,ii,rhob)+shift])));

        }
        //wtemp2[encode3_hdv1(p,i+1,j+1,k+1,tmpnui)]=wtemp[fencode3_hdv1(p,ii,tmp6)];



#endif



        wd[fencode3_hdv1(p,ii,hdnur+hand)]=0;
   }

}
   __syncthreads();




}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdv1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifvisc1(struct params **p,  struct params **d_p,   real **d_wmod,  real **d_wd, int order, real **d_wtemp, real **d_wtemp1, real **d_wtemp2, int field, int dim,int hand)
{

  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

// dim3 dimBlock(dimblock, 1);
 
 //   dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;

    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

     hyperdifvisc1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim,hand);
     hipDeviceSynchronize();
     hyperdifvisc1a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim,hand);
     hipDeviceSynchronize();
     hyperdifvisc2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim,hand);
     hipDeviceSynchronize();
     hyperdifvisc3_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim,hand);
     hipDeviceSynchronize();
     hyperdifvisc4_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim,hand);
     hipDeviceSynchronize();
   
  /*hyperdifvisc5_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim,hand);
     hipDeviceSynchronize();

    hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);

  if(hand==0)
    printf("field right hdmean hdmax %d %8.8g %8.8g \n",field, (*p)->hdmean, (*p)->hdmax);
  else
    printf("field left hdmean hdmax %d %8.8g %8.8g \n",field, (*p)->hdmean, (*p)->hdmax);*/
}







