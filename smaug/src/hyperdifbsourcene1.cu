#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"
#include "../include/iobparams.h"
/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/smaugcukernels.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_hdbne1.cuh"



__global__ void hyperdifbsourcene6_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb, real dt)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
 
  int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  



int shift=order*NVAR*dimp;

     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
     if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
    if(i<((p->n[0])) && j<((p->n[1])))
     #endif

  //if(i<((p->n[0])) && j<((p->n[1])))
	{		               

//dwn1[fencode3_hdbne1(p,iia,energy)]=sb*wtemp[fencode3_hdbne1(p,iia,tmp6)];

dwn1[fencode3_hdbne1(p,iia,b1+ii0)]=sb*wtemp[fencode3_hdbne1(p,iia,tmp4)];


   }

 //__syncthreads();


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
     if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
    if(i<((p->n[0])) && j<((p->n[1])))
     #endif
                         //if(i<(ni) && j<(nj))
                         {
                              //                                                                                  - sign here same as vac maybe a +
                              wmod[fencode3_hdbne1(p,iia,b1+ii0)+(ordero*NVAR*dimp)]=wmod[fencode3_hdbne1(p,iia,b1+ii0)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdbne1(p,iia,b1+ii0)]; 
                             //wmod[fencode3_hdbne1(p,iia,energy)+(ordero*NVAR*dimp)]=wmod[fencode3_hdbne1(p,iia,energy)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdbne1(p,iia,energy)]; 

                         }
              //  }	

  //__syncthreads();  
  
}




__global__ void hyperdifbsourcene6a_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb, real dt)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
 
  int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  



int shift=order*NVAR*dimp;

     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
     if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
    if(i<((p->n[0])) && j<((p->n[1])))
     #endif

  //if(i<((p->n[0])) && j<((p->n[1])))
	{		               

dwn1[fencode3_hdbne1(p,iia,energy)]=sb*wtemp[fencode3_hdbne1(p,iia,tmp6)];

//dwn1[fencode3_hdbne1(p,iia,b1+ii0)]=sb*wtemp[fencode3_hdbne1(p,iia,tmp4)];


   }

 //__syncthreads();


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
     if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
    if(i<((p->n[0])) && j<((p->n[1])))
     #endif
                         //if(i<(ni) && j<(nj))
                         {
                              //                                                                                  - sign here same as vac maybe a +
                              //wmod[fencode3_hdbne1(p,iia,b1+ii0)+(ordero*NVAR*dimp)]=wmod[fencode3_hdbne1(p,iia,b1+ii0)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdbne1(p,iia,b1+ii0)]; 
                             wmod[fencode3_hdbne1(p,iia,energy)+(ordero*NVAR*dimp)]=wmod[fencode3_hdbne1(p,iia,energy)+(ordero*NVAR*dimp)]+dt*dwn1[fencode3_hdbne1(p,iia,energy)]; 


  //  if(i==127 && j==252)
  //    p->test=wmod[fencode3_hdbne1(p,iia,energy)+(ordero*NVAR*dimp)];



                         }
              //  }	

  //__syncthreads();  
  
}





__global__ void hyperdifbsourcene5_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb, real dt)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  real dy=p->dx[1];
  real dx=p->dx[0];

  int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
     if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1)  && k<((p->n[2])-1))
     #else
    if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
     #endif
 
  //if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {



       wtemp[fencode3_hdbne1(p,iia,tmp6)]=grad13n_hdbne1(wtemp,wd,p,iia,tmp5,mm);

   }


//__syncthreads();




}



__global__ void hyperdifbsourcene4_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb, real dt)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  //real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];

  int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  


int shift=order*NVAR*dimp;



     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
     if( i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
    if( i<((p->n[0])) && j<((p->n[1])))
     #endif
  //if( i<((p->n[0])) && j<((p->n[1])))
  {
wtemp[fencode3_hdbne1(p,iia,tmp5)]=wtemp[fencode3_hdbne1(p,iia,tmp3)]*wmod[(shift)+fencode3_hdbne1(p,iia,b1+jj)];
   }


//__syncthreads();



  
}



__global__ void hyperdifbsourcene3_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int m,ii1;
  real fip,fim1,tmpc;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];

   int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  


int shift=order*NVAR*dimp;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
     if(i>0 && j>0 &&  k>0 && i<((p->n[0])-1) && j<((p->n[1])-1) && k<((p->n[2])-1))
     #else
     if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
     #endif
  //if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {

 wtemp[fencode3_hdbne1(p,iia,tmp4)]=grad13n_hdbne1(wtemp,wd,p,iia,tmp3,mm);

   // if(i==252 && j==127)
   //   p->test=wtemp[fencode3_hdbne1(p,iia,tmp3)];


   }


//__syncthreads();








   

  
}





__global__ void hyperdifbsourcene2_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,k;
  int m,ii1;
  //real fip,fim1,tmpc;
  //int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  //real dt=p->dt;
  //real dy=p->dx[1];
  //real dx=p->dx[0];

   int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;


  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
     if(i<((p->n[0])) && j<((p->n[1]))&& k<((p->n[2])))
     #else
     if(i<((p->n[0])) && j<((p->n[1])))
     #endif
     {



      wtemp[fencode3_hdbne1(p,iia,tmp3)]=wtemp[fencode3_hdbne1(p,iia,tmp2)]*(wd[fencode3_hdbne1(p,iia,hdnul)]+wd[fencode3_hdbne1(p,iia,hdnur)])/2;

 //wtemp[fencode3_hdbne1(p,iia,tmp3)]=wtemp[fencode3_hdbne1(p,iia,tmp2)]*3.75;

    //if(i==127 && j==252)
    //  p->test=wtemp[fencode3_hdbne1(p,iia,tmp2)];
   // if(i==127 && j==252)
   //   p->test=wtemp[fencode3_hdbne1(p,iia,tmp3)];

     }


//__syncthreads();



  
}



__global__ void hyperdifbsourcene1b_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,k;
  int m,ii1;
  //real fip,fim1,tmpc;
  //int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  //real dt=p->dt;
  //real dy=p->dx[1];
  //real dx=p->dx[0];


   int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  


int shift=order*NVAR*dimp;


     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i>0 && j >0 && k>0 && i<((p->n[0])-1) && j<((p->n[1])-1) && k<((p->n[2])-1))
     #else
     if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
     #endif
 // if(i>0 && j >0 && i<((p->n[0])-1) && j<((p->n[1])-1))
  {


       wtemp[fencode3_hdbne1(p,iia,tmp2)]=/*0.25**/grad13n_hdbne1(wtemp,wd,p,iia,tmp1,dim);
   //   wtemp[fencode3_hdbne1(p,iia,tmp2)]=/*0.25**/grad13n_hdbne1(wmod+shift,wd,p,iia,b1+field,dim);
//wmod[(shift)+fencode3_hdbne1(p,iia,b1+field)]
    //if(i==127 && j==252)
    //  p->test=grad13n_hdbne1(wtemp,wd,p,iia,tmp2,dim);
    //if(i==127 && j==252)
    //  p->test=grad13n_hdbne1(wmod+shift,wd,p,iia,b1+field,dim);
//if(i==127 && j==252)
//    p->test=wtemp[fencode3_hdbne1(p,iia,tmp2)];
   }


//__syncthreads();



  
}


__global__ void hyperdifbsourcene1a_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,k;
  int m,ii1;
  //real fip,fim1,tmpc;
  //int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  //real dt=p->dt;
  //real dy=p->dx[1];
  //real dx=p->dx[0];
   int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  


int shift=order*NVAR*dimp;


   

     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

 



     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
      if(i<((p->n[0])) && j<((p->n[1])))
     #endif
  //if( i<((p->n[0])) && j<((p->n[1])))
  {

wtemp[fencode3_hdbne1(p,iia,tmp1)]=wmod[(shift)+fencode3_hdbne1(p,iia,b1+field)];

//wtemp[fencode3_hdbne1(p,iia,tmp1)]=wmod[fencode3_hdbne1(p,iia,b1+field)];


  //  if(i==127 && j==127)
  //    p->test=wmod[shift+fencode3_hdbne1(p,iia,b1+field)];


   }


//__syncthreads();



}




__global__ void hyperdifbsourcene1_parallel(struct params *p,  real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real *wtemp, int field, int dim, int jj, int ii0,int mm,real sb)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,k;
  int m,ii1;
  //real fip,fim1,tmpc;
  //int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  //real dt=p->dt;
  //real dy=p->dx[1];
  //real dx=p->dx[0];
   int ip,jp;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  


int shift=order*NVAR*dimp;


   

     iia[0]=ip;
     iia[1]=jp;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1])) && k<((p->n[2])))
     #else
      if(i<((p->n[0])) && j<((p->n[1])))
     #endif
  //init rhol and rhor
  //if(i<((p->n[0])) && j<((p->n[1])))
  {
    for(int f=tmp1; f<=tmp8; f++)	
        wtemp[fencode3_hdbne1(p,iia,f)]=0.0;

   dwn1[fencode3_hdbne1(p,iia,energy)]=0.0;
   dwn1[fencode3_hdbne1(p,iia,b1+ii0)]=0.0;

  }

 //__syncthreads();




}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdbne1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}






int cuhyperdifbsourcene1(struct params **p,  struct params **d_p,   real **d_wmod, real **d_dwn1, real **d_wd, int order,int ordero, real **d_wtemp, int field, int dim, int jj, int ii0,int mm,real sb,real dt)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     hyperdifbsourcene1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb);
    hipDeviceSynchronize();
     hyperdifbsourcene1a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb);
    hipDeviceSynchronize();

     hyperdifbsourcene1b_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb);
    hipDeviceSynchronize();

     hyperdifbsourcene2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb);
    hipDeviceSynchronize();
     hyperdifbsourcene3_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb);
    hipDeviceSynchronize();
     hyperdifbsourcene4_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb,dt);
    hipDeviceSynchronize();
     hyperdifbsourcene6_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb,dt);
    hipDeviceSynchronize(); 
     hyperdifbsourcene5_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb,dt);
    hipDeviceSynchronize();
     hyperdifbsourcene6a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod, *d_dwn1,  *d_wd, order,ordero,*d_wtemp, field, dim,jj,ii0,mm,sb,dt);
    hipDeviceSynchronize(); 

hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);
//printf("e %d  %10.20g\n",mm,(*p)->test);
}







