#include "hip/hip_runtime.h"
//#define MODID pre


#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"
#include "../include/iobparams.h"
/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/smaugcukernels.h"
#include "../include/gradops_cdf.cuh"
#include "../include/dervfields_cdf.cuh"
/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
__global__ void computevels_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     









  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))



     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		               

                        switch(dir)
                        {
                         case 0:
                          #ifdef USE_SAC_3D
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2))
     			  #endif
                         //if(i<(ni)  && j >1 &&  j<(nj-1))
                                           computevel3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
                         break;
                         case 1:
                          #ifdef USE_SAC_3D
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2))
     			  #endif
                         //if(i>1 &&  i<(ni-1) && j<(nj))
                                           computevel3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
                         break;
                          #ifdef USE_SAC_3D
                         case 2:

       				if(ii[2]<p->n[2] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[1]>1 && ii[1]<(p->n[1]-2))

                         //if(i>1 &&  i<(ni-1) && j<(nj))
                                           computevel3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
                         break;
                         #endif
                        }


         }


              __syncthreads();











  
}


__global__ void computept_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     




     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		               

                        switch(dir)
                        {
                         case 0:
                          #ifdef USE_SAC_3D
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2))
     			  #endif
                         //if(i<(ni)  && j >1 &&  j<(nj-1))
                                           
                           computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
                         break;
                         case 1:
                          #ifdef USE_SAC_3D
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2))
     			  #endif
                         //if(i>1 &&  i<(ni-1) && j<(nj))
                                           
                            computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
                         break;
                          #ifdef USE_SAC_3D
                         case 2:

       				if(ii[2]<p->n[2] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[1]>1 && ii[1]<(p->n[1]-2))

                         //if(i>1 &&  i<(ni-1) && j<(nj))
                                          
                                computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
                         break;
                         #endif
                        }


         }




  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))



  /*   ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		               

	     #ifdef ADIABHYDRO
	       
	       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
	     #else
	       
	       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
	     #endif */        
              /* switch(dir)
                        {
                         case 0:
                          #ifdef USE_SAC_3D
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2))
     			  #endif
				     {
				     #ifdef ADIABHYDRO
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #else
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #endif
				     }
                         break;
                         case 1:
                          #ifdef USE_SAC_3D
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2))
     			  #endif
				     {
				     #ifdef ADIABHYDRO
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #else
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #endif
				     }
                         break;
                          #ifdef USE_SAC_3D
                         case 2:

       				if(ii[2]<p->n[2] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[1]>1 && ii[1]<(p->n[1]-2))
				     {

				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);

				     }
                         break;
                         #endif
                        }*/


        /* }*/


              __syncthreads();











  
}


__global__ void computeptzero_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     




     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		               

                        wd[fencode3_cdf(p,ii,pressuret)]=0.0;
                        


         }




  


              __syncthreads();











  
}



__global__ void computepk_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     









  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))



     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		               

	     #ifdef ADIABHYDRO
	       
	       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
	     #else
	       
	       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
	     #endif         
              /* switch(dir)
                        {
                         case 0:
                          #ifdef USE_SAC_3D
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2))
     			  #endif
				     {
				     #ifdef ADIABHYDRO
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #else
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #endif
				     }
                         break;
                         case 1:
                          #ifdef USE_SAC_3D
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2))
     			  #endif
				     {
				     #ifdef ADIABHYDRO
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #else
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #endif
				     }
                         break;
                          #ifdef USE_SAC_3D
                         case 2:

       				if(ii[2]<p->n[2] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[1]>1 && ii[1]<(p->n[1]-2))
				     {

				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);

				     }
                         break;
                         #endif
                        }*/


         }


              __syncthreads();











  
}


__global__ void computepbg_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp;
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#else
   dimp=((p->n[0]))*((p->n[1]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     









  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))



     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		               

	     #ifdef ADIABHYDRO
	       computepbg3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
	       
	     #else
	       computepbg3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
	       
	     #endif         
              /* switch(dir)
                        {
                         case 0:
                          #ifdef USE_SAC_3D
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2))
     			  #endif
				     {
				     #ifdef ADIABHYDRO
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #else
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #endif
				     }
                         break;
                         case 1:
                          #ifdef USE_SAC_3D
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2))
     			  #endif
				     {
				     #ifdef ADIABHYDRO
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #else
				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				     #endif
				     }
                         break;
                          #ifdef USE_SAC_3D
                         case 2:

       				if(ii[2]<p->n[2] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[1]>1 && ii[1]<(p->n[1]-2))
				     {

				       computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
				       computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);

				     }
                         break;
                         #endif
                        }*/


         }


              __syncthreads();











  
}


__global__ void computemaxc_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     




   /*for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               computec3_cdf(wmod+(order*dimp*NVAR),wd,p,ii,dir);
               //p->cmax=0.0;
        }

}*/
              __syncthreads();



if(iindex==0)
{
   
 //  for(ipg=0;ipg<(p->npgp[0]);ipg++)
 //  for(jpg=0;jpg<(p->npgp[1]);jpg++)
  // {

  //   i=ip*(p->npgp[0])+ipg;
 //    j=jp*(p->npgp[1])+jpg;
   //if( i<((p->n[0])) && j<((p->n[1])))
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
    p->cmax=0.0;
    for(ii[0]=0;ii[0]<((p->n[0]));ii[0]++)
      for(ii[1]=0;ii[1]<((p->n[1]));ii[1]++)
     #ifdef USE_SAC_3D
        for(ii[2]=0;ii[2]<((p->n[2]));ii[2]++)
     #endif
	{ 
               computecmax3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);




	}

 //  }
}
 __syncthreads(); 

//p->cmax=1.0;



  
}

//from http://www.nvidia.com/object/cuda_sample_data-parallel.html#reduction
/*
    This version uses n/2 threads --
    it performs the first level of reduction when reading from global memory
*/
__global__ void fastcomputemaxc_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp,ipg,jpg;
        extern __shared__ real sdata[];
  // __shared__ float sdata[];
 //real sdata[dimp];
  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     



    // perform first level of reduction,
    // reading from global memory, writing to shared memory
   sdata[tid]=0.0;


     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               //computec3_cdf(wmod+(order*dimp*NVAR),wd,p,ii,dir);
               //p->cmax=0.0;
               
               if(wd[fencode3_cdf(p,ii,cfast)]>sdata[tid])
                    sdata[tid]=wd[fencode3_cdf(p,ii,cfast)];
        }


              __syncthreads();

    // do reduction in shared mem
    for(unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            //sdata[tid] += sdata[tid + s];
            //if(sdata[tid]>sdata[0])
             //   sdata[0]=sdata[tid];
            if(sdata[tid+s]>sdata[tid])
                sdata[tid]=sdata[tid+s];
        }
        __syncthreads();
    }


    if (tid == 0) p->cmax = sdata[0];

  /* for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     ii[0]=ip*(p->npgp[0])+ipg;
     ii[1]=jp*(p->npgp[1])+jpg;
     #ifdef USE_SAC_3D
	   ii[2]=kp*(p->npgp[2])+kpg;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               //computec3_cdf(wmod+(order*dimp*NVAR),wd,p,ii,dir);
               //p->cmax=0.0;
               
               if(wd[fencode3_cdf(p,ii,cfast)]>(p->cmax))
                    sdata[tid]=wd[fencode3_cdf(p,ii,cfast)];
        }

}
              __syncthreads();*/

    // do reduction in shared mem
    /*for(unsigned int s=blockDim.x/2; s>0; s>>=1) {
        if (tid < s) {
            //sdata[tid] += sdata[tid + s];
            //if(sdata[tid]>sdata[0])
             //   sdata[0]=sdata[tid];
            if(sdata[tid+s]>sdata[tid])
                sdata[tid]=sdata[tid+s];
        }
        __syncthreads();
    }


    if (tid == 0) p->cmax = sdata[0];*/

  
}



/*
    Parallel sum reduction using shared memory
    - takes log(n) steps for n input elements
    - uses n threads
    - only works for power-of-2 arrays
*/

/* This reduction interleaves which threads are active by using the modulo
   operator.  This operator is very expensive on GPUs, and the interleaved 
   inactivity means that no whole warps are active, which is also very 
   inefficient */


//from http://www.nvidia.com/object/cuda_sample_data-parallel.html#reduction
/* This reduction interleaves which threads are active by using the modulo
   operator.  This operator is very expensive on GPUs, and the interleaved 
   inactivity means that no whole warps are active, which is also very 
   inefficient */
__global__ void reduction0computemaxc_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;
        extern __shared__ real sdata[];
 
  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif

int tnumThreadsPerBlock = 128;
    
int numBlocks = (dimp+tnumThreadsPerBlock-1) / tnumThreadsPerBlock;
  real temp[1024];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
   //sdata[tid]=0.0;
    if(iindex<1024)
      temp[iindex]=0.0;

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

   /*  #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif*/
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
//	{
 //determin cmax
               //computec3_cdf(wmod+(order*dimp*NVAR),wd,p,ii,dir);
               //p->cmax=0.0;
               
              // if(wd[fencode3_cdf(p,ii,cfast)]>(p->cmax))


         if(iindex<dimp)
                    sdata[tid]=wd[fencode3_cdf(p,ii,cfast)];

              __syncthreads();


    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x; s *= 2) {



        // modulo arithmetic is slow!
        if ((tid % (2*s)) == 0) {
            if(sdata[tid+s]>sdata[tid])
                 sdata[tid]=sdata[tid + s];
            
        }
        // strided indexing using sequential addressing is better!
        /*int tindex=2*s*tid;
        if (tindex<blockDim.x) {
            if(sdata[tid+s]>sdata[tid])
                 sdata[tid]=sdata[tid + s];
        }
        __syncthreads();*/
         __syncthreads();
    }

    __syncthreads();
    if(tid==0)
      temp[blockIdx.x]=sdata[0];
__syncthreads();
    if(iindex==0)
       for(int i=0; i<numBlocks; i++)
         if(temp[i]>(p->cmax)) p->cmax=temp[i];
     if (tid == 0 && p->cmax<sdata[0] ) p->cmax = sdata[0];
 


    /* ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               //computec3_cdf(wmod+(order*dimp*NVAR),wd,p,ii,dir);
               //p->cmax=0.0;
               
               if(wd[fencode3_cdf(p,ii,cfast)]>(p->cmax))
                    p->cmax=wd[fencode3_cdf(p,ii,cfast)];
        }


              __syncthreads();*/
 
//        }

//}
//p->cmax=1.0;
 
}




__global__ void newreduction0computemax_parallel(real *cmax, real *temp,int ndimp)
{
  //real *cmax, real *temp, int ndimp

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  extern __shared__ double partialResult[];

  int i;
   partialResult[tid]=0.0;
   if(iindex<ndimp)
              partialResult[tid]=temp[iindex];
  __syncthreads();


for(unsigned int s=1; s < blockDim.x; s *= 2) {
        if ((tid % (2*s)) == 0) {
            if(partialResult[tid+s]>partialResult[tid])
                 partialResult[tid]=partialResult[tid + s];
        }
        __syncthreads();
    }

    __syncthreads();
    if(tid==0)
    {
      cmax[blockIdx.x]=partialResult[0];
      temp[blockIdx.x]=partialResult[0];
     }
     __syncthreads();
}



__global__ void myreduction0computemaxc_parallel(struct params *p,   real *wmod, real *wd, int order, int dir, real *temp,int ndimp,int s)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;
//        extern __shared__ real sdata[];
 
  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif

int tnumThreadsPerBlock = 128;
    
int numBlocks = (dimp+tnumThreadsPerBlock-1) / tnumThreadsPerBlock;
  //real temp[dimp];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
   //sdata[tid]=0.0;
   // if(iindex<1024)
    //  temp[iindex]=0.0;

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif
    //int s=1;


   
    //while(((s*=2)<=((ndimp/2)-1)) && ((iindex+s)<ndimp)) {
    if((iindex+s)<ndimp)
            if(temp[iindex+s]>temp[iindex])
                 temp[iindex]=temp[iindex + s];
            
       // }

       //  __syncthreads();
    

   // __syncthreads();

   if(iindex==0)
      p->cmax=temp[0];


 
}



__global__ void myreduction0computemaxcourant_parallel(struct params *p,   real *wmod, real *wd, int order, int dir, real *temp,int ndimp,int s)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;
//        extern __shared__ real sdata[];
 
  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif

int tnumThreadsPerBlock = 128;
    
int numBlocks = (dimp+tnumThreadsPerBlock-1) / tnumThreadsPerBlock;
  //real temp[dimp];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
   //sdata[tid]=0.0;
   // if(iindex<1024)
    //  temp[iindex]=0.0;

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif
    //int s=1;


   
    //while(((s*=2)<=((ndimp/2)-1)) && ((iindex+s)<ndimp)) {
    if((iindex+s)<ndimp)
            if(temp[iindex+s]>temp[iindex])
                 temp[iindex]=temp[iindex + s];
            
       // }

       //  __syncthreads();
    

   // __syncthreads();

   if(iindex==0  && (p->maxcourant<temp[0]))
      p->maxcourant=temp[0];


 
}




__global__ void zeropadmaxc_parallel(struct params *p,   real *wmod, real *wd, int order, int dir, real *temp, int ndimp)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
 
  if(iindex<ndimp)
      temp[iindex]=0.0;

}

__global__ void zeropadmaxcourant_parallel(struct params *p,   real *wmod, real *wd, int order, int dir, real *temp, int ndimp)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
 
  //if(iindex<ndimp)
  //    temp[iindex]=0.0;

  unsigned int tid = threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;
//        extern __shared__ real sdata[];
 
  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif

    //if(iindex<ndimp)
    //  temp[iindex]=0.0;
  
//int numBlocks = (dimp+tnumThreadsPerBlock-1) / tnumThreadsPerBlock;
  //real temp[dimp];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
   //sdata[tid]=0.0;
   // if(iindex<1024)
    //  temp[iindex]=0.0;

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif
    //int s=1;

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
             temp[iindex]=temp[iindex]/(wd[fencode3_cdf(p,ii,delx1+dir)]);
       else
            temp[index]=0.0;







}

__global__ void reduction0computemaxcfast_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


 // int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

    
    unsigned int iindex = blockIdx.x*(blockDim.x*2) + threadIdx.x;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;
    extern __shared__ real sdata[];



    // perform first level of reduction,
    // reading from global memory, writing to shared memory
   sdata[tid]=0.0;

                   sdata[tid]=wd[blockDim.x+(cfast*dimp)+iindex];

              __syncthreads();


    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x; s *= 2) {
        // modulo arithmetic is slow!
        if ((tid % (2*s)) == 0) {
            if(sdata[tid+s]>sdata[tid])
                 sdata[tid]=sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0 && (p->cmax)<sdata[0] ) p->cmax = sdata[0];
 __syncthreads();



 
}




//from http://www.nvidia.com/object/cuda_sample_data-parallel.html#reduction
/* This reduction interleaves which threads are active by using the modulo
   operator.  This operator is very expensive on GPUs, and the interleaved 
   inactivity means that no whole warps are active, which is also very 
   inefficient */
__global__ void reductiona0computemaxc_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;
        extern __shared__ real sdata[];
   //__shared__ float sdata[];
 //real sdata[dimp];
  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     



    // perform first level of reduction,
    // reading from global memory, writing to shared memory
   sdata[tid]=0.0;

   if(iindex<dimp)
      sdata[tid]=wd[iindex+(dimp*cfast)];

       /* if(iindex<dimp)
               if(wd[iindex+(dimp*cfast)]>(p->cmax))
                    sdata[tid]=wd[iindex+(dimp*cfast)];*/

              __syncthreads();


    // do reduction in shared mem
    for(unsigned int s=1; s < blockDim.x; s *= 2) {
        // modulo arithmetic is slow!
        if ((tid % (2*s)) == 0) {
            if(sdata[tid+s]>sdata[tid])
                 sdata[tid]=sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) p->cmax = sdata[0];



     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               //computec3_cdf(wmod+(order*dimp*NVAR),wd,p,ii,dir);
               //p->cmax=0.0;
               
               if(wd[fencode3_cdf(p,ii,cfast)]>(p->cmax))
                    p->cmax=wd[fencode3_cdf(p,ii,cfast)];
        }


              __syncthreads();
  
}


__global__ void computec_parallel(struct params *p,   real *wmod, real *wd, int order, int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     






 p->cmax=0.0;


     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
	{
 //determin cmax
               computec3_cdf(wmod+(order*dimp*NVAR),wd,p,ii,dir);
               //p->cmax=0.0;
        }


              __syncthreads();












  
}


__global__ void computedervfields_parallel(struct params *p,   real *wmod, real *wd, int order)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
//  real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     
int dir=0;

 

/*                if(jp==0 && ip==63)
                {
     ii[0]=ip;
     ii[1]=jp;

             // printf("density at 128,128=%12.10f %d\n", wmod[fencode3_cdf(p,ii,rho)], order);    
          printf("density at 128,128=%d %d %12.10f %12.10f  \n",ip,jp, wd[fencode3_cdf(p,ii,delx1)], wd[fencode3_cdf(p,ii,delx2)]);          
}

                if(jp==2 && ip==63)
                {
     ii[0]=ip;
     ii[1]=jp;

             // printf("density at 128,128=%12.10f %d\n", wmod[fencode3_cdf(p,ii,rho)], order);    
          printf("density at 128,128=%d %d %12.10f %12.10f  \n",ip,jp, wd[fencode3_cdf(p,ii,delx1)], wd[fencode3_cdf(p,ii,delx2)]);          
}



                if(jp==255 && ip==63)
                {
     ii[0]=ip;
     ii[1]=jp;

             // printf("density at 128,128=%12.10f %d\n", wmod[fencode3_cdf(p,ii,rho)], order);    
          printf("density at 128,128=%d %d %12.10f %12.10f  \n", ip,jp,wd[fencode3_cdf(p,ii,delx1)], wd[fencode3_cdf(p,ii,delx2)]);          
}



                if(jp==253 && ip==63)
                {
     ii[0]=ip;
     ii[1]=jp;

             // printf("density at 128,128=%12.10f %d\n", wmod[fencode3_cdf(p,ii,rho)], order);    
          printf("density at 128,128=%d %d %12.10f %12.10f  \n", ip,jp,wd[fencode3_cdf(p,ii,delx1)], wd[fencode3_cdf(p,ii,delx2)]);          
}






                if(jp==63 && ip==0)
                {
     ii[0]=ip;
     ii[1]=jp;

             // printf("density at 128,128=%12.10f %d\n", wmod[fencode3_cdf(p,ii,rho)], order);    
          printf("density at 128,128=%d %d %12.10f %12.10f  \n",ip,jp, wd[fencode3_cdf(p,ii,delx1)], wd[fencode3_cdf(p,ii,delx2)]);          
}



                if(jp==63 && ip==2)
                {
     ii[0]=ip;
     ii[1]=jp;

             // printf("density at 128,128=%12.10f %d\n", wmod[fencode3_cdf(p,ii,rho)], order);    
          printf("density at 128,128=%d %d %12.10f %12.10f  \n",ip,jp, wd[fencode3_cdf(p,ii,delx1)], wd[fencode3_cdf(p,ii,delx2)]);          
}






                if(jp==63 && ip==255)
                {
     ii[0]=ip;
     ii[1]=jp;

             // printf("density at 128,128=%12.10f %d\n", wmod[fencode3_cdf(p,ii,rho)], order);    
          printf("density at 128,128=%d %d %12.10f %12.10f  \n", ip,jp,wd[fencode3_cdf(p,ii,delx1)], wd[fencode3_cdf(p,ii,delx2)]);          
}




                if(jp==63 && ip==253)
                {
     ii[0]=ip;
     ii[1]=jp;

             // printf("density at 128,128=%12.10f %d\n", wmod[fencode3_cdf(p,ii,rho)], order);    
          printf("density at 128,128=%d %d %12.10f %12.10f  \n", ip,jp,wd[fencode3_cdf(p,ii,delx1)], wd[fencode3_cdf(p,ii,delx2)]);          
}*/







if(order == 0)
{

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		
               for(int f=vel1; f<=pkb; f++)
                        wd[fencode3_cdf(p,ii,f)]=0; 

                //Here we set the current field values (order==0)to the
                //values which were updated (order=1)
                //hence the dimp*NVAR term in wmod on RHS of expression 
		#ifdef USE_SAC_3D
		  for(int f=rho; f<=b3; f++)
                  	//wmod[fencode3_cdf(p,ii,f)+dimp*NVAR]=wmod[fencode3_cdf(p,ii,f)];                    
                        wmod[fencode3_cdf(p,ii,f)]=wmod[fencode3_cdf(p,ii,f)+dimp*NVAR]; 




		#else
		  for(int f=rho; f<=b2; f++)
                  	//wmod[fencode3_cdf(p,ii,f)+dimp*NVAR]=wmod[fencode3_cdf(p,ii,f)]; 
                         wmod[fencode3_cdf(p,ii,f)]=wmod[fencode3_cdf(p,ii,f)+dimp*NVAR];


                      
		#endif               



// for(int field=rho;field<=rho ; field++)
//if(  (p->ipe)==0  && ((p)->it)==1 && ( isnan(wmod[fencode3_cdf(p,ii,field)])|| wmod[fencode3_cdf(p,ii,field)]==0 ))
//if(  /*(p->ipe)==0  &&*/ (  wmod[fencode3_cdf(p,ii,field)]==0 ))
//       { 
//    				printf("nant %d %d %d %d %lg %lg \n",ii[0],ii[1],field,dir, wmod[fencode3_cdf(p,ii,rho)],wmod[fencode3_cdf(p,ii,field)+dimp*NVAR] );
//;//wmod[fencode3_cdf(p,ii,rho)]=0.221049;
//;//wmod[fencode3_cdf(p,ii,field)+dimp*NVAR]=0.221049;
//}



        }




}

               __syncthreads();



  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))



     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if( ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
	{		               
             #ifdef ADIABHYDRO
               //computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
               //computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
             #else
               //computevel3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
               //computej3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
               //computepk3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
               //computept3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);

               computebdotv3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);
               //computedivb3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);

             #endif

         }


              __syncthreads();

  
}

/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cdf(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cucomputedervfields(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   ////hipSetDevice(selectedDevice);
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif  

 dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;

hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
     computedervfields_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order);

     hipDeviceSynchronize();
 

    hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //checkErrors("copy data from device");


 


}

int cucomputevels(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order, int dir)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   ////hipSetDevice(selectedDevice);
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

 //dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     computevels_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);

     hipDeviceSynchronize();
 

   // hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //checkErrors("copy data from device");


 


}

int cucomputemaxc(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order, int dir, real **wd, real **d_wtemp)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));
  
  real fn,fractn,in;
  int ndimp;

  double *d_cmax;
  double *d_bmax;

////hipSetDevice(selectedDevice);
   int nit=100;
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

    fn=log(dimp)/log(2.0);
    fractn=modf(fn,&in);
    
  if(fractn>0)
  {
   fn+=1;
   ndimp=(int)pow(2,fn);
  }
  else
   ndimp=dimp;

  //Number threads per block
  int NTPB=512;

  //Num blocks is determined by size of zeropadded 2^n size array
  int numBlocks = (ndimp+NTPB-1) / NTPB;

  //Shared memory
  int smemSize = NTPB * sizeof(double);

  //Array to store maximum values for reduction in host memory 
  double *h_cmax = (double*)malloc(numBlocks*sizeof(double));

  hipMalloc((void**)&d_cmax, numBlocks*sizeof(double)); 
  hipMalloc((void**)&d_bmax, numBlocks*sizeof(double)); //Array to store maximum values for reduction in GPU global memory

  //set maximum value to zero and update values in GPU memory
  (*p)->cmax=0.0;
  hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
 
  //determine maximum value of magneto-acoustic fast mode
  zeropadmaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir, *d_wtemp,ndimp);

  hipMemcpy(*wd, *d_wd, NDERV*dimp*sizeof(real), hipMemcpyDeviceToHost);
  hipMemcpy(*d_wtemp, ((*wd)+(cfast*dimp)), dimp*sizeof(real), hipMemcpyHostToDevice);
  int i=0;


  //find the maximum in each block
  for(i=0;i<numBlocks;i++)
                h_cmax[i]=0;
  hipMemcpy(d_bmax, h_cmax, numBlocks*sizeof(double), hipMemcpyHostToDevice);

  newreduction0computemax_parallel<<<numBlocks,NTPB,smemSize>>>(d_bmax,*d_wtemp,ndimp);
  hipDeviceSynchronize();
  hipMemcpy(h_cmax, d_bmax, numBlocks*sizeof(double), hipMemcpyDeviceToHost);

  //compare the maxima for all of the blocks and determine maximum value
  for( i=0;i<numBlocks;i++)          		
                if(h_cmax[i]>((*p)->cmax)) ((*p)->cmax)=h_cmax[i];


 //determine maximum value of sound speed
 hipMemcpy(*d_wtemp, ((*wd)+(soundspeed*dimp)), dimp*sizeof(real), hipMemcpyHostToDevice);
 zeropadmaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir, *d_wtemp,ndimp);
 for(i=0;i<numBlocks;i++)
                h_cmax[i]=0;
 hipMemcpy(d_bmax, h_cmax, numBlocks*sizeof(double), hipMemcpyHostToDevice);

 newreduction0computemax_parallel<<<numBlocks,NTPB,smemSize>>>(d_bmax,*d_wtemp,ndimp);
 hipDeviceSynchronize();

 hipMemcpy(h_cmax, d_bmax, numBlocks*sizeof(double), hipMemcpyDeviceToHost);
  //compare the maxima for all of the blocks and determine maximum value
  for( i=0;i<numBlocks;i++)          		
                if(h_cmax[i]>((*p)->cmax)) ((*p)->cmax)=h_cmax[i];

 
  int oldnumBlocks,newnumBlocks;
  newnumBlocks=numBlocks;

  //printf("loop over blocks %d\n\n\n",newnumBlocks);
  /*while(newnumBlocks>1)
  {

      
       //hipMemcpy(d_wtemp, h_cmax, newnumBlocks*sizeof(double), hipMemcpyHostToDevice);
    //hipMemcpy(h_cmax, d_wtemp, newnumBlocks*sizeof(double), hipMemcpyDeviceToHost);
  //for (i=0; i<oldnumBlocks; i++)
   // {
    //  fprintf(stdout,"cmax# %d %f\n",i, h_cmax[i]);
   // }


	  for( i=0;i<newnumBlocks;i++)
          {
           //printf("gt10 %d %g\n",i, h_cmax[i]);		
        h_cmax[i]=0;
               
          }

	  hipMemcpy(d_bmax, h_cmax, newnumBlocks*sizeof(double), hipMemcpyHostToDevice);



       oldnumBlocks=newnumBlocks;
  	newnumBlocks = (newnumBlocks+NTPB-1) / NTPB;
            // printf("blocsk  %d %d\n",newnumBlocks,oldnumBlocks);

  	newreduction0computemax_parallel<<<newnumBlocks,NTPB,smemSize>>>(d_bmax,*d_wtemp,oldnumBlocks);
       hipDeviceSynchronize();
       hipMemcpy(h_cmax, d_bmax, oldnumBlocks*sizeof(double), hipMemcpyDeviceToHost);
     //hipMemcpy(h_cmax, d_wtemp, oldnumBlocks*sizeof(double), hipMemcpyDeviceToHost);*/
  /*for (i=0; i<oldnumBlocks; i++)
    {
      fprintf(stdout,"cmax# %d %f\n",i, h_cmax[i]);
    }
       fprintf(stdout,"\n");*/


  //}


//(*p)->cmax=h_cmax[0];


//printf("cmax fast=%g\n",h_cmax[0]);


//reduction0computemaxcfast_parallel<<<numBlocks, numThreadsPerBlock,smemSize>>>(*d_p, *d_wmod,  *d_wd, order, dir);
//myreduction0computemaxcfast_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd,*d_wtemp, order, dir);

 //reductiona0computemaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);
 //  computemaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);



    // fastcomputemaxc_parallel<<<numBlocks, numThreadsPerBlock,smemSize>>>(*d_p, *d_wmod,  *d_wd, order, dir);
hipDeviceSynchronize();
//hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);

//printf("cmax slow=%g\n",(*p)->cmax);

//(*p)->cmax=2.0;
hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
//hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);

//printf("cmax on device %.8f\n",(*p)->cmax);
/*(*p)->cmax=0.0;
hipMemcpy(*wd, *d_wd, NDERV*dimp*sizeof(real), hipMemcpyDeviceToHost);
for(int i=0; i<dimp;i++)
{
if(((*wd)[i+(soundspeed*dimp)])>((*p)->cmax))
                    (*p)->cmax=(*wd)[i+(soundspeed*dimp)];
if(((*wd)[i+(cfast*dimp)])>((*p)->cmax))
                    (*p)->cmax=(*wd)[i+(cfast*dimp)];
}*/
/*printf("cmax on cpu %.8f\n",(*p)->cmax);*/
//hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
 /*for(int i=0; i<nit;i++)
{
 reduction0computemaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);
   // computemaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);
     hipDeviceSynchronize();
}*/



//    hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);

  // hipFree(*d_ttemp);
  //checkErrors("copy data from device");


   free(h_cmax);
  hipFree(d_bmax);
  hipFree(d_cmax);


}



int cucomputemaxcourant(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order, int dir, real **wd, real **d_wtemp)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));
    double *d_cmax;
  double *d_bmax;

  real fn,fractn,in;
  int i,ndimp;
////hipSetDevice(selectedDevice);
   int nit=100;
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

    fn=log(dimp)/log(2.0);
    fractn=modf(fn,&in);
    
    if(fractn>0)
    {
       fn+=1;
       ndimp=(int)pow(2,fn);
     }
     else
       ndimp=dimp;
       
       int NTPB=512;
  int numBlocks = (ndimp+NTPB-1) / NTPB;

  int smemSize = NTPB * sizeof(double);
 double *h_cmax = (double*)malloc(numBlocks*sizeof(double));

  hipMalloc((void**)&d_cmax, numBlocks*sizeof(double)); 
  hipMalloc((void**)&d_bmax, numBlocks*sizeof(double)); 


   ((*p)->maxcourant)=0;
   //(*p)->maxcourant=0.0;
   // int smemSize = numThreadsPerBlock * sizeof(real);
  hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
 //dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   //int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;




//hipMemcpy(*d_wtemp, *d_wd, NDERV*dimp*sizeof(real), hipMemcpyDeviceToHost);
//  zeropadmaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir, *d_wtemp,ndimp);
hipMemcpy(*wd, *d_wd, NDERV*dimp*sizeof(real), hipMemcpyDeviceToHost);
hipMemcpy(*d_wtemp, ((*wd)+(cfast*dimp)), dimp*sizeof(real), hipMemcpyHostToDevice);
 zeropadmaxcourant_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir, *d_wtemp,ndimp);


   for(i=0;i<numBlocks;i++)
                h_cmax[i]=0;
  hipMemcpy(d_bmax, h_cmax, numBlocks*sizeof(double), hipMemcpyHostToDevice);


  newreduction0computemax_parallel<<<numBlocks,NTPB,smemSize>>>(d_bmax,*d_wtemp,ndimp);
  hipDeviceSynchronize();
  hipMemcpy(h_cmax, d_bmax, numBlocks*sizeof(double), hipMemcpyDeviceToHost);


  int oldnumBlocks,newnumBlocks;
  newnumBlocks=numBlocks;

  /*while(newnumBlocks>1)
  {
        for(i=0;i<numBlocks;i++)
                h_cmax[i]=0;
        hipMemcpy(d_bmax, h_cmax, numBlocks*sizeof(double), hipMemcpyHostToDevice);
        for(i=0;i<numBlocks;i++)
                h_cmax[i]=0;
        hipMemcpy(d_bmax, h_cmax, numBlocks*sizeof(double), hipMemcpyHostToDevice);


       //hipMemcpy(d_wtemp, h_cmax, numBlocks*sizeof(double), hipMemcpyHostToDevice);
       oldnumBlocks=newnumBlocks;
  	newnumBlocks = (newnumBlocks+NTPB-1) / NTPB;

  	newreduction0computemax_parallel<<<newnumBlocks,NTPB,smemSize>>>(d_bmax,*d_wtemp,oldnumBlocks);
       hipDeviceSynchronize();
       hipMemcpy(h_cmax, d_bmax, newnumBlocks*sizeof(double), hipMemcpyDeviceToHost);*/

  /*for (i=0; i<numBlocks; i++)
    {
      fprintf(stdout,"cmax# %d %f\n",i, h_cmax[i]);
    }
       fprintf(stdout,"\n");*/


  //}
  hipMemcpy(h_cmax, d_bmax, numBlocks*sizeof(double), hipMemcpyDeviceToHost);

  for( i=0;i<numBlocks;i++)          		
                if(h_cmax[i]>((*p)->maxcourant)) ((*p)->maxcourant)=h_cmax[i];







//(*p)->maxcourant=h_cmax[0];
hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
















/*int s=1;
myreduction0computemaxcourant_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir, *d_wtemp,ndimp,s);
hipDeviceSynchronize();
while(((s*=2)<=((ndimp/2)-1)) ) 
{
   myreduction0computemaxcourant_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir, *d_wtemp,ndimp,s);
   hipDeviceSynchronize();
}*/
//reduction0computemaxcfast_parallel<<<numBlocks, numThreadsPerBlock,smemSize>>>(*d_p, *d_wmod,  *d_wd, order, dir);
//myreduction0computemaxcfast_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd,*d_wtemp, order, dir);

 //reductiona0computemaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);
  // computemaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);
    // fastcomputemaxc_parallel<<<numBlocks, numThreadsPerBlock,smemSize>>>(*d_p, *d_wmod,  *d_wd, order, dir);
hipDeviceSynchronize();

//(*p)->cmax=2.0;
//hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);

//printf("cmax on device %.8f\n",(*p)->cmax);
//(*p)->cmax=0.0;
//hipMemcpy(*wd, *d_wd, NDERV*dimp*sizeof(real), hipMemcpyDeviceToHost);
/*for(int i=0; i<dimp;i++)
{

if(((*wd)[i+(cfast*dimp)])>((*p)->cmax))
                    (*p)->cmax=(*wd)[i+(cfast*dimp)];
}
printf("cmax on cpu %.8f\n",(*p)->cmax);*/
//hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
 /*for(int i=0; i<nit;i++)
{
 reduction0computemaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);
   // computemaxc_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);
     hipDeviceSynchronize();
}*/



//    hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);

  // hipFree(*d_ttemp);
  //checkErrors("copy data from device");


    free(h_cmax);
  hipFree(d_bmax);
  hipFree(d_cmax);



}




int cucomputec(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order, int dir)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));
////hipSetDevice(selectedDevice);
   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

 //dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     computec_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);

     hipDeviceSynchronize();
 

    hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //checkErrors("copy data from device");


 


}

int cucomputept(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order, int dir)
{

 int dimp=(((*p)->n[0]))*(((*p)->n[1]));
////hipSetDevice(selectedDevice);
   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

 //dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;

    computeptzero_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);

     hipDeviceSynchronize();
     computept_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);

     hipDeviceSynchronize();
 

   // hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //checkErrors("copy data from device");


 


}

int cucomputepk(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order, int dir)
{

 int dimp=(((*p)->n[0]))*(((*p)->n[1]));
////hipSetDevice(selectedDevice);
   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

 //dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     computepk_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);

     hipDeviceSynchronize();
 

   // hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //checkErrors("copy data from device");


 


}


int cucomputepbg(struct params **p,  struct params **d_p, real **d_wmod,  real **d_wd, int order, int dir)
{

 int dimp=(((*p)->n[0]))*(((*p)->n[1]));
////hipSetDevice(selectedDevice);
   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

 //dim3 dimBlock(dimblock, 1);
    //dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   // dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


     computepbg_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dir);

     hipDeviceSynchronize();
 

   // hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //checkErrors("copy data from device");


 


}







