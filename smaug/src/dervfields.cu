#include "hip/hip_runtime.h"
/*Operators for derived fields*/


//These operators for the derived fields are routines which may be used by the kernel functions
//They are used as follows during make the field MODID is replaced by a unique identifier
//for the particular cuda source file 
//For example the file centdiff1.cu has identifier cd1
//so that computej3_MODID becomes computej3_cd1

//The make routine copies the resulting file to a new file called dervfields_cd1.cuh
//This file is then included using the line #include "../include/dervfields_cd1.cuh"
//in centdiff1.cu

//The routines in centdiff1.cu must call these routines with _MODID replaced by _cd1



__device__ __host__
void computej3_MODID(real *wmod,real *wd,struct params *p,int *ii)
{


 /* wd[fencode3_MODID(p,ii,current1)]=(grad3d_MODID(wmod,p,ii,b3,1))/(p->mu);
  wd[fencode3_MODID(p,ii,current2)]=(grad3d_MODID(wmod,p,ii,b3,0))/(p->mu);
  wd[fencode3_MODID(p,ii,current3)]=(grad3d_MODID(wmod,p,ii,b2,0)-grad3d_MODID(wmod,p,ii,b1,1))/(p->mu);*/
  
          #ifdef USE_SAC
	 /* wd[fencode3_MODID(p,ii,current1)]+=(grad3d_MODID(wmod,p,ii,b3b,1))/(p->mu);
	  wd[fencode3_MODID(p,ii,current2)]+=(grad3d_MODID(wmod,p,ii,b3b,0))/(p->mu);
	  wd[fencode3_MODID(p,ii,current3)]+=(grad3d_MODID(wmod,p,ii,b2b,0)-grad3d_MODID(wmod,p,ii,b1b,1))/(p->mu);*/
         #endif

          #ifdef USE_SAC_3D

         /* wd[fencode3_MODID(p,ii,current1)]-=(  (grad3d_MODID(wmod,p,ii,b2b,2))+ (grad3d_MODID(wmod,p,ii,b2,2)) )/(p->mu)
          wd[fencode3_MODID(p,ii,current2)]+=(  (grad3d_MODID(wmod,p,ii,b1b,2))+ (grad3d_MODID(wmod,p,ii,b1,2)) )/(p->mu)*/

	 /* wd[fencode3_MODID(p,ii,current1)]+=(grad3d_MODID(wmod,p,ii,b3b,1))/(p->mu);
	  wd[fencode3_MODID(p,ii,current2)]+=(grad3d_MODID(wmod,p,ii,b3b,0))/(p->mu);
	  wd[fencode3_MODID(p,ii,current3)]+=(grad3d_MODID(wmod,p,ii,b2b,0)-grad3d_MODID(wmod,p,ii,b1b,1))/(p->mu);*/
         #endif

}

__device__ __host__
void computebdotv3_MODID(real *wmod,real *wd,struct params *p,int *ii)
{
        #ifdef USE_SAC


wd[fencode3_MODID(p,ii,bdotv)]=((wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b1b)])*wmod[fencode3_MODID(p,ii,mom1)]+(wmod[fencode3_MODID(p,ii,b2)]+wmod[fencode3_MODID(p,ii,b2b)])*wmod[fencode3_MODID(p,ii,mom2)])/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]);

         #endif
        #ifdef USE_SAC_3D


wd[fencode3_MODID(p,ii,bdotv)]=((wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b1b)])*wmod[fencode3_MODID(p,ii,mom1)]+(wmod[fencode3_MODID(p,ii,b2)]+wmod[fencode3_MODID(p,ii,b2b)])*wmod[fencode3_MODID(p,ii,mom2)]+(wmod[fencode3_MODID(p,ii,b3)]+wmod[fencode3_MODID(p,ii,b3b)])*wmod[fencode3_MODID(p,ii,mom3)])/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]);

         #endif
 // return ( status);
}

__device__ __host__
void computedivb3_MODID(real *wmod,real *wd,struct params *p,int *ii)
{
      //#ifdef USE_SAC

		wd[fencode3_MODID(p,ii,divb)]=grad3d_MODID(wmod,p,ii,b1,0)+grad3d_MODID(wmod,p,ii,b2,1);
		wd[fencode3_MODID(p,ii,divb)]+=grad3d_MODID(wmod,p,ii,b1b,0)+grad3d_MODID(wmod,p,ii,b2b,1);
        //#endif
        #ifdef USE_SAC_3D
		wd[fencode3_MODID(p,ii,divb)]=grad3d_MODID(wmod,p,ii,b2,2);
		wd[fencode3_MODID(p,ii,divb)]+=grad3d_MODID(wmod,p,ii,b2b,2);		
         #endif
 // return ( status);
}

__device__ __host__
void computevel3_MODID(real *wmod,real *wd,struct params *p,int *ii)
{

        #ifdef USE_SAC_3D
		wd[fencode3_MODID(p,ii,vel1)]=wmod[fencode3_MODID(p,ii,mom1)]/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]);
                wd[fencode3_MODID(p,ii,vel2)]=wmod[fencode3_MODID(p,ii,mom2)]/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]);
                wd[fencode3_MODID(p,ii,vel3)]=wmod[fencode3_MODID(p,ii,mom3)]/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]);
        #endif

        #ifdef USE_SAC
		wd[fencode3_MODID(p,ii,vel1)]=wmod[fencode3_MODID(p,ii,mom1)]/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]);
                wd[fencode3_MODID(p,ii,vel2)]=wmod[fencode3_MODID(p,ii,mom2)]/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]);
        #endif
       #ifdef ADIABHYDRO
		wd[fencode3_MODID(p,ii,vel1)]=wmod[fencode3_MODID(p,ii,mom1)]/(wmod[fencode3_MODID(p,ii,rho)]);
                wd[fencode3_MODID(p,ii,vel2)]=wmod[fencode3_MODID(p,ii,mom2)]/(wmod[fencode3_MODID(p,ii,rho)]);

         #endif
 // return ( status);
}







__device__ __host__
void computept3_MODID(real *wmod,real *wd,struct params *p,int *ii)
{
 // int status=0;

#ifdef ADIABHYDRO

/*below used for adiabatic hydrodynamics*/
 wd[fencode3_MODID(p,ii,pressuret)]=(p->adiab)*pow(wmod[fencode3_MODID(p,ii,rho)],p->gamma);

#elif defined(USE_SAC)
 //wmod[fencode3_MODID(p,ii,b1b)]=0;
// wmod[fencode3_MODID(p,ii,b2b)]=0;
wd[fencode3_MODID(p,ii,pressuret)]=((p->gamma)-2.0)*((wmod[fencode3_MODID(p,ii,b1)]*wmod[fencode3_MODID(p,ii,b1b)]+wmod[fencode3_MODID(p,ii,b2)]*wmod[fencode3_MODID(p,ii,b2b)])+0.5*(wmod[fencode3_MODID(p,ii,b1)]*wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b2)]*wmod[fencode3_MODID(p,ii,b2)]));
wd[fencode3_MODID(p,ii,pressuret)]=((p->gamma)-1.0)*( wmod[fencode3_MODID(p,ii,energy)]-0.5*(wmod[fencode3_MODID(p,ii,mom1)]*wmod[fencode3_MODID(p,ii,mom1)]+wmod[fencode3_MODID(p,ii,mom2)]*wmod[fencode3_MODID(p,ii,mom2)])/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]))-wd[fencode3_MODID(p,ii,pressuret)];



#elif defined(USE_SAC_3D)

wd[fencode3_MODID(p,ii,pressuret)]=((p->gamma)-2.0)*((wmod[fencode3_MODID(p,ii,b1)]*wmod[fencode3_MODID(p,ii,b1b)]+wmod[fencode3_MODID(p,ii,b2)]*wmod[fencode3_MODID(p,ii,b2b)]+wmod[fencode3_MODID(p,ii,b3)]*wmod[fencode3_MODID(p,ii,b3b)])+0.5*(wmod[fencode3_MODID(p,ii,b1)]*wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b2)]*wmod[fencode3_MODID(p,ii,b2)]+wmod[fencode3_MODID(p,ii,b3)]*wmod[fencode3_MODID(p,ii,b3)]));




//wd[fencode3_MODID(p,ii,pressuret)]=0.0;
//wd[fencode3_MODID(p,ii,pressuret)]=((p->gamma)-1.0)*( wmod[fencode3_MODID(p,ii,energy)]);
//wd[fencode3_MODID(p,ii,pressuret)]=((p->gamma)-1.0)*( wmod[fencode3_MODID(p,ii,energy)]-0.5*(wmod[fencode3_MODID(p,ii,mom1)]*wmod[fencode3_MODID(p,ii,mom1)]+wmod[fencode3_MODID(p,ii,mom2)]*wmod[fencode3_MODID(p,ii,mom2)]+wmod[fencode3_MODID(p,ii,mom3)]*wmod[fencode3_MODID(p,ii,mom3)])/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]))-wd[fencode3_MODID(p,ii,pressuret)];



wd[fencode3_MODID(p,ii,pressuret)]=((p->gamma)-1.0)*(wmod[fencode3_MODID(p,ii,energy)] -0.5*(wmod[fencode3_MODID(p,ii,mom1)]*wmod[fencode3_MODID(p,ii,mom1)]+wmod[fencode3_MODID(p,ii,mom2)]*wmod[fencode3_MODID(p,ii,mom2)]+wmod[fencode3_MODID(p,ii,mom3)]*wmod[fencode3_MODID(p,ii,mom3)])/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]))-wd[fencode3_MODID(p,ii,pressuret)];

#else

wd[fencode3_MODID(p,ii,pressuret)]=  ((p->gamma)-1.0)*wmod[fencode3_MODID(p,ii,energy)]+(1.0-0.5*(p->gamma))*(wmod[fencode3_MODID(p,ii,b1)]*wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b2)]*wmod[fencode3_MODID(p,ii,b2)])+0.5*(1.0-(p->gamma))*(wmod[fencode3_MODID(p,ii,mom1)]*wmod[fencode3_MODID(p,ii,mom1)]+wmod[fencode3_MODID(p,ii,mom2)]*wmod[fencode3_MODID(p,ii,mom2)])/wmod[fencode3_MODID(p,ii,rho)];

#endif



  //if(wd[fencode3_MODID(p,ii,pressuret)]<0)
              //wd[fencode3_MODID(p,ii,pressuret)]=1.0e-10;
	//      wd[fencode3_MODID(p,ii,pressuret)]=0.01;


 // return ( status);
}


__device__ __host__
void computepbg3_MODID(real *wmod,real *wd,struct params *p,int *ii)
{
 // int status=0;

#ifdef ADIABHYDRO

/*below used for adiabatic hydrodynamics*/
;// wd[fencode3_MODID(p,ii,pressuret)]=(p->adiab)*pow(wmod[fencode3_MODID(p,ii,rho)],p->gamma);

#elif defined(USE_SAC)
 //wmod[fencode3_MODID(p,ii,b1b)]=0;
// wmod[fencode3_MODID(p,ii,b2b)]=0;



 wd[fencode3_MODID(p,ii,ptb)]=  ((p->gamma)-1)*wmod[fencode3_MODID(p,ii,energyb)]- 0.5*((p->gamma)-2)*(wmod[fencode3_MODID(p,ii,b1b)]*wmod[fencode3_MODID(p,ii,b1b)]+wmod[fencode3_MODID(p,ii,b2b)]*wmod[fencode3_MODID(p,ii,b2b)]) ;

#elif defined(USE_SAC_3D)



 wd[fencode3_MODID(p,ii,ptb)]=  ((p->gamma)-1)*wmod[fencode3_MODID(p,ii,energyb)]- 0.5*((p->gamma)-2)*(wmod[fencode3_MODID(p,ii,b1b)]*wmod[fencode3_MODID(p,ii,b1b)]+wmod[fencode3_MODID(p,ii,b2b)]*wmod[fencode3_MODID(p,ii,b2b)]+wmod[fencode3_MODID(p,ii,b3b)]*wmod[fencode3_MODID(p,ii,b3b)]) ;


#endif



  //if(wd[fencode3_MODID(p,ii,pressuret)]<0)
              //wd[fencode3_MODID(p,ii,pressuret)]=1.0e-10;
	//      wd[fencode3_MODID(p,ii,pressuret)]=0.01;


 // return ( status);
}





__device__ __host__
void computepk3_MODID(real *wmod,real *wd,struct params *p,int *ii)
{
  //int status=0;

#ifdef ADIABHYDRO

/*below used for adiabatic hydrodynamics*/
wd[fencode3_MODID(p,ii,pressurek)]=(p->adiab)*pow(wmod[fencode3_MODID(p,ii,rho)],p->gamma);
wd[fencode3_MODID(p,ii,vel1)]=wmod[fencode3_MODID(p,ii,mom1)]/(wmod[fencode3_MODID(p,ii,rho)]);
wd[fencode3_MODID(p,ii,vel2)]=wmod[fencode3_MODID(p,ii,mom2)]/(wmod[fencode3_MODID(p,ii,rho)]);

#elif defined(USE_SAC)
 wd[fencode3_MODID(p,ii,pressurek)]= 0.5*((wmod[fencode3_MODID(p,ii,mom1)]*wmod[fencode3_MODID(p,ii,mom1)]+wmod[fencode3_MODID(p,ii,mom2)]*wmod[fencode3_MODID(p,ii,mom2)])/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]));

wd[fencode3_MODID(p,ii,pressurek)]=wd[fencode3_MODID(p,ii,pressurek)]+(0.5*(wmod[fencode3_MODID(p,ii,b1)]*wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b2)]*wmod[fencode3_MODID(p,ii,b2)]) +(wmod[fencode3_MODID(p,ii,b1)]*wmod[fencode3_MODID(p,ii,b1b)]+wmod[fencode3_MODID(p,ii,b2)]*wmod[fencode3_MODID(p,ii,b2b)]) );

wd[fencode3_MODID(p,ii,pressurek)]=((p->gamma)-1)*wmod[fencode3_MODID(p,ii,energy)]-wd[fencode3_MODID(p,ii,pressurek)];


#elif defined(USE_SAC_3D)

 wd[fencode3_MODID(p,ii,pressurek)]= 0.5*((wmod[fencode3_MODID(p,ii,mom1)]*wmod[fencode3_MODID(p,ii,mom1)]+wmod[fencode3_MODID(p,ii,mom2)]*wmod[fencode3_MODID(p,ii,mom2)]+wmod[fencode3_MODID(p,ii,mom3)]*wmod[fencode3_MODID(p,ii,mom3)])/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]));

wd[fencode3_MODID(p,ii,pressurek)]=wd[fencode3_MODID(p,ii,pressurek)]+(0.5*(wmod[fencode3_MODID(p,ii,b1)]*wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b2)]*wmod[fencode3_MODID(p,ii,b2)]+wmod[fencode3_MODID(p,ii,b3)]*wmod[fencode3_MODID(p,ii,b3)]) +(wmod[fencode3_MODID(p,ii,b1)]*wmod[fencode3_MODID(p,ii,b1b)]+wmod[fencode3_MODID(p,ii,b2)]*wmod[fencode3_MODID(p,ii,b2b)]+wmod[fencode3_MODID(p,ii,b3)]*wmod[fencode3_MODID(p,ii,b3b)]) );

wd[fencode3_MODID(p,ii,pressurek)]=((p->gamma)-1)*wmod[fencode3_MODID(p,ii,energy)]-wd[fencode3_MODID(p,ii,pressurek)];


#else

 wd[fencode3_MODID(p,ii,pressurek)]=((p->gamma)-1)*(wmod[fencode3_MODID(p,ii,energy)]- 0.5*(wmod[fencode3_MODID(p,ii,mom1)]*wmod[fencode3_MODID(p,ii,mom1)]+wmod[fencode3_MODID(p,ii,mom2)]*wmod[fencode3_MODID(p,ii,mom2)])/wmod[fencode3_MODID(p,ii,rho)]-0.5*(wmod[fencode3_MODID(p,ii,b1)]*wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b2)]*wmod[fencode3_MODID(p,ii,b2)]) );


#endif



  //if(wd[fencode3_MODID(p,ii,pressurek)]<0)
  //           wd[fencode3_MODID(p,ii,pressurek)]=0.001;
  //return ( status);
}

__device__ __host__
void computec3_MODID(real *wmod,real *wd,struct params *p,int *ii,int dir)
{

 real cfasti,pk; 
#ifdef ADIABHYDRO
/*below used for adiabatic hydrodynamics*/
  wd[fencode3_MODID(p,ii,soundspeed)]=sqrt((p->adiab)/wmod[fencode3_MODID(p,ii,rho)]);
#elif defined(USE_SAC)

pk=((p->gamma)-1)*(wmod[fencode3_MODID(p,ii,energy)]
- 0.5*((wmod[fencode3_MODID(p,ii,mom1)]*wmod[fencode3_MODID(p,ii,mom1)]+wmod[fencode3_MODID(p,ii,mom2)]*wmod[fencode3_MODID(p,ii,mom2)])/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]))-0.5*(wmod[fencode3_MODID(p,ii,b1)]*wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b2)]*wmod[fencode3_MODID(p,ii,b2)]) -(wmod[fencode3_MODID(p,ii,b1b)]*wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b2b)]*wmod[fencode3_MODID(p,ii,b2)]) );

wd[fencode3_MODID(p,ii,soundspeed)]=(((p->gamma))
*(pk+(((p->gamma))-1)*(
wmod[fencode3_MODID(p,ii,energyb)] -0.5*(wmod[fencode3_MODID(p,ii,b1b)]*wmod[fencode3_MODID(p,ii,b1b)]+wmod[fencode3_MODID(p,ii,b2b)]*wmod[fencode3_MODID(p,ii,b2b)])))
/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]));


wd[fencode3_MODID(p,ii,cfast)]=( ((wmod[fencode3_MODID(p,ii,b1)]*wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b2)]*wmod[fencode3_MODID(p,ii,b2)]) + (wmod[fencode3_MODID(p,ii,b1b)]*wmod[fencode3_MODID(p,ii,b1b)]+wmod[fencode3_MODID(p,ii,b2b)]*wmod[fencode3_MODID(p,ii,b2b)]) +2.0*(wmod[fencode3_MODID(p,ii,b1b)]*wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b2b)]*wmod[fencode3_MODID(p,ii,b2)]))/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]))+(wd[fencode3_MODID(p,ii,soundspeed)]);

cfasti=0.5*(
wd[fencode3_MODID(p,ii,cfast)]
+sqrt(wd[fencode3_MODID(p,ii,cfast)]*wd[fencode3_MODID(p,ii,cfast)]
-4.0*wd[fencode3_MODID(p,ii,soundspeed)]*((wmod[fencode3_MODID(p,ii,b1b+dir)]+wmod[fencode3_MODID(p,ii,b1+dir)])*(wmod[fencode3_MODID(p,ii,b1b+dir)]+wmod[fencode3_MODID(p,ii,b1+dir)]))
/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)])));

wd[fencode3_MODID(p,ii,cfast)]=sqrt(cfasti)+sacdabs_MODID(wmod[fencode3_MODID(p,ii,mom1+dir)]/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]));
wd[fencode3_MODID(p,ii,soundspeed)]=sqrt(wd[fencode3_MODID(p,ii,soundspeed)]);
//wd[fencode3_MODID(p,ii,cfast)]=( ((wmod[fencode3_MODID(p,ii,b1)]*wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b2)]*wmod[fencode3_MODID(p,ii,b2)]) + (wmod[fencode3_MODID(p,ii,b1b)]*wmod[fencode3_MODID(p,ii,b1b)]+wmod[fencode3_MODID(p,ii,b2b)]*wmod[fencode3_MODID(p,ii,b2b)]) +2.0*(wmod[fencode3_MODID(p,ii,b1b)]*wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b2b)]*wmod[fencode3_MODID(p,ii,b2)]))/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]));

//wd[fencode3_MODID(p,ii,cfast)]=cfasti;

#elif defined(USE_SAC_3D)


pk=((p->gamma)-1)*(wmod[fencode3_MODID(p,ii,energy)]
- 0.5*((wmod[fencode3_MODID(p,ii,mom1)]*wmod[fencode3_MODID(p,ii,mom1)]+wmod[fencode3_MODID(p,ii,mom2)]*wmod[fencode3_MODID(p,ii,mom2)]+wmod[fencode3_MODID(p,ii,mom3)]*wmod[fencode3_MODID(p,ii,mom3)])/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]))-0.5*(wmod[fencode3_MODID(p,ii,b1)]*wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b2)]*wmod[fencode3_MODID(p,ii,b2)]+wmod[fencode3_MODID(p,ii,b3)]*wmod[fencode3_MODID(p,ii,b3)]) -(wmod[fencode3_MODID(p,ii,b1b)]*wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b2b)]*wmod[fencode3_MODID(p,ii,b2)]+wmod[fencode3_MODID(p,ii,b3b)]*wmod[fencode3_MODID(p,ii,b3)]) );

wd[fencode3_MODID(p,ii,soundspeed)]=(((p->gamma))
*(pk+(((p->gamma))-1)*(
wmod[fencode3_MODID(p,ii,energyb)] -0.5*(wmod[fencode3_MODID(p,ii,b1b)]*wmod[fencode3_MODID(p,ii,b1b)]+wmod[fencode3_MODID(p,ii,b2b)]*wmod[fencode3_MODID(p,ii,b2b)]+wmod[fencode3_MODID(p,ii,b3b)]*wmod[fencode3_MODID(p,ii,b3b)])))
/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]));


wd[fencode3_MODID(p,ii,cfast)]=( ((wmod[fencode3_MODID(p,ii,b1)]*wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b2)]*wmod[fencode3_MODID(p,ii,b2)]+wmod[fencode3_MODID(p,ii,b3)]*wmod[fencode3_MODID(p,ii,b3)]) + (wmod[fencode3_MODID(p,ii,b1b)]*wmod[fencode3_MODID(p,ii,b1b)]+wmod[fencode3_MODID(p,ii,b2b)]*wmod[fencode3_MODID(p,ii,b2b)]+wmod[fencode3_MODID(p,ii,b3b)]*wmod[fencode3_MODID(p,ii,b3b)]) +2.0*(wmod[fencode3_MODID(p,ii,b1b)]*wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b2b)]*wmod[fencode3_MODID(p,ii,b2)]+wmod[fencode3_MODID(p,ii,b3b)]*wmod[fencode3_MODID(p,ii,b3)]))/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]))+(wd[fencode3_MODID(p,ii,soundspeed)]);

cfasti=0.5*(
wd[fencode3_MODID(p,ii,cfast)]
+sqrt(wd[fencode3_MODID(p,ii,cfast)]*wd[fencode3_MODID(p,ii,cfast)]
-4.0*wd[fencode3_MODID(p,ii,soundspeed)]*((wmod[fencode3_MODID(p,ii,b1b+dir)]+wmod[fencode3_MODID(p,ii,b1+dir)])*(wmod[fencode3_MODID(p,ii,b1b+dir)]+wmod[fencode3_MODID(p,ii,b1+dir)]))
/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)])));

wd[fencode3_MODID(p,ii,cfast)]=sqrt(cfasti)+sacdabs_MODID(wmod[fencode3_MODID(p,ii,mom1+dir)]/(wmod[fencode3_MODID(p,ii,rho)]+wmod[fencode3_MODID(p,ii,rhob)]));
wd[fencode3_MODID(p,ii,soundspeed)]=sqrt(wd[fencode3_MODID(p,ii,soundspeed)]);



#else
wd[fencode3_MODID(p,ii,soundspeed)]=sqrt(((p->gamma))*wd[fencode3_MODID(p,ii,pressuret)]/wmod[fencode3_MODID(p,ii,rho)]);


wd[fencode3_MODID(p,ii,cfast)]=sqrt(((wmod[fencode3_MODID(p,ii,b1)]*wmod[fencode3_MODID(p,ii,b1)]+wmod[fencode3_MODID(p,ii,b2)]*wmod[fencode3_MODID(p,ii,b2)])/wmod[fencode3_MODID(p,ii,rho)])+(wd[fencode3_MODID(p,ii,soundspeed)]*wd[fencode3_MODID(p,ii,soundspeed)]));

#endif



  
}
//uptohere so far thursday  24th march
__device__ __host__
void computecmax3_MODID(real *wmod,real *wd,struct params *p,int *ii)
{
 //p->cmax=0.02;
#ifdef ADIABHYDRO
       if(wd[fencode3_MODID(p,ii,soundspeed)]>(p->cmax))
                    // atomicExch(&(p->cmax),(wd[fencode3_MODID(p,ii,soundspeed)]));
                    p->cmax=(wd[fencode3_MODID(p,ii,soundspeed)]);
#else
       if(wd[fencode3_MODID(p,ii,soundspeed)]>(p->cmax))
                    // atomicExch(&(p->cmax),(wd[fencode3_MODID(p,ii,soundspeed)]));
                    p->cmax=(wd[fencode3_MODID(p,ii,soundspeed)]);
       if(wd[fencode3_MODID(p,ii,cfast)]>(p->cmax))
                     //atomicExch(&(p->cmax),(wd[fencode3_MODID(p,ii,cfast)]));
                    p->cmax=(wd[fencode3_MODID(p,ii,cfast)]);
#endif

}


