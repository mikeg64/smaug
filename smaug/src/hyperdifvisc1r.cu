#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"
#include "../include/iobparams.h"
/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/smaugcukernels.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_hdv1r.cuh"

__device__ __host__
void bc_hyperdifr(real *wt, struct params *p,int *ii, int f,int dir) {

   int i=ii[0];
   int j=ii[1];
   int k=0;
 #ifdef USE_SAC_3D
	k=ii[2];
 #endif

int is=1;
 #ifdef USE_SAC
   if(  (dir == 0) && (i==(p->n[0])-1)   && j>=0   && j<(p->n[1])           )
   {
      //for(int is=0;is<3-2*(j<((p->n[1])-1));is++)   
         wt[encode3p2_hdv1r(p,i+2,j+is,k,f)]=wt[encode3p2_hdv1r(p,(p->n[0])-5,j+is,k,f)];
         
   }
   else if((dir == 1) && (j==(p->n[1])-1)    && i>0   && i<((p->n[0]))  )
    //for(int is=0;is<3-2*(j<((p->n[1])-1));is++)
       wt[encode3p2_hdv1r(p,i+is,j+2,k,f)]=wt[encode3p2_hdv1r(p,i+is,(p->n[1])-5,k,f)];
  else if((dir == 0) && (i==0)    && j>0   && j<((p->n[1]))   )
    //for(int is=0;is<3-2*(j<((p->n[1])-1));is++)
       wt[encode3p2_hdv1r(p,0,j+is,k,f)]=wt[encode3p2_hdv1r(p,6,j+is,k,f)];
   else if((dir == 1) && (j==0)    && i>0   && i<((p->n[0]))   )
    //for(int is=0;is<3-2*(j<((p->n[1])-1));is++)
       wt[encode3p2_hdv1r(p,i+is,0,k,f)]=wt[encode3p2_hdv1r(p,i+is,6,k,f)];

#endif
 #ifdef USE_SAC_3D
   if(  (dir == 0) && (i==(p->n[0])-1)   && j>0   && j<(p->n[1])      && k>0   && k<(p->n[2])     )
         wt[encode3p2_hdv1r(p,i+2,j+is,k+is,f)]=wt[encode3p2_hdv1r(p,(p->n[0])-5,j+is,k+is,f)];
   else if((dir == 1) && (j==(p->n[1])-1)    && i>0   && i<((p->n[0])) && k>0   && k<((p->n[2]))  )
       wt[encode3p2_hdv1r(p,i+is,j+2,k+is,f)]=wt[encode3p2_hdv1r(p,i+is,(p->n[1])-5,k+is,f)];
   else if((dir == 2) && (k==(p->n[2])-1)    && i>0   && i<((p->n[0])) && j>0   && j<((p->n[1]))  )
       wt[encode3p2_hdv1r(p,i+is,j+is,k+2,f)]=wt[encode3p2_hdv1r(p,i+is,j+is,(p->n[2])-5,f)];
  else if((dir == 0) && (i==0)    && j>0   && j<((p->n[1])) && k>0   && k<((p->n[2]))  )
       wt[encode3p2_hdv1r(p,0,j+is,k+is,f)]=wt[encode3p2_hdv1r(p,6,j+is,k+is,f)];
   else if((dir == 1) && (j==0)    && i>0   && i<((p->n[0]))  && k>0   && k<((p->n[2]))  )
       wt[encode3p2_hdv1r(p,i+is,0,k+is,f)]=wt[encode3p2_hdv1r(p,i+is,6,k+is,f)];
   else if((dir == 2) && (k==0)    && i>0   && i<((p->n[0])) && j>0   && j<((p->n[1]))   )
       wt[encode3p2_hdv1r(p,i+is,j+is,0,f)]=wt[encode3p2_hdv1r(p,i+is,j+is,6,f)];
#endif




 
}


/*__device__ __host__
void bc_periodic1_temp2(real *wt, struct params *p,int i, int j, int f) {

                if(i==1 )                
                    wt[fencode_hdv1r(p,i,j,f)]=wt[fencode_hdv1r(p,6,j,f)];
                else if((i==((p->n[0]))) )                
                    wt[fencode_hdv1r(p,i,j,f)]=wt[fencode_hdv1r(p,i-4,j,f)];
                else if(j==1  )                
                  wt[fencode_hdv1r(p,i,j,f)]=wt[fencode_hdv1r(p,i,6,f)];
                else if((j==((p->n[1]))) )                
                  wt[fencode_hdv1r(p,i,j,f)]=wt[fencode_hdv1r(p,i,j-4,f)];
}*/

/*__device__ __host__
void bc_periodic2_temp2(real *wt, struct params *p,int i, int j, int f) {


               if(i<1 && j<1)
                {
                  if(i==j)
                    //wt[fencode_hdv1r(p,i,j,f)]=wt[fencode_hdv1r(p,(p->n[0])-3+i,j,f)];
                    wt[fencode_hdv1r(p,i,j,f)]=wt[fencode_hdv1r(p,i,6,f)];
                  else                  
                    //wt[fencode_hdv1r(p,i,j,f)]=wt[fencode_hdv1r(p,i,(p->n[1])-3+j,f)];
                    wt[fencode_hdv1r(p,i,j,f)]=wt[fencode_hdv1r(p,6,j,f)];                                    
                }
                else if(i<1 && j>((p->n[1])-1))
                {
                  if(i==(j-(p->n[1])-1))                  
                    //wt[fencode_hdv1r(p,i,j,f)]=wt[fencode_hdv1r(p,(p->n[0])-3+i,4-(p->n[1])+j,f)];
                    wt[fencode_hdv1r(p,i,j,f)]=wt[fencode_hdv1r(p,6,j,f)];                                     
                  else                  
                    wt[fencode_hdv1r(p,i,j,f)]=wt[fencode_hdv1r(p,i,j-6,f)];                                     
                }
                else if(i>((p->n[0])-1) && j<1)
                {
                  if((i-(p->n[0])+1)==j)                  
                    wt[fencode_hdv1r(p,i,j,f)]=wt[fencode_hdv1r(p,i-5,j,f)];                                    
                  else                  
                   wt[fencode_hdv1r(p,i,j,f)]=wt[fencode_hdv1r(p,i,4,f)];                                    
                }
                else if(i>((p->n[0])-1) && j>((p->n[1])-1))
                {
                  if(i==j)                  
                    wt[fencode_hdv1r(p,i,j,f)]=wt[fencode_hdv1r(p,i,j-5,f)];                                    
                  else                  
                    wt[fencode_hdv1r(p,i,j,f)]=wt[fencode_hdv1r(p,i-5,j,f)];                                    
                }                       
                 
                




}*/



__global__ void zeropadmaxviscr_parallel(struct params *p,   real *wmod, real *wd, int order, int dir, real *temp, int ndimp)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
 
  if(iindex<ndimp)
      temp[iindex]=0.0;

}

__global__ void newreduction0computemaxviscr_parallel(real *cmax, real *temp,int ndimp)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  extern __shared__ double partialResult[];

  int i;
   partialResult[tid]=0.0;
   if(iindex<ndimp)
              partialResult[tid]=temp[iindex];
  __syncthreads();


for(unsigned int s=1; s < blockDim.x; s *= 2) {
        if ((tid % (2*s)) == 0) {
            if(partialResult[tid+s]>partialResult[tid])
                 partialResult[tid]=partialResult[tid + s];
        }
        __syncthreads();
    }

    __syncthreads();
    if(tid==0)
    {
      cmax[blockIdx.x]=partialResult[0];
      //temp[blockIdx.x]=partialResult[0];
     }
    __syncthreads();

}

__global__ void myreduction0computemaxviscr_parallel(struct params *p,   real *wmod, real *wd, int order, int dir, real *temp,int ndimp,int s)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int tid = threadIdx.x;
  int i,j;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;
//        extern __shared__ real sdata[];
 
  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif

int tnumThreadsPerBlock = 128;
    
int numBlocks = (dimp+tnumThreadsPerBlock-1) / tnumThreadsPerBlock;
  //real temp[dimp];
    // perform first level of reduction,
    // reading from global memory, writing to shared memory
   //sdata[tid]=0.0;
   // if(iindex<1024)
    //  temp[iindex]=0.0;

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif
    //int s=1;


   
    //while(((s*=2)<=((ndimp/2)-1)) && ((iindex+s)<ndimp)) {
    if((iindex+s)<ndimp)
            if(temp[iindex+s]>temp[iindex])
                 temp[iindex]=temp[iindex + s];
            
       // }

       //  __syncthreads();
    

   // __syncthreads();

   if(iindex==0)
      p->maxviscoef=temp[0];


 
}




__global__ void hyperdifvisc5r_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  real maxt=0,max3=0, max1=0;
  
   int ip,jp;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;

   
if(iindex==0)
{
  p->hdmean=0.0;
  p->hdmax=0;
 //  for(ipg=0;ipg<(p->npgp[0]);ipg++)
 //  for(jpg=0;jpg<(p->npgp[1]);jpg++)
  // {

  //   i=ip*(p->npgp[0])+ipg;
 //    j=jp*(p->npgp[1])+jpg;
   //if( i<((p->n[0])) && j<((p->n[1])))
  //if(i>1 && j >1 && i<((p->n[0])-2) && j<((p->n[1])-2))
    //p->cmax=0.0;
    for(ii[0]=1;ii[0]<((p->n[0])-1);ii[0]++)
      for(ii[1]=1;ii[1]<((p->n[1])-1);ii[1]++)
     #ifdef USE_SAC_3D
        for(ii[2]=1;ii[2]<((p->n[2])-1);ii[2]++)
     #endif
	{ 
              // computecmax3_cdf(wmod+(order*dimp*NVAR),wd,p,ii);

             
                    // atomicExch(&(p->cmax),(wd[fencode3_MODID(p,ii,soundspeed)]));
               #ifdef USE_SAC_3D
                if(wd[encode3_hdv1r(p,ii[0],ii[1],ii[2],hdnur)]>(p->maxviscoef))
                    p->maxviscoef=(wd[encode3_hdv1r(p,ii[0],ii[1],ii[2],hdnur)]);
               #else
                 if(wd[encode3_hdv1r(p,ii[0],ii[1],0,hdnur)]>(p->maxviscoef))
                    p->maxviscoef=(wd[encode3_hdv1r(p,ii[0],ii[1],0,hdnur)]);
               #endif

             /* if(wd[encode3_hdv1r(p,ii[0],ii[1],0,hdnur)]>(p->hdmax))
                    p->hdmax=(wd[encode3_hdv1r(p,ii[0],ii[1],0,hdnur)]);

              p->hdmean=(p->hdmean)+wd[encode3_hdv1r(p,ii[0],ii[1],0,hdnur)];*/

	}
//p->hdmean=(p->hdmean)/(((p->n[0])-2)*((p->n[1]))-2);
 //  }
}
 //__syncthreads();



 
}







__global__ void hyperdifvisc4r_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  real maxt=0,max3=0, max1=0;
  
   int ip,jp;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;


   //tmp1  tmp_nuI
   //tmp2  d3r
    //tmp3 d1r
//tmp4    md3r
//tmp5    md1r
//tmp6    d3l
//tmp7    d1l
//tmp8    md3l
//tmp9    md1l







 //  p->maxviscoef=0;
//  p->cmax=1.0;

    //finally update nur and nul
//tmp4    md3r
//tmp5    md1r
   

     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif
      
     #ifdef USE_SAC_3D
       if(i>1 && i<((p->n[0])-1) && j>1 && j<((p->n[1])-1) && k>1 && k<((p->n[2])-1))
     #else
       if(i>1 && i<((p->n[0])-1) && j>1 && j<((p->n[1])-1))
     #endif
   //if(i>1 && i<((p->n[0])-2) && j>1 && j<((p->n[1])-2))
   {
     //wd[encode3_hdv1r(p,i,j,hdnur)]=wtemp2[encode3_hdv1r(p,i+1,j+1,tmpnui)];
     if(wtemp[encode3_hdv1r(p,i,j,k,tmp5)]>0)
{
//p->cmax=1.0;
     #ifdef USE_SAC_3D
	wd[encode3_hdv1r(p,i,j,k,hdnur)]=((dim==0)*(wd[encode3_hdv1r(p,i,j,k,delx1)])+(dim==1)*(wd[encode3_hdv1r(p,i,j,k,delx2)])+(dim==2)*(wd[encode3_hdv1r(p,i,j,k,delx3)]))*(p->cmax)*(p->chyp[field])*wtemp[encode3_hdv1r(p,i,j,k,tmp4)]/wtemp[encode3_hdv1r(p,i,j,k,tmp5)];
     #else
	wd[encode3_hdv1r(p,i,j,k,hdnur)]=((dim==0)*(wd[encode3_hdv1r(p,i,j,k,delx1)])+(dim==1)*(wd[encode3_hdv1r(p,i,j,k,delx2)]))*(p->cmax)*(p->chyp[field])*wtemp[encode3_hdv1r(p,i,j,k,tmp4)]/wtemp[encode3_hdv1r(p,i,j,k,tmp5)];
     #endif
        //wd[encode3_hdv1r(p,i,j,k,hdnur)]=1.0e-1; 
          //wd[encode3_hdv1r(p,i,j,hdnur)]=wtemp[encode3_hdv1r(p,i,j,tmp4)];
	//wd[encode3_hdv1r(p,i,j,k,hdnur)]=0.01;
       // wd[encode3_hdv1r(p,i,j,k,hdnur)]=0.0005; 
}
     else
        wd[encode3_hdv1r(p,i,j,k,hdnur)]=0;


     /*switch(field)
        {
            case 0:
             wd[encode3_hdv1r(p,i,j,k,hdnur)]=6.744e-6;
            break;
            case 3:
             wd[encode3_hdv1r(p,i,j,k,hdnur)]=1.8e-6;
            break;
            case 1:
             wd[encode3_hdv1r(p,i,j,k,hdnur)]=1.9e-6;
            break;
            case 2:
             wd[encode3_hdv1r(p,i,j,k,hdnur)]=1.9e-6;
            break;
            case 5:
             wd[encode3_hdv1r(p,i,j,k,hdnur)]=9.4e-8;
            break;
            case 4:
             wd[encode3_hdv1r(p,i,j,k,hdnur)]=3.8e-7;
            break;
          

         }   */


   }

 //__syncthreads();



 
}





__global__ void hyperdifvisc3r_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int is,js,ks;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  real maxt1=0,max3=0, maxt2=0;
  
   int ip,jp;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;


   //tmp1  tmp_nuI
   //tmp2  d3r
    //tmp3 d1r
//tmp4    md3r
//tmp5    md1r
//tmp6    d3l
//tmp7    d1l
//tmp8    md3l
//tmp9    md1l





  //compute md3r and md1r
//tmp4    md3r
//tmp5    md1r
  //js=0;
 // is=0;

     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       //if(ii[0]>1 && ii[1]>1 && ii[2]>1 && ii[0]<p->n[0] && ii[1]<p->n[1]  && ii[2]<p->n[2])
       if(i>1 && j>1 && k>1 && i<((p->n[0])-2) && j<((p->n[1])-2)   && k<((p->n[2]))-2)
     #else
       //if(ii[0]>1 && ii[1]>1 && ii[0]<p->n[0] && ii[1]<p->n[1])
       if(i>1 && j>1 && i<((p->n[0])-2) && j<((p->n[1])-2))
     #endif

 // if( i>1 && j>1 && i<((p->n[0])-2) && j<((p->n[1])-2))            
   {
         maxt1=0;

     #ifdef USE_SAC_3D
         for(is=-(dim==0); is<=(dim==0); is++)
                for(js=-(dim==1); js<=(dim==1); js++)
                   for(ks=-(dim==2); ks<=(dim==2); ks++)
                {
                   if(wtemp1[encode3p1_hdv1r(p,i+1+is,j+1+js,k+1+ks,d3)]>maxt1)
                         maxt1=wtemp1[encode3p1_hdv1r(p,i+1+is,j+1+js,k+1+ks,d3)];

                }
	#else
         for(is=-(dim==0); is<=(dim==0); is++)
                for(js=-(dim==1); js<=(dim==1); js++)
                {
                   if(wtemp1[encode3p1_hdv1r(p,i+1+is,j+1+js,k,d3)]>maxt1)
                         maxt1=wtemp1[encode3p1_hdv1r(p,i+1+is,j+1+js,k,d3)];

                }
	#endif
          wtemp[encode3_hdv1r(p,i,j,k,tmp4)]=maxt1;

         maxt2=0;

     #ifdef USE_SAC_3D
         for(is=-2*(dim==0); is<=2*(dim==0); is++)
                for(js=-2*(dim==1); js<=2*(dim==1); js++)
                   for(ks=-2*(dim==2); ks<=2*(dim==2); ks++)
                {
                   if(wtemp1[encode3p1_hdv1r(p,i+1+is,j+1+js,k+1+ks,d1)]>maxt2)
                        maxt2=wtemp1[encode3p1_hdv1r(p,i+1+is,j+1+js,k+1+ks,d1)];

                }
	#else
         for(is=-2*(dim==0); is<=2*(dim==0); is++)
                for(js=-2*(dim==1); js<=2*(dim==1); js++)
                {
                   if(wtemp1[encode3p1_hdv1r(p,i+1+is,j+1+js,k,d1)]>maxt2)
                        maxt2=wtemp1[encode3p1_hdv1r(p,i+1+is,j+1+js,k,d1)];

                }
	#endif
          wtemp[encode3_hdv1r(p,i,j,k,tmp5)]=maxt2;
   }

   //__syncthreads();







 
}




__global__ void hyperdifvisc2r_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim)
{
  // compute the global index in the vector from
  // the number of the current block, blockIdx,
  // the number of threads per block, blockDim,
  // and the number of the current thread within the block, threadIdx
  //int i = blockIdx.x * blockDim.x + threadIdx.x;
  //int j = blockIdx.y * blockDim.y + threadIdx.y;

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  //real g=p->g;
 //  dt=1.0;
//dt=0.05;
//enum vars rho, mom1, mom2, mom3, energy, b1, b2, b3;

  real maxt=0,max3=0, max1=0;
  
   int ip,jp;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;






   //tmp1  tmp_nuI
 
//compute d3r and d1r
   //tmp2  d3r
    //tmp3 d1r


     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]>1 && ii[1]>1 && ii[2]>1 && ii[0]<p->n[0] && ii[1]<p->n[1]  && ii[2]<p->n[2])
     #else
       if(ii[0]>1 && ii[1]>1 && ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
 
   //if(i>1 && j>1 && i<((p->n[0])) && j<((p->n[1])))       
   { 

	#ifdef USE_SAC_3D
		   wtemp1[encode3p1_hdv1r(p,i,j,k,d3)]=fabs(3.0*(wtemp2[encode3p2_hdv1r(p,i+(dim==0),j+(dim==1),k+(dim==2),tmpnui)] - wtemp2[encode3p2_hdv1r(p,i,j,k,tmpnui)] ) - (wtemp2[encode3p2_hdv1r(p,i+2*(dim==0),j+2*(dim==1),k+2*(dim==2),tmpnui)] - wtemp2[encode3p2_hdv1r(p,i-(dim==0),j-(dim==1),k-(dim==2),tmpnui)]    ));
	#else
		   wtemp1[encode3p1_hdv1r(p,i,j,k,d3)]=fabs(3.0*(wtemp2[encode3p2_hdv1r(p,i+(dim==0),j+(dim==1),k,tmpnui)] - wtemp2[encode3p2_hdv1r(p,i,j,k,tmpnui)] ) - (wtemp2[encode3p2_hdv1r(p,i+2*(dim==0),j+2*(dim==1),k,tmpnui)] - wtemp2[encode3p2_hdv1r(p,i-(dim==0),j-(dim==1),k,tmpnui)]    ));
	#endif

   }

   //__syncthreads();








     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(i<((p->n[0])) && j<((p->n[1]))   && k<((p->n[2])))
      // if(i>0 && j>0 && k>0 && i<=((p->n[0])) && j<=((p->n[1]))   && k<=((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
       //if(i>0 && j>0 && i<=((p->n[0])) && j<=((p->n[1])))
     #endif

   //if(i>0 && j>0 && i<=((p->n[0])) && j<=((p->n[1])))            
   { 

     #ifdef USE_SAC_3D
           wtemp1[encode3p1_hdv1r(p,i+1,j+1,k+1,d1)]=fabs((wtemp2[encode3p2_hdv1r(p,i+(dim==0)+1,j+(dim==1)+1,k+(dim==2)+1,tmpnui)] - wtemp2[encode3p2_hdv1r(p,i+1,j+1,k+1,tmpnui)] ));
           
     #else
           //wtemp1[encode3p1_hdv1r(p,i,j,k,d1)]=fabs((wtemp2[encode3p2_hdv1r(p,i+(dim==0),j+(dim==1),k,tmpnui)] - wtemp2[encode3p2_hdv1r(p,i,j,k,tmpnui)] ));
           wtemp1[encode3p1_hdv1r(p,i+1,j+1,k,d1)]=fabs((wtemp2[encode3p2_hdv1r(p,i+(dim==0)+1,j+(dim==1)+1,k,tmpnui)] - wtemp2[encode3p2_hdv1r(p,i+1,j+1,k,tmpnui)] ));
     #endif

   }

   //__syncthreads();



}



__global__ void hyperdifvisc1ar_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  const int blockdim=blockDim.x;
  const int SZWT=blockdim;
  const int SZWM=blockdim*NVAR;
  int tid=threadIdx.x;
  int i,j,iv;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  real maxt=0,max3=0, max1=0;
  
   int ip,jp;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;
 // __shared__ real wts[512];
 // __shared__ real wms[512];




     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
    //set viscosities
   //if( i<((p->n[0])) && j<((p->n[1])))
   {
     #ifdef USE_SAC_3D
     wtemp2[encode3p2_hdv1r(p,i+1,j+1,k+1,tmpnui)]=wtemp[encode3_hdv1r(p,i,j,k,tmp6)];
     #else
     wtemp2[encode3p2_hdv1r(p,i+1,j+1,0,tmpnui)]=wtemp[encode3_hdv1r(p,i,j,0,tmp6)];
     #endif

   }

   
   //__syncthreads();




 /*    ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<(p->n[1]) && ii[2]<(p->n[2]))
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
    //set viscosities
   //if(i<((p->n[0])) && j<((p->n[1])))
   {
	
        bc_hyperdifr(wtemp2, p,ii, tmpnui,dim);

   }*/


    
   //__syncthreads();





 
}








__global__ void hyperdifvisc1arb0_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  const int blockdim=blockDim.x;
  const int SZWT=blockdim;
  const int SZWM=blockdim*NVAR;
  int tid=threadIdx.x;
  int i,j,iv;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  real maxt=0,max3=0, max1=0;
  
   int ip,jp;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;
 // __shared__ real wts[512];
 // __shared__ real wms[512];


     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

is=1;
field=tmpnui;
  

 #ifdef USE_SAC
   if(   (i==(p->n[0])-1)   && j>=0   && j<(p->n[1])           )
   {
      //for(int is=0;is<3-2*(j<((p->n[1])-1));is++)   
         wtemp2[encode3p2_hdv1r(p,i+2,j+is,k,field)]=wtemp2[encode3p2_hdv1r(p,(p->n[0])-5,j+is,k,field)];
         
   }
 
  if( (i==0)    && j>0   && j<((p->n[1]))   )
    //for(int is=0;is<3-2*(j<((p->n[1])-1));is++)
       wtemp2[encode3p2_hdv1r(p,0,j+is,k,field)]=wtemp2[encode3p2_hdv1r(p,6,j+is,k,field)];

#endif
 #ifdef USE_SAC_3D
   if(   (i==(p->n[0])-1)   && j>0   && j<(p->n[1])      && k>0   && k<(p->n[2])     )
         wtemp2[encode3p2_hdv1r(p,i+2,j+is,k+is,field)]=wtemp2[encode3p2_hdv1r(p,(p->n[0])-5,j+is,k+is,field)];
 
  if( (i==0)    && j>0   && j<((p->n[1])) && k>0   && k<((p->n[2]))  )
       wtemp2[encode3p2_hdv1r(p,0,j+is,k+is,field)]=wtemp2[encode3p2_hdv1r(p,6,j+is,k+is,field)];
#endif

    
   //__syncthreads();





 
}





__global__ void hyperdifvisc1arb1_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  const int blockdim=blockDim.x;
  const int SZWT=blockdim;
  const int SZWM=blockdim*NVAR;
  int tid=threadIdx.x;
  int i,j,iv;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  real maxt=0,max3=0, max1=0;
  
   int ip,jp;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;
 // __shared__ real wts[512];
 // __shared__ real wms[512];


     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

  is=1;
field=tmpnui;


 #ifdef USE_SAC
if( (j==(p->n[1])-1)    && i>0   && i<((p->n[0]))  )
    //for(int is=0;is<3-2*(j<((p->n[1])-1));is++)
       wtemp2[encode3p2_hdv1r(p,i+is,j+2,k,field)]=wtemp2[encode3p2_hdv1r(p,i+is,(p->n[1])-5,k,field)];
 if( (j==0)    && i>0   && i<((p->n[0]))   )
    //for(int is=0;is<3-2*(j<((p->n[1])-1));is++)
       wtemp2[encode3p2_hdv1r(p,i+is,0,k,field)]=wtemp2[encode3p2_hdv1r(p,i+is,6,k,field)];

#endif
 #ifdef USE_SAC_3D
   else if( (j==(p->n[1])-1)    && i>0   && i<((p->n[0])) && k>0   && k<((p->n[2]))  )
       wtemp2[encode3p2_hdv1r(p,i+is,j+2,k+is,field)]=wtemp2[encode3p2_hdv1r(p,i+is,(p->n[1])-5,k+is,field)];
   else if( (j==0)    && i>0   && i<((p->n[0]))  && k>0   && k<((p->n[2]))  )
       wtemp2[encode3p2_hdv1r(p,i+is,0,k+is,field)]=wtemp2[encode3p2_hdv1r(p,i+is,6,k+is,field)];
#endif

    
   //__syncthreads();





 
}





__global__ void hyperdifvisc1arb2_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  const int blockdim=blockDim.x;
  const int SZWT=blockdim;
  const int SZWM=blockdim*NVAR;
  int tid=threadIdx.x;
  int i,j,iv;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];
  real maxt=0,max3=0, max1=0;
  
   int ip,jp;
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int shift=order*NVAR*dimp;
 // __shared__ real wts[512];
 // __shared__ real wms[512];


     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

  

is=1;
field=tmpnui;



 #ifdef USE_SAC_3D
 
   if( (k==(p->n[2])-1)    && i>0   && i<((p->n[0])) && j>0   && j<((p->n[1]))  )
       wtemp2[encode3p2_hdv1r(p,i+is,j+is,k+2,field)]=wtemp2[encode3p2_hdv1r(p,i+is,j+is,(p->n[2])-5,field)];
   if( (k==0)    && i>0   && i<((p->n[0])) && j>0   && j<((p->n[1]))   )
       wtemp2[encode3p2_hdv1r(p,i+is,j+is,0,field)]=wtemp2[encode3p2_hdv1r(p,i+is,j+is,6,field)];
#endif

    
   //__syncthreads();





 
}












__global__ void hyperdifvisc1r_parallel(struct params *p,real *wmod, 
     real *wd, int order, real *wtemp, real *wtemp1, real *wtemp2, int field, int dim)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  const int blockdim=blockDim.x;
  const int SZWT=blockdim;
  const int SZWM=blockdim*NVAR;
  int tid=threadIdx.x;
  real maxt=0,max3=0, max1=0;
  int i,j,iv;
  int is,js;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dt=p->dt;
  real dy=p->dx[1];
  real dx=p->dx[0];


  
   int ip,jp;



  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif  

int bfac1,bfac2,bfac3;
//int bfac1=(field==rho || field>mom2)+(field>rho && field<energy);
//int bfac2= (field==rho || field>mom2);
//int bfac3=(field>rho && field<energy);
int shift=order*NVAR*dimp;
  //__shared__ real wts[512];
  //__shared__ real wms[512];




//init temp1 and temp2 to zero 
//the compute element initialising n[0] or n[1] element must do +1 and +2
//this is because we fit the problem geometrically to nixnj elements 

     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
    //set viscosities
   //if(i<((p->n[0])) && j<((p->n[1])))
   {


        for(int f=tmp1; f<=tmp8; f++)
                 wtemp[fencode3_hdv1r(p,ii,f)]=0;

        for(int f=d1; f<=d3; f++)
     #ifdef USE_SAC_3D
                 wtemp1[encode3p1_hdv1r(p,ii[0],ii[1],ii[2],f)]=0;
                 wtemp2[encode3p2_hdv1r(p,ii[0],ii[1],ii[2],tmpnui)]=0;
     #else
                 wtemp1[encode3p1_hdv1r(p,ii[0],ii[1],k,f)]=0;
                 wtemp2[encode3p2_hdv1r(p,ii[0],ii[1],k,tmpnui)]=0;
     #endif

      if(i==((p->n[0])-1))
      {
        for(int f=d1; f<=d3; f++)
                 wtemp1[encode3p1_hdv1r(p,ii[0]+1,ii[1],k,f)]=0;
        wtemp2[encode3p2_hdv1r(p,i+1,j,k,tmpnui)]=0;
        wtemp2[encode3p2_hdv1r(p,i+2,j,k,tmpnui)]=0;
      }
      if(j==((p->n[1])-1))
      {
          for(int f=d1; f<=d3; f++)
                 wtemp1[encode3p1_hdv1r(p,i,j+1,k,f)]=0;
          wtemp2[encode3p2_hdv1r(p,i,j+1,k,tmpnui)]=0;
          wtemp2[encode3p2_hdv1r(p,i,j+2,k,tmpnui)]=0;
      }

     #ifdef USE_SAC_3D
      if(k==((p->n[2])-1))
      {
          for(int f=d1; f<=d3; f++)
                 wtemp1[encode3p1_hdv1r(p,i,j,k+1,f)]=0;
          wtemp2[encode3p2_hdv1r(p,i,j,k+1,tmpnui)]=0;
          wtemp2[encode3p2_hdv1r(p,i,j,k+2,tmpnui)]=0;
      }

     #endif
      if(j==((p->n[1])-1)  && i==((p->n[0])-1))
      {
          for(int f=d1; f<=d3; f++)
                 wtemp1[encode3p1_hdv1r(p,i+1,j+1,k,f)]=0;



          for(int di=0; di<2; di++)
             for(int dj=0; dj<2; dj++)
                   wtemp2[encode3p2_hdv1r(p,i+1+di,j+1+dj,k,tmpnui)]=0;
               

      }
     #ifdef USE_SAC_3D
      if(i==((p->n[0])-1)  && k==((p->n[2])-1))
      {
          for(int f=d1; f<=d3; f++)
                 wtemp1[encode3p1_hdv1r(p,i+1,j,k+1,f)]=0;
          for(int di=0; di<2; di++)
             for(int dk=0; dk<2; dk++)
                   wtemp2[encode3p2_hdv1r(p,i+1+di,j,k+1+dk,tmpnui)]=0;


      }
      #endif
     #ifdef USE_SAC_3D
      if(j==((p->n[1])-1)  && k==((p->n[2])-1))
      {
          for(int f=d1; f<=d3; f++)
                 wtemp1[encode3p1_hdv1r(p,i+1,j+1,k,f)]=0;

          for(int dk=0; dk<2; dk++)
             for(int dj=0; dj<2; dj++)
                   wtemp2[encode3p2_hdv1r(p,i,j+1+dj,k+1+dk,tmpnui)]=0;


      }
      #endif

     #ifdef USE_SAC_3D
      if(i==((p->n[0])-1) && j==((p->n[1])-1)  && k==((p->n[2])-1))
      {
          for(int f=d1; f<=d3; f++)
                 wtemp1[encode3p1_hdv1r(p,i+1,j+1,k+1,f)]=0;
       
          for(int dk=0; dk<2; dk++)
             for(int dj=0; dj<2; dj++)
               for(int di=0; di<2; di++)
                   wtemp2[encode3p2_hdv1r(p,i+1+di,j+1+dj,k+1+dk,tmpnui)]=0;


      }
      #endif

   }



  

   //__syncthreads();



     ii[0]=ip;
     ii[1]=jp;
     i=ii[0];
     j=ii[1];
     k=0;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
           k=ii[2];
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
    //set viscosities
   //if(i<((p->n[0])) && j<((p->n[1])))
   {

        //for(iv=0;iv<NVAR;iv++)
        //               wms[tid+iv*blockdim]=wmod[fencode_hdv1r(p,i,j,iv)+shift];
        //wts[tid]=wtemp[fencode_hdv1r(p,i,j,tmp6)];
        //temp value for viscosity

       //tmp6  tmpnu
#ifdef USE_SAC
        if(field==energy)
        wtemp[fencode3_hdv1r(p,ii,tmp6)]=wmod[fencode3_hdv1r(p,ii,energy)+shift]-0.5*((wmod[fencode3_hdv1r(p,ii,b1)+shift]*wmod[fencode3_hdv1r(p,ii,b1)+shift]+wmod[fencode3_hdv1r(p,ii,b2)+shift]*wmod[fencode3_hdv1r(p,ii,b2)+shift])+(wmod[fencode3_hdv1r(p,ii,mom1)+shift]*wmod[fencode3_hdv1r(p,ii,mom1)+shift]+wmod[fencode3_hdv1r(p,ii,mom2)+shift]*wmod[fencode3_hdv1r(p,ii,mom2)+shift])/(wmod[fencode3_hdv1r(p,ii,rho)+shift]+wmod[fencode3_hdv1r(p,ii,rhob)+shift] ));
        else
        {
           wtemp[fencode3_hdv1r(p,ii,tmp6)]=wmod[fencode3_hdv1r(p,ii,field)+shift];
	   if((field ==mom1 || field == mom2))
		wtemp[fencode3_hdv1r(p,ii,tmp6)]=wmod[fencode3_hdv1r(p,ii,field)+shift]/(((wmod[fencode3_hdv1r(p,ii,rho)+shift] +wmod[fencode3_hdv1r(p,ii,rhob)+shift])));
        }
        //wtemp2[encode3_hdv1r(p,i+1,j+1,k,tmpnui)]=wtemp[fencode3_hdv1r(p,ii,tmp6)];



#endif

#ifdef USE_SAC_3D
       if(field==energy)
        wtemp[fencode3_hdv1r(p,ii,tmp6)]=wmod[fencode3_hdv1r(p,ii,energy)+shift]-0.5*((wmod[fencode3_hdv1r(p,ii,b1)+shift]*wmod[fencode3_hdv1r(p,ii,b1)+shift]+wmod[fencode3_hdv1r(p,ii,b2)+shift]*wmod[fencode3_hdv1r(p,ii,b2)+shift]+wmod[fencode3_hdv1r(p,ii,b3)+shift]*wmod[fencode3_hdv1r(p,ii,b3)+shift])
+(wmod[fencode3_hdv1r(p,ii,mom1)+shift]*wmod[fencode3_hdv1r(p,ii,mom1)+shift]+wmod[fencode3_hdv1r(p,ii,mom2)+shift]*wmod[fencode3_hdv1r(p,ii,mom2)+shift]+wmod[fencode3_hdv1r(p,ii,mom3)+shift]*wmod[fencode3_hdv1r(p,ii,mom3)+shift])/(wmod[fencode3_hdv1r(p,ii,rho)+shift]+wmod[fencode3_hdv1r(p,ii,rhob)+shift] ));       
       else
       {
          wtemp[fencode3_hdv1r(p,ii,tmp6)]=wmod[fencode3_hdv1r(p,ii,field)+shift];
	if((field ==mom1 || field == mom2 || field == mom3))
		wtemp[fencode3_hdv1r(p,ii,tmp6)]=wmod[fencode3_hdv1r(p,ii,field)+shift]/(((wmod[fencode3_hdv1r(p,ii,rho)+shift] +wmod[fencode3_hdv1r(p,ii,rhob)+shift])));

        }
        //wtemp2[encode3_hdv1r(p,i+1,j+1,k+1,tmpnui)]=wtemp[fencode3_hdv1r(p,ii,tmp6)];



#endif



        wd[fencode3_hdv1r(p,ii,hdnur)]=0;
   }


   //__syncthreads();




}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_hdv1r(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}





int cuhyperdifvisc1r(struct params **p,  struct params **d_p,   real **d_wmod,real **wd,  real **d_wd, int order, real **d_wtemp, real **d_wtemp1, real **d_wtemp2, int field, int dim)
{
  int dimp=(((*p)->n[0]))*(((*p)->n[1]));
     double *d_cmax;
  double maxviscoef;
  double *d_bmax;
  real fn,fractn,in;
  int ndimp;
  int i;
////hipSetDevice(selectedDevice);
   int nit=100;
double *h_cmax;
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

       int NTPB=tnumThreadsPerBlock;
   
  int smemSize = NTPB * sizeof(double);


    fn=log(dimp)/log(2.0);
    fractn=modf(fn,&in);
    
    if(fractn>0)
    {
       fn+=1;
       ndimp=(int)pow(2,fn);
     }
     else
       ndimp=dimp;
       


// dim3 dimBlock(dimblock, 1);
 
 //   dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;


    (*p)->hdmax=0;
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

     hyperdifvisc1r_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim);
     hipDeviceSynchronize();
     hyperdifvisc1ar_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim);
     hipDeviceSynchronize();


if(dim==0)
{
     hyperdifvisc1arb0_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim);
     hipDeviceSynchronize();
}

if(dim==1)
{
     hyperdifvisc1arb1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim);
     hipDeviceSynchronize();
}

if(dim==2)
{
     hyperdifvisc1arb2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim);
     hipDeviceSynchronize();
}





     hyperdifvisc2r_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim);
     hipDeviceSynchronize();
     hyperdifvisc3r_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim);
     hipDeviceSynchronize();
     hyperdifvisc4r_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim);
     hipDeviceSynchronize();


    //compute max hyperviscosity (only used by dt modifier)
     if(((*p)->moddton)==1 )
    {
     // hyperdifvisc5r_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim);
    // hipDeviceSynchronize();

numBlocks = (ndimp+NTPB-1) / NTPB;
    h_cmax = (double*)malloc(numBlocks*sizeof(double));

  hipMalloc((void**)&d_cmax, numBlocks*sizeof(double)); 
  hipMalloc((void**)&d_bmax, numBlocks*sizeof(double)); 

     maxviscoef=(*p)->maxviscoef;
     
     zeropadmaxviscr_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dim, *d_wtemp,ndimp);
      hipDeviceSynchronize();
	hipMemcpy(*wd, *d_wd, NDERV*dimp*sizeof(real), hipMemcpyDeviceToHost);
	hipMemcpy(*d_wtemp, ((*wd)+(hdnur*dimp)), dimp*sizeof(real), hipMemcpyHostToDevice);
 
	/*int s=1;
	while(((s*=2)<=((ndimp/2)-1)) ) 
	{
	   myreduction0computemaxviscr_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,  *d_wd, order, dim, *d_wtemp,ndimp,s);
	   hipDeviceSynchronize();
	}*/
	  for(i=0;i<numBlocks;i++)
		       h_cmax[i]=0;
	  hipMemcpy(d_bmax, h_cmax, numBlocks*sizeof(double), hipMemcpyHostToDevice);

	  newreduction0computemaxviscr_parallel<<<numBlocks,NTPB,smemSize>>>(d_bmax,*d_wtemp,ndimp);
	  hipDeviceSynchronize();
	  hipMemcpy(h_cmax, d_bmax, numBlocks*sizeof(double), hipMemcpyDeviceToHost);

   for( i=0;i<numBlocks;i++)          		
                if(h_cmax[i]>maxviscoef) maxviscoef=h_cmax[i];


       if((*p)->maxviscoef<maxviscoef)
              (*p)->maxviscoef=maxviscoef;

     free(h_cmax);
     hipFree(d_bmax);
     hipFree(d_cmax);


    }
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

    //hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);



  //  printf("field right hdmean hdmax %d %8.8g %8.8g \n",field, (*p)->hdmean, (*p)->hdmax);
}

int cuhyperdifvisc1ir(struct params **p,  struct params **d_p,   real **d_wmod,  real **d_wd, int order, real **d_wtemp, real **d_wtemp1, real **d_wtemp2, int field, int dim)
{

  int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

// dim3 dimBlock(dimblock, 1);
 
 //   dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;

    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

     hyperdifvisc1r_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_wmod,   *d_wd, order, *d_wtemp,*d_wtemp1,*d_wtemp2, field, dim);
     hipDeviceSynchronize();

    hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);


  //  printf("field right hdmean hdmax %d %8.8g %8.8g \n",field, (*p)->hdmean, (*p)->hdmax);
}







