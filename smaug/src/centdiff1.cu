#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"
#include "../include/iobparams.h"
/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/smaugcukernels.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_cd1.cuh"
#include "../include/dervfields_cd1.cuh"
#include "../include/usersource_cd1.cuh"

__device__ __host__
int divflux1(real *dw, real *wd, real *w, struct params *p,int *ii,int field,int dir) {

  int direction;
  int status=0;
  real divflux=0;
  //real g = grad3dn_cd1(wd,wd,p,ii,flux,dir);

	dw[fencode3_cd1(p,ii,field)]+= grad3dn_cd1(wd,wd,p,ii,flux,dir);

	/*if(field==rho && (p->ipe)==0  && ((p)->it)==1 && isnan(g))
        { 
    				printf("nant %d %d %lg %lg %lg  %lg\n",ii[0],ii[1],g,wd[fencode3_cd1(p,ii,flux)],wd[fencode3_cd1(p,ii,delx1)],wd[fencode3_cd1(p,ii,delx2)] );

				ii[0]+=1;									
                                printf("nant 0+1 %d %d  %lg %lg  %lg\n",ii[0]+1,ii[1],wd[fencode3_cd1(p,ii,flux)],wd[fencode3_cd1(p,ii,delx1)],wd[fencode3_cd1(p,ii,delx2)] );
				ii[0]-=1;
				printf("nant 0-1 %d %d  %lg %lg  %lg\n",ii[0]-1,ii[1],wd[fencode3_cd1(p,ii,flux)],wd[fencode3_cd1(p,ii,delx1)],wd[fencode3_cd1(p,ii,delx2)] );
				ii[1]+=1;
				printf("nant 1+1 %d %d  %lg %lg  %lg\n",ii[0],ii[1]+1,wd[fencode3_cd1(p,ii,flux)],wd[fencode3_cd1(p,ii,delx1)],wd[fencode3_cd1(p,ii,delx2)] );
				ii[0]-=1;
				printf("nant %1-1 d %d  %lg %lg  %lg\n\n",ii[0],ii[1]-1,wd[fencode3_cd1(p,ii,flux)],wd[fencode3_cd1(p,ii,delx1)],wd[fencode3_cd1(p,ii,delx2)] );
        }*/
   
  return ( status);
}






__device__ __host__
real transportflux (real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {

        #if defined USE_SAC || defined USE_SAC_3D
     		return(w[fencode3_cd1(p,ii,mom1+direction)]*w[fencode3_cd1(p,ii,field)]/(w[fencode3_cd1(p,ii,rho)]+w[fencode3_cd1(p,ii,rhob)]));
        #else
     		return(w[fencode3_cd1(p,ii,mom1+direction)]*w[fencode3_cd1(p,ii,field)]/w[fencode3_cd1(p,ii,rho)]);
        #endif
	
 
}






__device__ __host__
real fluxmom1 (real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {


         #if defined USE_SAC || defined USE_SAC_3D
     		return( -(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]+w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)]);
        #endif


}




__device__ __host__
real fluxmom10 (real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {


/*real gtest=(direction==0?wd[fencode3_cd1(p,ii,pressuret)]-(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]+w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)]:-(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]+w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)]);

if( (p->ipe)==0  && ((p)->it)==1 && (isnan(gtest) || isnan(w[fencode3_cd1(p,ii,field)]) || w[fencode3_cd1(p,ii,field)]==0      ))
        { 
    	printf("nant %d %d %d %d %lg %lg\n",ii[0],ii[1],field, direction, w[fencode3_cd1(p,ii,rho)],w[fencode3_cd1(p,ii,field)] );
}*/
         #if defined USE_SAC || defined USE_SAC_3D
         return(direction==0?wd[fencode3_cd1(p,ii,pressuret)]-(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]+w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)]:-(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]+w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)]);
        #endif


}

__device__ __host__
real fluxmom11 (real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {


/*real gtest=(direction==1?wd[fencode3_cd1(p,ii,pressuret)]-(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]+w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)]:-(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]+w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)]);
if(  (p->ipe)==0  && ((p)->it)==2 && (isnan(gtest) || isnan(w[fencode3_cd1(p,ii,field)])|| w[fencode3_cd1(p,ii,field)]==0 ))
        { 
    				printf("nant %d %d %d %d %lg %lg \n",ii[0],ii[1],field,direction, w[fencode3_cd1(p,ii,rho)],w[fencode3_cd1(p,ii,field)] );
}*/
         #if defined USE_SAC || defined USE_SAC_3D
         return(direction==1?wd[fencode3_cd1(p,ii,pressuret)]-(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]+w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)]:-(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]+w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)]);
        #endif


}


__device__ __host__
real fluxmom12 (real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {


         #if defined USE_SAC || defined USE_SAC_3D
         return(direction==2?wd[fencode3_cd1(p,ii,pressuret)]-(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]+w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)]:-(w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1b+direction)]+w[fencode3_cd1(p,ii,field+(2*NDIM+3))]*w[fencode3_cd1(p,ii,b1+direction)])-w[fencode3_cd1(p,ii,field+(NDIM+1))]*w[fencode3_cd1(p,ii,b1+direction)]);
        #endif


}







__device__ __host__
int computefluxrho (real *dw, real *wd, real *w, struct params *p,int *ii,int direction) {

  int field;
  int status=0;
      wd[fencode3_cd1(p,ii,flux)]=0.0;
 
         #if defined USE_SAC || defined USE_SAC_3D
	      wd[fencode3_cd1(p,ii,flux)]=  transportflux(dw,wd,w,p,ii,rho,direction)+(w[fencode3_cd1(p,ii,rhob)]*w[fencode3_cd1(p,ii,mom1+direction)])/(w[fencode3_cd1(p,ii,rhob)]+w[fencode3_cd1(p,ii,rho)]);
         #else
             wd[fencode3_cd1(p,ii,flux)]= transportflux(dw,wd,w,p,ii,rho,direction);
         #endif
  
  return ( status);
}


__device__ __host__
int computefluxmom3 (real *dw, real *wd, real *w, struct params *p,int *ii, int field,int direction) {

 
  int status=0;

#ifdef USE_SAC_3D
               wd[fencode3_cd1(p,ii,flux)]=0.0;
    		wd[fencode3_cd1(p,ii,flux)]+=transportflux(dw,wd,w,p,ii,field,direction)+fluxmom12(dw,wd,w,p,ii,field,direction);
               

#endif

  return ( status);
}




__device__ __host__
int computefluxmom2 (real *dw, real *wd, real *w, struct params *p,int *ii, int field,int direction) {

 
  int status=0;

               wd[fencode3_cd1(p,ii,flux)]=0.0;
 
        #ifdef USE_SAC
    		wd[fencode3_cd1(p,ii,flux)]+=  transportflux(dw,wd,w,p,ii,field,direction)+fluxmom11(dw,wd,w,p,ii,field,direction);

 
        #endif
        #ifdef USE_SAC_3D
    		wd[fencode3_cd1(p,ii,flux)]+= transportflux(dw,wd,w,p,ii,field,direction)+fluxmom11(dw,wd,w,p,ii,field,direction);
 
 
        #endif

  return ( status);
}







__device__ __host__
int computefluxmom1 (real *dw, real *wd, real *w, struct params *p,int *ii, int field,int direction) {

 
  int status=0;

               wd[fencode3_cd1(p,ii,flux)]=0.0;



        #ifdef ADIABHYDRO
     		wd[fencode3_cd1(p,ii,flux)]+= transportflux(dw,wd,w,p,ii,field,direction);
        #endif
        #ifdef USE_SAC
    		wd[fencode3_cd1(p,ii,flux)]+=  transportflux(dw,wd,w,p,ii,field,direction)+fluxmom10(dw,wd,w,p,ii,field,direction);
 
        #endif
        #ifdef USE_SAC_3D
    		wd[fencode3_cd1(p,ii,flux)]= transportflux(dw,wd,w,p,ii,field,direction)+fluxmom10(dw,wd,w,p,ii,field,direction);
 
        #endif
        
  return ( status);
}







//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void computeflux (real *dw, real *wd, real *w, struct params *p,int *ii, int field,int dir) {




  switch(field)
  {
     case rho:

       computefluxrho(dw,wd,w,p,ii,dir);

     break;
     case mom1:
      computefluxmom1(dw,wd,w,p,ii,field,dir);

      break;
     case mom2:
       computefluxmom2(dw,wd,w,p,ii,field,dir);

      break;
     #ifdef USE_SAC_3D
       case mom3:
        computefluxmom3(dw,wd,w,p,ii,field,dir);
        break;
     #endif
  }

}




__global__ void centdiff1init_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dy=p->dx[1];
  real dx=p->dx[0];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     
 

   fid=0;
   

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
                        {
                            dwn1[fencode3_cd1(p,ii,f)]=0.0;
                               wd[fencode3_cd1(p,ii,flux)]=0.0;


 
                         }

   
 __syncthreads();                       




}



__global__ void centdiff1_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dy=p->dx[1];
  real dx=p->dx[0];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


   fid=0;




     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif



                        switch(dir)
                        {
                         case 0:
                          #ifdef USE_SAC_3D
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2))
     			  #endif
                         
                            computeflux(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,0); 
                         break;
                         case 1:
                          #ifdef USE_SAC_3D
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2))
     			  #endif
                         
                            computeflux(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,1); 
                         break;
                          #ifdef USE_SAC_3D
                         case 2:

       				if(ii[2]<p->n[2] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[1]>1 && ii[1]<(p->n[1]-2))

                         
                            computeflux(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,2); 
                         break;
                         #endif
                        }



__syncthreads();                        



}










__global__ void centdiff1a_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  real dy=p->dx[1];
  real dx=p->dx[0];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
    int nk=p->n[2];
    real dz=p->dx[2];
#endif
 #ifdef USE_SAC_3D
   int kp;
   
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     

   fid=0;





     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

			     #ifdef USE_SAC
				   if(ii[0]>1 && ii[1] >1 && ii[0]<(ni-2) && ii[1]<(nj-2))
			     #endif
			     #ifdef USE_SAC_3D
				  if(ii[0]>1 && ii[1] >1 && ii[2] >1 && ii[0]<(ni-2) && ii[1]<(nj-2) && ii[2]<(nk-2))
			     #endif                        
                               divflux1(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,dir);  


 __syncthreads();


}

__global__ void centdiff1af_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];


  real dy=p->dx[1];
  real dx=p->dx[0];
 

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
    int nk=p->n[2];
    real dz=p->dx[2];
#endif
 #ifdef USE_SAC_3D
   int kp;
   
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


   fid=0;

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif



                        switch(dir)
                        {
                         case 0:

 			     #ifdef USE_SAC
				   if(ii[1]>1 && ii[1] <(nj-2) && ii[0]<(ni) )
			     #endif
			     #ifdef USE_SAC_3D
				   if(ii[1]>1 && ii[1] <(nj-2) && ii[0]<(ni) &&  ii[2]>1 && ii[2] <(nk-2) )
			     #endif                          
                              wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd1(p,ii,f)]; 
                         break;
                         case 1:
			     #ifdef USE_SAC
				   if(ii[0]>1 && ii[1] <(nj) && ii[0]<(ni-2) )
			     #endif
			     #ifdef USE_SAC_3D
				   if(ii[0]>1 && ii[1] <(nj) && ii[0]<(ni-2) &&  ii[2]>1 && ii[2] <(nk-2) )
			     #endif 
                         
                              wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd1(p,ii,f)];
                         break;
                         case 2:

 
			     #ifdef USE_SAC_3D
				   if(ii[0]>1 &&  ii[0]<(ni-2)  && ii[1]>1 &&  ii[1]<(nj-2) && ii[2] <(nk) )
                               wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd1(p,ii,f)];
			     #endif                         
                             
                         break;
                        }


               /* if( ii[1] <(nj) && ii[0]<(ni) )
                           if(p->ipe==0 && ii[0]==124  && (p->it)==2)
                           {
                               wmod[fencode3_cd1(p,ii,rho)]=0.225;
 			       w[fencode3_cd1(p,ii,rho)]=0.225;
                           }*/

               /* if( ii[1] <(nj) && ii[0]<(ni) )
                           if(p->ipe==3 && ii[1]==3  && (p->it)==2)
                           {
                               wmod[fencode3_cd1(p,ii,rho)]=0.22114;
 			       w[fencode3_cd1(p,ii,rho)]=0.22114;
                           }*/
	

  __syncthreads();


}


__global__ void centdiff1binit_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];
  real dy=p->dx[1];
  real dx=p->dx[0];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     
 

   fid=0;
   

     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
                        {
                            dwn1[fencode3_cd1(p,ii,f)]=0.0;

                        }

   
 __syncthreads();                       




}


__global__ void centdiff1b_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{
 
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  real dy=p->dx[1];
  real dx=p->dx[0];
 
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
    int nk=p->n[2];
    real dz=p->dx[2];
#endif
 #ifdef USE_SAC_3D
   int kp;
   
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     

   fid=0;





     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif


#if(defined(USE_USERSOURCE))
   {

     ii[0]=ip;
     ii[1]=jp;
#endif
     #if(defined(USE_SAC_3D) && defined(USE_USERSOURCE))
	   ii[2]=kp;
     #endif


     #if(defined(USE_SAC_3D) && defined(USE_USERSOURCE))

       if(ii[0]<((p->n[0])) && ii[1]<((p->n[1])) && ii[2]<((p->n[2]))    )
     #endif
     #if(defined(USE_SAC) && defined(USE_USERSOURCE))
 
      if(ii[0]<(p->n[0]) && ii[1]<(p->n[1]))
     #endif

                     #ifdef USE_USERSOURCE
                            addsourceterms1_cd1(dwn1,wd,wmod+ordero*NVAR*dimp,p,s,ii,f,dir); 


                      }
                    __syncthreads();
                     #endif



               // }
    


}



__global__ void centdiff1bf_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt, int f, int dir)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j;
  int fid;
  int index,k;
  int ni=p->n[0];
  int nj=p->n[1];

  real dy=p->dx[1];
  real dx=p->dx[0];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
    int nk=p->n[2];
    real dz=p->dx[2];
#endif
 #ifdef USE_SAC_3D
   int kp;
   
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


   fid=0;



     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif


			     #ifdef USE_SAC
				   if( ii[1] <(nj) && ii[0]<(ni) )
			     #endif
			     #ifdef USE_SAC_3D
				   if(ii[1] <(nj) && ii[0]<(ni) &&   ii[2] <(nk) )
			     #endif                          
                              wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd1(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd1(p,ii,f)];

            

  __syncthreads();


}



/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cd1(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

int cucentdiff1(struct params **p, struct params **d_p,struct state **d_s, real **d_w,  real **d_wmod, real **d_dwn1, real **d_wd, int order, int ordero, real dt, int field, int dir)
{
 int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
 
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;

    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);
 
     centdiff1init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_s,*d_w,*d_wmod, *d_dwn1,  *d_wd, order, ordero,dt,field,dir);
  
     hipDeviceSynchronize();
     centdiff1_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_s,*d_w,*d_wmod, *d_dwn1,  *d_wd, order, ordero,dt,field,dir);

     hipDeviceSynchronize();
     centdiff1a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_s,*d_w,*d_wmod, *d_dwn1,  *d_wd, order, ordero,dt,field,dir);
     hipDeviceSynchronize();


     centdiff1af_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_s,*d_w,*d_wmod, *d_dwn1,  *d_wd, order, ordero,dt,field,dir);
     hipDeviceSynchronize();
     

     
}


