#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"
#include "../include/iobparams.h"
/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/smaugcukernels.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_cd2.cuh"
#include "../include/dervfields_cd2.cuh"

#include "../include/usersource_cd2.cuh"

__device__ __host__
real fluxe2(real *dw, real *wd, real *w, real *wmod, struct params *p,int *ii, int dir) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;

  real ddcx=0,ddcy=0;

   real fluxt=0;


        #ifdef USE_SAC



      		fluxt= +wd[fencode3_cd2(p,ii,ptb)]*grad3dn_cd2(wd,wd,p,ii,vel1+dir,dir);
	fluxt += +w[fencode3_cd2(p,ii,b1b)]*w[fencode3_cd2(p,ii,b1b+dir)]*grad3dn_cd2(wd,wd,p,ii,vel1,0)+w[fencode3_cd2(p,ii,b2b)]*w[fencode3_cd2(p,ii,b1b+dir)]*grad3dn_cd2(wd,wd,p,ii,vel1+1,1);
          #endif


        #ifdef USE_SAC_3D
      		fluxt= +wd[fencode3_cd2(p,ii,ptb)]*grad3dn_cd2(wd,wd,p,ii,vel1+dir,dir);

               fluxt += +w[fencode3_cd2(p,ii,b1b)]*w[fencode3_cd2(p,ii,b1b+dir)]*grad3dn_cd2(wd,wd,p,ii,vel1,0)+w[fencode3_cd2(p,ii,b2b)]*w[fencode3_cd2(p,ii,b1b+dir)]*grad3dn_cd2(wd,wd,p,ii,vel1+1,1)+w[fencode3_cd2(p,ii,b3b)]*w[fencode3_cd2(p,ii,b1b+dir)]*grad3dn_cd2(wd,wd,p,ii,vel1+2,2);
        #endif

  return fluxt;


}



__device__ __host__
int divflux_cd2(real *dw, real *wd, real *w, struct params *p,int *ii,int field,int dir) {

  int direction;
  int status=0;
  real divflux=0;



  dw[fencode3_cd2(p,ii,field)]= grad3dn_cd2(wd,wd,p,ii,flux,dir);

 
 #ifdef USE_SAC

  //commented out to test against vac
  /*if(field==energy)
  {    
     dw[fencode3_cd2(p,ii,field)]+=fluxe2(dw, wd, w, p,ii,dir)-w[fencode3_cd2(p,ii,rho)]*((p->g[dir])*w[fencode3_cd2(p,ii,mom1+dir)]    )/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);
   }*/


 #endif
  return ( status);
}


__device__ __host__
int addenergyterms_cd2(real *dw, real *wd, real *w, real *wmod, struct params *p,int *ii,int field,int dir) {

  int direction;
  int status=0;
  real divflux=0;
  

 #if defined USE_SAC  ||  defined USE_SAC_3D

  
  if(field==energy)
  {    
     //computept3_cd2(w,wd,p,ii);
     //wmod[fencode3_cd2(p,ii,field)]+=fluxe2(dw, wd, wmod, p,ii,dir);/*+w[fencode3_cd2(p,ii,rho)]*((p->g[dir])*w[fencode3_cd2(p,ii,mom1+dir)]    )/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);*/
     

        
              		wmod[fencode3_cd2(p,ii,field)]-= +(p->dt)*wd[fencode3_cd2(p,ii,ptb)]*grad3dn_cd2(wd,wd,p,ii,vel1+dir,dir);
                      //wmod[fencode3_cd2(p,ii,field)]-= +(p->dt)*wd[fencode3_cd2(p,ii,ptb)]*grad3d_cd2(wd,p,ii,vel1+dir,dir);

                    for(int idim=0;idim<NDIM;idim++)
                         wmod[fencode3_cd2(p,ii,field)]+=(p->dt)*wmod[fencode3_cd2(p,ii,b1b+idim)]*wmod[fencode3_cd2(p,ii,b1b+dir)]*grad3dn_cd2(wd,wd,p,ii,vel1+dir,idim);
                        //wmod[fencode3_cd2(p,ii,field)]+=(p->dt)*wmod[fencode3_cd2(p,ii,b1b+idim)]*wmod[fencode3_cd2(p,ii,b1b+dir)]*grad3d_cd2(wd,p,ii,vel1+idim,idim);

		//fluxt= +(((p->gamma)-1)*w[fencode3_cd2(p,ii,energyb)]- 0.5*((p->gamma)-2)*(w[fencode3_cd2(p,ii,b1b)]*w[fencode3_cd2(p,ii,b1b)]+w[fencode3_cd2(p,ii,b2b)]*w[fencode3_cd2(p,ii,b2b)]+w[fencode3_cd2(p,ii,b3b)]*w[fencode3_cd2(p,ii,b3b)]))*grad3d_cd2(wd,p,ii,vel1+dir,dir);

               
               //flux= -(((p->gamma)-1)*w[fencode3_cd2(p,ii,energyb)]- 0.5*((p->gamma)-2)*(w[fencode3_cd2(p,ii,b1b)]*w[fencode3_cd2(p,ii,b1b)]+w[fencode3_cd2(p,ii,b2b)]*w[fencode3_cd2(p,ii,b2b)]+w[fencode3_cd2(p,ii,b3b)]*w[fencode3_cd2(p,ii,b3b)]))*grad3d_cd2(wd,p,ii,vel1+dir,dir);
              // fluxt += +w[fencode3_cd2(p,ii,b1b)]*w[fencode3_cd2(p,ii,b1b+dir)]*grad3d_cd2(wd,p,ii,vel1,0)+w[fencode3_cd2(p,ii,b2b)]*w[fencode3_cd2(p,ii,b1b+dir)]*grad3d_cd2(wd,p,ii,vel1+1,1)+w[fencode3_cd2(p,ii,b3b)]*w[fencode3_cd2(p,ii,b1b+dir)]*grad3d_cd2(wd,p,ii,vel1+2,2);

   }


 #endif
  return ( status);
}

__device__ __host__
int addgrav_cd2(real *dw, real *wd, real *w, real *wmod, struct params *p,int *ii) {

  //int direction;
  int status=0;
  int field,dir;
  //real divflux=0;
  //dw[fencode3_cd2(p,ii,field)]= grad_cd2(wd,p,ii,flux,dir);//+grad_cd2(wd,p,ii,f2,1); 


  for(field=rho;field<NVAR;field++)
  {
    switch(field)
    {
               case mom1:
               case mom2:
                    #ifdef USE_SAC_3D
                    case mom3:
                    #endif  
                         dir=field-mom1;
                         wmod[fencode3_cd2(p,ii,field)]+=(p->dt)* (p->g[dir])*w[fencode3_cd2(p,ii,rho)];

                 break;
                 
                 case energy:
                      for(dir=0; dir<NDIM; dir++)
                        wmod[fencode3_cd2(p,ii,field)]+=(p->dt)*w[fencode3_cd2(p,ii,rho)]*((p->g[dir])*w[fencode3_cd2(p,ii,mom1+dir)]    )/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);

                 break;
                 }                               
                                   
  }
  
 


  return ( status);
}


__device__ __host__
real transportflux_cd2 (real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {

 

  // real fluxt=0;

   //transport flux
   //use versions with velocity less ops may improve performance
        #if defined USE_SAC  || defined USE_SAC_3D
     return(w[fencode3_cd2(p,ii,mom1+direction)]*w[fencode3_cd2(p,ii,field)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]));
 // flux= wd[fencode3_cd2(p,ii,vel1+direction)]*w[fencode3_cd2(p,ii,field)];
        #else
     return(w[fencode3_cd2(p,ii,mom1+direction)]*w[fencode3_cd2(p,ii,field)]/w[fencode3_cd2(p,ii,rho)]);
//flux= w[fencode3_cd2(p,ii,vel1+direction)]*w[fencode3_cd2(p,ii,field)];
        #endif

 
}




__device__ __host__
real fluxb1(real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {

 
   real fluxt=0;

       #if defined USE_SAC  || defined USE_SAC_3D

  fluxt= -(w[fencode3_cd2(p,ii,b1+direction)]+w[fencode3_cd2(p,ii,field+(NDIM+2)+direction)])*w[fencode3_cd2(p,ii,mom1+(field-b1))]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);

fluxt+= (w[fencode3_cd2(p,ii,field+(NDIM+2))])*w[fencode3_cd2(p,ii,mom1+direction)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);



         #endif

 
  return fluxt;
}



__device__ __host__
real fluxe1(real *dw, real *wd, real *w, struct params *p,int *ii, int direction) {

  real ddc=0;
  real fi, fim1;
  real  fip2=0, fim2=0;
  real ddcx=0,ddcy=0;

   real fluxt=0;

//computept3_cd2(w,wd,p,ii);

         #if defined USE_SAC


fluxt = w[fencode3_cd2(p,ii,mom1+direction)]*(wd[fencode3_cd2(p,ii,pressuret)]);


fluxt  -= w[fencode3_cd2(p,ii,b1+direction)]*(w[fencode3_cd2(p,ii,b1b)]*w[fencode3_cd2(p,ii,mom1)]+w[fencode3_cd2(p,ii,b2b)]*w[fencode3_cd2(p,ii,mom2)]);
fluxt -= w[fencode3_cd2(p,ii,b1b+direction)]*(w[fencode3_cd2(p,ii,b1)]*w[fencode3_cd2(p,ii,mom1)]+w[fencode3_cd2(p,ii,b2)]*w[fencode3_cd2(p,ii,mom2)]);
fluxt /= (w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);
fluxt += w[fencode3_cd2(p,ii,mom1+direction)]*w[fencode3_cd2(p,ii,energyb)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);
fluxt -=w[fencode3_cd2(p,ii,b1+direction)]*(w[fencode3_cd2(p,ii,b1)]*w[fencode3_cd2(p,ii,mom1)]+w[fencode3_cd2(p,ii,b2)]*w[fencode3_cd2(p,ii,mom2)])/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);


         #endif

#ifdef USE_SAC_3D
fluxt = w[fencode3_cd2(p,ii,mom1+direction)]*(wd[fencode3_cd2(p,ii,pressuret)]);


fluxt  -= w[fencode3_cd2(p,ii,b1+direction)]*(w[fencode3_cd2(p,ii,b1b)]*w[fencode3_cd2(p,ii,mom1)]+w[fencode3_cd2(p,ii,b2b)]*w[fencode3_cd2(p,ii,mom2)]+w[fencode3_cd2(p,ii,b3b)]*w[fencode3_cd2(p,ii,mom3)]);
fluxt -= w[fencode3_cd2(p,ii,b1b+direction)]*(w[fencode3_cd2(p,ii,b1)]*w[fencode3_cd2(p,ii,mom1)]+w[fencode3_cd2(p,ii,b2)]*w[fencode3_cd2(p,ii,mom2)]+w[fencode3_cd2(p,ii,b3)]*w[fencode3_cd2(p,ii,mom3)]);
fluxt /= (w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);
fluxt +=w[fencode3_cd2(p,ii,mom1+direction)]*w[fencode3_cd2(p,ii,energyb)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);
fluxt -=w[fencode3_cd2(p,ii,b1+direction)]*(w[fencode3_cd2(p,ii,b1)]*w[fencode3_cd2(p,ii,mom1)]+w[fencode3_cd2(p,ii,b2)]*w[fencode3_cd2(p,ii,mom2)]+w[fencode3_cd2(p,ii,b3)]*w[fencode3_cd2(p,ii,mom3)])/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);

#endif

  return fluxt;

}








__device__ __host__
int computefluxe(real *dw, real *wd, real *w, struct params *p,int *ii,int direction) {

  int field;//, direction;
  int status=0;
wd[fencode3_cd2(p,ii,flux)]=0.0;
         #if defined USE_SAC  || defined USE_SAC_3D
	     wd[fencode3_cd2(p,ii,flux)]= transportflux_cd2(dw,wd,w,p,ii,energy,direction)+fluxe1(dw,wd,w,p,ii,direction);
         #endif

        
  return ( status);
}

__device__ __host__
int computefluxb1 (real *dw, real *wd, real *w, struct params *p,int *ii, int field,int direction) {


  int status=0;
wd[fencode3_cd2(p,ii,flux)]=0.0;
        
      if(direction==0)
wd[fencode3_cd2(p,ii,flux)]= 0.0;
      else
 #if defined USE_SAC  || defined USE_SAC_3D  
wd[fencode3_cd2(p,ii,flux)]=  transportflux_cd2(dw,wd,w,p,ii,field,direction)-(w[fencode3_cd2(p,ii,b1+direction)]+w[fencode3_cd2(p,ii,b1b+direction)])*w[fencode3_cd2(p,ii,mom1)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)])+ (w[fencode3_cd2(p,ii,b1b)])*w[fencode3_cd2(p,ii,mom1+direction)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);//+fluxb1(dw,wd,w,p,ii,field,direction);

         #endif

  return ( status);
}

__device__ __host__
int computefluxb2 (real *dw, real *wd, real *w, struct params *p,int *ii, int field,int direction) {


  int status=0;
   wd[fencode3_cd2(p,ii,flux)]=0.0;      
      if(direction==1)
wd[fencode3_cd2(p,ii,flux)]= 0.0;
else
#if defined USE_SAC  || defined USE_SAC_3D 


wd[fencode3_cd2(p,ii,flux)]= transportflux_cd2(dw,wd,w,p,ii,field,direction)-(w[fencode3_cd2(p,ii,b1+direction)]+w[fencode3_cd2(p,ii,b1b+direction)])*w[fencode3_cd2(p,ii,mom2)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)])+ (w[fencode3_cd2(p,ii,b2b)])*w[fencode3_cd2(p,ii,mom1+direction)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);//+fluxb1(dw,wd,w,p,ii,field,direction);

         #endif


  return ( status);
}


__device__ __host__
int computefluxb3 (real *dw, real *wd, real *w, struct params *p,int *ii, int field,int direction) {

wd[fencode3_cd2(p,ii,flux)]=0.0;
  int status=0;
 #ifdef USE_SAC_3D
 

      if(direction==2)
wd[fencode3_cd2(p,ii,flux)]= 0.0;
else
wd[fencode3_cd2(p,ii,flux)]= transportflux_cd2(dw,wd,w,p,ii,field,direction)-(w[fencode3_cd2(p,ii,b1+direction)]+w[fencode3_cd2(p,ii,b1b+direction)])*w[fencode3_cd2(p,ii,mom3)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)])+ (w[fencode3_cd2(p,ii,b3b)])*w[fencode3_cd2(p,ii,mom1+direction)]/(w[fencode3_cd2(p,ii,rho)]+w[fencode3_cd2(p,ii,rhob)]);//+fluxb1(dw,wd,w,p,ii,field,direction);



 
  #endif
  return ( status);
}



//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void computeflux_cd2 (real *dw, real *wd, real *w, struct params *p,int *ii, int field,int dir) {

  //int status=0;
  switch(field)
  {
     case energy:
      computefluxe(dw,wd,w,p,ii,dir);
      
      // add the following terms for SAC
      // del((b bb+ bb b).v)+ptb del v - bb bb del v
     break;
     case b1:
      computefluxb1(dw,wd,w,p,ii,field,dir);
     break;
     case b2:
       computefluxb2(dw,wd,w,p,ii,field,dir);



     break;
#ifdef USE_SAC_3D
     case b3:
      computefluxb3(dw,wd,w,p,ii,field,dir);
     break;
#endif
  }
  //return ( status);
}



__global__ void centdiff2a_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt,int f,int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,fid;
 // int index;
  int ni=p->n[0];
  int nj=p->n[1];
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int nk=p->n[2];
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     

   fid=0;




     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif
                          #ifdef USE_SAC_3D
       				if(ii[0]<((p->n[0])-2) && ii[0]>1 && ii[1]>1 && ii[1]<((p->n[1])-2) && ii[2]>1 && ii[2]<((p->n[2])-2))
     			  #else
       				if(ii[0]<((p->n[0]))-2 && ii[0]>1  && ii[1]>1 && ii[1]<((p->n[1])-2))
     			  #endif
                                divflux_cd2(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,dir); 



__syncthreads();
                        

                         
}

__global__ void centdiff2b_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt,int f,int dir)
{

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,fid;
 
  int ni=p->n[0];
  int nj=p->n[1];
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int nk=p->n[2];
   int kp;

   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


   fid=0;






     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif
  real del;



                        switch(dir)
                        {
                         case 0:

                         #ifdef USE_SAC_3D
       				if(ii[0]<((p->n[0]))  && ii[1]>1 && ii[1]<((p->n[1])-2) && ii[2]>1 && ii[2]<((p->n[2])-2))
     			  #else
       				if(ii[0]<((p->n[0]))   && ii[1]>1 && ii[1]<((p->n[1])-2))
     			  #endif
 
                              wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd2(p,ii,f)]; 
                         break;
                         case 1:
                         #ifdef USE_SAC_3D
       				if(ii[0]>1 && ii[0]<((p->n[0])-2)  &&  ii[1]<((p->n[1])) && ii[2]>1 && ii[2]<((p->n[2])-2))
     			  #else
       				if(ii[0]>1 && ii[0]<((p->n[0])-2)   && ii[1]<((p->n[1])) )
     			  #endif
  
                              wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd2(p,ii,f)]; 

  
                         break;
                         #ifdef USE_SAC_3D
                         case 2:

 
      			if(ii[0]>1 && ii[0]<((p->n[0])-2)  && ii[1]>1 && ii[1]<((p->n[1])-2)  && ii[2]<((p->n[2])))
 
                              wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd2(p,ii,f)]; 
                         break;
                         #endif
                        }



__syncthreads(); 


}

__global__ void centdiff2ci_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt,int f,int dir)
{


  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,fid;
 
  int ni=p->n[0];
  int nj=p->n[1];
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int nk=p->n[2];
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


   fid=0;

   //compute pbg used in next source term


     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif



     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
   {
   //computevel3_cd2(wmod+(order*NVAR*dimp),wd,p,ii);  
   computepbg3_cd2(wmod+(ordero*NVAR*dimp),wd,p,ii);  
    }
      
    __syncthreads();





     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif


     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
                            dwn1[fencode3_cd2(p,ii,f)]=0.0;




__syncthreads();



                         
}


__global__ void centdiff2c_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt,int f,int dir)
{
 

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,fid;
 // int index;
  int ni=p->n[0];
  int nj=p->n[1];
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int nk=p->n[2];
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


   fid=0;

   //compute pbg used in next source term


     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif







     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif




			//if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
     #ifdef USE_SAC_3D
       if(ii[0]<((p->n[0])-2) && ii[1]<((p->n[1])-2) && ii[2]<((p->n[2])-2)     && ii[0]>1    &&  ii[1]>1   && ii[2]>1   )
     #else
       if(ii[0]<(p->n[0])-2 && ii[1]<(p->n[1])-2)
     #endif
                                addenergyterms_cd2(dwn1,wd,w,wmod+ordero*NVAR*dimp,p,ii,f,dir);

    /* #if(defined(USE_SAC_3D) && defined(USE_USERSOURCE))
       //if(ii[0]<((p->n[0])-2) && ii[1]<((p->n[1])-2) && ii[2]<((p->n[2])-2)     && ii[0]>1    &&  ii[1]>1   && ii[2]>1   )
       if(ii[0]<((p->n[0])) && ii[1]<((p->n[1])) && ii[2]<((p->n[2]))    )
     #endif
     #if(defined(USE_SAC) && defined(USE_USERSOURCE))
       //if(ii[0]<(p->n[0])-2 && ii[1]<(p->n[1])-2)
      if(ii[0]<(p->n[0]) && ii[1]<(p->n[1]))
     #endif

                     #ifdef USE_USERSOURCE
                                addsourceterms2_cd2(dwn1,wd,wmod+ordero*NVAR*dimp,p,s,ii,f,dir); 
                     #endif*/





                /*if( ii[1] <(nj) && ii[0]<(ni) )
                           if(p->ipe==1 && ii[1]==125  && (p->it)==2)
                           {
                               wmod[fencode3_cd2(p,ii,rho)]=0.22113;
 			       w[fencode3_cd2(p,ii,rho)]=0.22113;
                           }*/

               /* if( ii[1] <(nj) && ii[0]<(ni) )
                           if(p->ipe==3 && ii[1]==3  && (p->it)==2)
                           {
                               wmod[fencode3_cd2(p,ii,rho)]=0.22118;
 			       w[fencode3_cd2(p,ii,rho)]=0.22118;
                           }*/

                /*if( ii[1] <(nj) && ii[0]<(ni) )
                           if(p->ipe==1 && ii[1]==127  && (p->it)==2)
                           {
                               wmod[fencode3_cd2(p,ii,rho)]=wmod[fencode_cd2(p,ii[0],ii[1]-4,rho)];
 			       w[fencode3_cd2(p,ii,rho)]= w[fencode_cd2(p,ii[0],ii[1]-4,rho)];
                           }

                if( ii[1] <(nj) && ii[0]<(ni) )
                           if(p->ipe==3 && ii[1]==0  && (p->it)==2)
                           {
                               wmod[fencode3_cd2(p,ii,rho)]=wmod[fencode_cd2(p,ii[0],ii[1]+4,rho)];
 			       w[fencode3_cd2(p,ii,rho)]= w[fencode_cd2(p,ii[0],ii[1]+4,rho)];
                           }

                if( ii[1] <(nj) && ii[0]<(ni) )
                           if(p->ipe==1 && ii[1]==126  && (p->it)==2)
                           {
                               wmod[fencode3_cd2(p,ii,rho)]=wmod[fencode_cd2(p,ii[0],ii[1]-4,rho)];
 			       w[fencode3_cd2(p,ii,rho)]= w[fencode_cd2(p,ii[0],ii[1]-4,rho)];
                           }

                if( ii[1] <(nj) && ii[0]<(ni) )
                           if(p->ipe==3 && ii[1]==1  && (p->it)==2)
                           {
                               wmod[fencode3_cd2(p,ii,rho)]=wmod[fencode_cd2(p,ii[0],ii[1]+4,rho)];
 			       w[fencode3_cd2(p,ii,rho)]= w[fencode_cd2(p,ii[0],ii[1]+4,rho)];
                           }*/



__syncthreads();



                         
}

__global__ void grav_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt,int f,int dir)
{
 
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,fid;
  int ni=p->n[0];
  int nj=p->n[1];
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int nk=p->n[2];
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


   fid=0;

   //compute pbg used in next source term


     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif







     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif


     #ifdef USE_SAC_3D
       if(ii[0]<((p->n[0])-2) && ii[1]<((p->n[1])-2) && ii[2]<((p->n[2])-2)     && ii[0]>1    &&  ii[1]>1   && ii[2]>1   )
     #else
       if(ii[0]<(p->n[0])-2 && ii[1]<(p->n[1])-2)
     #endif
                                addgrav_cd2(dwn1,wd,w,wmod+ordero*NVAR*dimp,p,ii);



__syncthreads();



                         
}

__global__ void source_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt)
{
  

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,fid;
  int f,dir;

  int ni=p->n[0];
  int nj=p->n[1];
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int nk=p->n[2];
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


   fid=0;

   //compute pbg used in next source term


     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif







     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif


     #if(defined(USE_SAC_3D) && defined(USE_USERSOURCE))
 
       if(ii[0]<((p->n[0])) && ii[1]<((p->n[1])) && ii[2]<((p->n[2]))    )
     #endif
     #if(defined(USE_SAC) && defined(USE_USERSOURCE))

      if(ii[0]<(p->n[0]) && ii[1]<(p->n[1]))
     #endif

                     #ifdef USE_USERSOURCE
                               addsourceterms2_cd2(dwn1,wd,wmod+ordero*NVAR*dimp,p,s,ii,f,dir); 
                     #endif







	


__syncthreads();



                         
}


__global__ void centdiff2d_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt,int f,int dir)
{
 

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,fid;

  int ni=p->n[0];
  int nj=p->n[1];
  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int nk=p->n[2];
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


   fid=0;




     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

			     #ifdef USE_SAC
				   if(ii[0]<ni  && ii[1]<(nj))
			     #endif
			     #ifdef USE_SAC_3D
				  if(ii[0]<ni    && ii[1]<(nj) && ii[2]<(nk))
			     #endif 
				{  
                              wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]=wmod[fencode3_cd2(p,ii,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_cd2(p,ii,f)];
				  
				} 



__syncthreads(); 

                         
}






__global__ void centdiff2_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt,int f,int dir)
{
 

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,fid;
 
  int ni=p->n[0];
  int nj=p->n[1];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     


   fid=0;
   



     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif


                        switch(dir)
                        {
                         case 0:
                          #ifdef USE_SAC_3D
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[0]<p->n[0] && ii[1]>1 && ii[1]<(p->n[1]-2))
     			  #endif
                         
                            computeflux_cd2(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,0); 
                         break;
                         case 1:
                          #ifdef USE_SAC_3D
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[2]>1 && ii[2]<(p->n[2]-2))
     			  #else
       				if(ii[1]<p->n[1] && ii[0]>1 && ii[0]<(p->n[0]-2))
     			  #endif
                         
                            computeflux_cd2(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,1); 
                         break;
                          #ifdef USE_SAC_3D
                         case 2:

       				if(ii[2]<p->n[2] && ii[0]>1 && ii[0]<(p->n[0]-2) && ii[1]>1 && ii[1]<(p->n[1]-2))

                         
                            computeflux_cd2(dwn1,wd,wmod+order*NVAR*dimp,p,ii,f,2); 
                         break;
                         #endif
                        }
              
 

__syncthreads();                        






                         
}


__global__ void centdiff2init_parallel(struct params *p, struct state *s, real *w, real *wmod, 
    real *dwn1, real *wd, int order, int ordero, real dt,int f,int dir)
{
 

  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,fid;

  int ni=p->n[0];
  int nj=p->n[1];

  int ii[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int kp;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   int ip,jp;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni);
   jp=(iindex-(kp*(nj*ni)))/ni;
   ip=iindex-(kp*nj*ni)-(jp*ni);
#else
    jp=iindex/ni;
   ip=iindex-(jp*ni);
#endif     

   fid=0;
   





     ii[0]=ip;
     ii[1]=jp;
     #ifdef USE_SAC_3D
	   ii[2]=kp;
     #endif

     #ifdef USE_SAC_3D
       if(ii[0]<p->n[0] && ii[1]<p->n[1] && ii[2]<p->n[2])
     #else
       if(ii[0]<p->n[0] && ii[1]<p->n[1])
     #endif
                        {
                            dwn1[fencode3_cd2(p,ii,f)]=0.0;

                               wd[fencode3_cd2(p,ii,flux)]=0.0;

                        }


  __syncthreads();   


                         
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_cd2(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}




int cucentdiff2(struct params **p, struct params **d_p, struct state **d_s, real **d_w,  real **d_wmod, real **d_dwn1, real **d_wd, int order,int ordero, real dt, int field,int dir)
{
 int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
 
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;
   //  hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
  
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);

     centdiff2init_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_s,*d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt,field,dir);
     hipDeviceSynchronize();

     centdiff2_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p, *d_s,*d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt,field,dir);
     hipDeviceSynchronize();



     centdiff2a_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_s, *d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt,field,dir);
     hipDeviceSynchronize();

     centdiff2b_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_s, *d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt,field,dir);
     hipDeviceSynchronize();

 

     centdiff2ci_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_s, *d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt,field,dir);
     hipDeviceSynchronize();


    centdiff2c_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_s, *d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt,field,dir);
     hipDeviceSynchronize();


    //hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);
    //printf("source params %G %f %f\n",(*p)->test, (*p)->chyp[0] , (*p)->chyp[1]);
    //printf("source params %G \n",(*p)->test);


     //centdiff2d_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_s, *d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt,field,dir);
     //hipDeviceSynchronize();


     // hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
     //hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
     //hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

     //checkErrors("copy data from device");

}

int cugrav(struct params **p, struct params **d_p, struct state **d_s, real **d_w,  real **d_wmod, real **d_dwn1, real **d_wd, int order,int ordero, real dt)
{
 int dimp=(((*p)->n[0]))*(((*p)->n[1]));

  int field=rho;
  int dir=0;   
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 

   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;
  
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);


     grav_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_s, *d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt,field,dir);
     hipDeviceSynchronize();


    //hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);
    //printf("source params %G %f %f %G\n",(*p)->test, (*p)->chyp[0] , (*p)->chyp[1] , (*p)->chyp[2]);



     //centdiff2d_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_s, *d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt,field,dir);
     //hipDeviceSynchronize();


     // hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
     //hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
     //hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

     //checkErrors("copy data from device");

}

int cusource(struct params **p, struct params **d_p, struct state **d_s, real **d_w,  real **d_wmod, real **d_dwn1, real **d_wd, int order,int ordero, real dt)
{
 int dimp=(((*p)->n[0]))*(((*p)->n[1]));

   int field=rho;
  int dir=0;     
 #ifdef USE_SAC_3D
   
  dimp=(((*p)->n[0]))*(((*p)->n[1]))*(((*p)->n[2]));
#endif 
  
   int numBlocks = (dimp+numThreadsPerBlock-1) / numThreadsPerBlock;
   //  hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
   // if(order==0)
    hipMemcpy(*d_p, *p, sizeof(struct params), hipMemcpyHostToDevice);


     //centdiff2ci_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_s, *d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt,field,dir);
     //hipDeviceSynchronize();


     source_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_s, *d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt);
     hipDeviceSynchronize();


    hipMemcpy(*p, *d_p, sizeof(struct params), hipMemcpyDeviceToHost);
    //printf("vx vy e %8.16G %8.16G %8.16G\n", (*p)->chyp[0] , (*p)->chyp[1] ,(*p)->test);



     //centdiff2d_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_s, *d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt,field,dir);
     //hipDeviceSynchronize();


     // hipMemcpy(*w, *d_w, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
     //hipMemcpy(*wnew, *d_wnew, NVAR*((*p)->n[0])* ((*p)->n[1])*sizeof(real), hipMemcpyDeviceToHost);
     //hipMemcpy(*b, *d_b, (((*p)->n[0])* ((*p)->n[1]))*sizeof(real), hipMemcpyDeviceToHost);

     //checkErrors("copy data from device");

}
