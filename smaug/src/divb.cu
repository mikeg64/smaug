#include "hip/hip_runtime.h"
#include "../include/cudapars.h"
#include "../include/paramssteeringtest1.h"

/////////////////////////////////////
// standard imports
/////////////////////////////////////
#include <stdio.h>
#include <math.h>
#include "../include/smaugcukernels.h"

/////////////////////////////////////
// kernel function (CUDA device)
/////////////////////////////////////
#include "../include/gradops_db.cuh"
#include "../include/dervfields_db.cuh"


__device__ __host__
real dbsourcerho (real *dw, real *wd, real *w, struct params *p,int *ii) {

  real src=0;

  
 
  return src;
}

__device__ __host__
real dbsourcemom (real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {

  real src=0;
  switch(direction)
  {
	case 0:
         src= -wd[fencode3_db(p,ii,divb)]*w[fencode3_db(p,ii,b1)];
	break;
	case 1:
         src= -wd[fencode3_db(p,ii,divb)]*w[fencode3_db(p,ii,b2)];
	break;
   #ifdef USE_SAC_3D
	case 2:
         src= -wd[fencode3_db(p,ii,divb)]*w[fencode3_db(p,ii,b3)];
	break;
   #endif
  }

  return(isnan(src)?0:src);


}

__device__ __host__
real dbsourceb (real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {

  real src=0;
  switch(direction)
  {
   #ifdef USE_SAC
	case 0:
         src= -wd[fencode3_db(p,ii,divb)]*w[fencode3_db(p,ii,mom1)]/(w[fencode3_db(p,ii,rho)]+w[fencode3_db(p,ii,rhob)]);
	break;
	case 1:
         src= -wd[fencode3_db(p,ii,divb)]*w[fencode3_db(p,ii,mom2)]/(w[fencode3_db(p,ii,rho)]+w[fencode3_db(p,ii,rhob)]);
	break;
   #endif
   #ifdef USE_SAC_3D
	case 2:
         src= -wd[fencode3_db(p,ii,divb)]*w[fencode3_db(p,ii,mom3)]/(w[fencode3_db(p,ii,rho)]+w[fencode3_db(p,ii,rhob)]);
	break;
   #endif
  }
   return(isnan(src)?0:src);
}

__device__ __host__
real dbsourceenergy (real *dw, real *wd, real *w, struct params *p,int *ii) {

 real src=0;
    src= -wd[fencode3_db(p,ii,divb)]*wd[fencode3_db(p,ii,bdotv)];
 
  return ( src);
}


__device__ __host__
int dbderivsourcerho (real *dw, real *wd, real *w, struct params *p,int *ii) {

  int status=0;
  int field=rho;
        dw[fencode3_db(p,ii,field)]=dw[fencode3_db(p,ii,field)]+dbsourcerho(dw,wd,w,p,ii);
     	//dw[fencode3_db(p,ii,field)]=w[fencode3_db(p,ii,field)]+10;
  return ( status);
}

__device__ __host__
int dbderivsourcemom (real *dw, real *wd, real *w, struct params *p,int *ii,int field, int direction) {

  int status=0;
     	//dw[fencode3_db(p,ii,field)]=w[fencode3_db(p,ii,field)]+20+5*(2*direction+1);
        dw[fencode3_db(p,ii,field)]=dw[fencode3_db(p,ii,field)]+dbsourcemom(dw,wd,w,p,ii,field,direction);
        //dw[fencode3_db(p,ii,field)]=-ddotcurrentmom(dw,wd,w,p,ii,field,direction);

  return ( status);
}

__device__ __host__
int dbderivsourceb (real *dw, real *wd, real *w, struct params *p,int *ii, int field, int direction) {

  int status=0;
        dw[fencode3_db(p,ii,field)]=dw[fencode3_db(p,ii,field)]+dbsourceb(dw,wd,w,p,ii,field,direction);

  return ( status);
}

__device__ __host__
int dbderivsourceenergy (real *dw, real *wd, real *w, struct params *p,int *ii) {

  int status=0;
  int field=energy;
        dw[fencode3_db(p,ii,field)]=dw[fencode3_db(p,ii,field)]+dbsourceenergy(dw,wd,w,p,ii);

  return ( status);
}

//rho, mom1, mom2, mom3, energy, b1, b2, b3
__device__ __host__
void dbderivsource (real *dw, real *wd, real *w, struct params *p,int *ii, int field) {

  //int status=0;
  switch(field)
  {
     case rho:
      dbderivsourcerho(dw,wd,w,p,ii);
     break;
     case mom1:
      dbderivsourcemom(dw,wd,w,p,ii,field,0);
     break;
     case mom2:
      dbderivsourcemom(dw,wd,w,p,ii,field,1);
     break;
   #ifdef USE_SAC_3D
     case mom3:
      dbderivsourcemom(dw,wd,w,p,ii,field,2);
     break;
   #endif
     case energy:
       dbderivsourceenergy(dw,wd,w,p,ii);
     break;
     case b1:
      dbderivsourceb(dw,wd,w,p,ii,field,0);
     break;
     case b2:
      dbderivsourceb(dw,wd,w,p,ii,field,1);
     break;
   #ifdef USE_SAC_3D
     case b3:
      dbderivsourceb(dw,wd,w,p,ii,field,2);
     break;
   #endif
  }
  //return ( status);
}


__global__ void divb_parallel(struct params *p, real *w, real *wmod, 
    real *dwn1, real *wd, int order,int ordero, real dt)
{
  int iindex = blockIdx.x * blockDim.x + threadIdx.x;
  int i,j,k;

  int ni=p->n[0];
  int nj=p->n[1];

   int ip,jp,ipg,jpg;
  int iia[NDIM];
  int dimp=((p->n[0]))*((p->n[1]));
 #ifdef USE_SAC_3D
   int nk,kp,kpg;
   real dz=p->dx[2];
   dimp=((p->n[0]))*((p->n[1]))*((p->n[2]));
#endif  
   //int ip,jp,ipg,jpg;

  #ifdef USE_SAC_3D
   kp=iindex/(nj*ni/((p->npgp[1])*(p->npgp[0])));
   jp=(iindex-(kp*(nj*ni/((p->npgp[1])*(p->npgp[0])))))/(ni/(p->npgp[0]));
   ip=iindex-(kp*nj*ni/((p->npgp[1])*(p->npgp[0])))-(jp*(ni/(p->npgp[0])));
#endif
 #if defined USE_SAC || defined ADIABHYDRO
    jp=iindex/(ni/(p->npgp[0]));
   ip=iindex-(jp*(ni/(p->npgp[0])));
#endif  


int shift=order*NVAR*dimp;

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     iia[0]=ip*(p->npgp[0])+ipg;
     iia[1]=jp*(p->npgp[1])+jpg;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp*(p->npgp[2])+kpg;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i<((p->n[0])) && j<((p->n[1]))  && k<((p->n[2])))
     #else
       if(i<((p->n[0])) && j<((p->n[1])))
     #endif
     for(int f=rho; f<=b2; f++)
                dwn1[fencode3_db(p,iia,f)]=0;
   }
 __syncthreads();

   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     iia[0]=ip*(p->npgp[0])+ipg;
     iia[1]=jp*(p->npgp[1])+jpg;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp*(p->npgp[2])+kpg;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i>2 && j>2 && k>2 && i<(ni-2) && j<(nj-2) && k<(nk-2))
     #else
       if(i>2 && j>2 && i<(ni-2) && j<(nj-2))
     #endif
  //if(i>2 && j>2 && i<(ni-2) && j<(nj-2))
	{
           if(p->divbfix)
           {   

               wd[fencode3_db(p,iia,divb)]=grad3d_db(wmod+order*NVAR*dimp,p,iia,b1,0)+grad3d_db(wmod+order*NVAR*dimp,p,iia,b2,1);
               #ifdef USE_SAC
		wd[fencode3_db(p,iia,divb)]+=grad3d_db(wmod+order*NVAR*dimp,p,iia,b1b,0)+grad3d_db(wmod+order*NVAR*dimp,p,iia,b2b,1);
                #endif
               #ifdef USE_SAC_3D
		wd[fencode3_db(p,iia,divb)]+=grad3d_db(wmod+order*NVAR*dimp,p,iia,b3,0)+grad3d_db(wmod+order*NVAR*dimp,p,iia,b3b,1);
                #endif
               for(int f=rho; f<=b2; f++) 
               {              
                  dbderivsource(dwn1,wd,wmod+order*NVAR*dimp,p,iia,f);
 
               }
            }

	}
}
 __syncthreads();


   for(ipg=0;ipg<(p->npgp[0]);ipg++)
   for(jpg=0;jpg<(p->npgp[1]);jpg++)
   #ifdef USE_SAC_3D
     for(kpg=0;kpg<(p->npgp[2]);kpg++)
   #endif
   {

     iia[0]=ip*(p->npgp[0])+ipg;
     iia[1]=jp*(p->npgp[1])+jpg;
     i=iia[0];
     j=iia[1];
     k=0;
     #ifdef USE_SAC_3D
	   iia[2]=kp*(p->npgp[2])+kpg;
           k=iia[2];
     #endif

     #ifdef USE_SAC_3D
      if(i>1 && j >1 && k>1 && i<(ni-2) && j<(nj-2) && k<(nk-2))
     #else
       if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
     #endif
   // if(i>1 && j >1 && i<(ni-2) && j<(nj-2))
                         {
                         if(p->divbfix)
                          { 
                             for(int f=rho; f<=b2; f++) 
                             //                                                  - sign here same as vac maybe a +
                              wmod[fencode3_db(p,iia,f)+(ordero*NVAR*dimp)]=wmod[fencode3_db(p,iia,f)+(ordero*NVAR*dimp)]-dt*dwn1[fencode3_db(p,iia,f)]; 
                          }

                         }
              //  }	
}
  __syncthreads();



  
}


/////////////////////////////////////
// error checking routine
/////////////////////////////////////
void checkErrors_db(char *label)
{
  // we need to synchronise first to catch errors due to
  // asynchroneous operations that would otherwise
  // potentially go unnoticed

  hipError_t err;

  err = hipDeviceSynchronize();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }

  err = hipGetLastError();
  if (err != hipSuccess)
  {
    char *e = (char*) hipGetErrorString(err);
    fprintf(stderr, "CUDA Error: %s (at %s)", e, label);
  }
}

int cudivb(struct params **p, struct params **d_p, real **d_w,  real **d_wmod, real **d_dwn1, real **d_wd,  int order,int ordero, real dt)
{
    int status=0;
    dim3 dimBlock(dimblock, 1);
    
    dim3 dimGrid(((*p)->n[0])/dimBlock.x,((*p)->n[1])/dimBlock.y);
   int numBlocks = (((*p)->n[0])*((*p)->n[1])+numThreadsPerBlock-1) / numThreadsPerBlock;


    divb_parallel<<<numBlocks, numThreadsPerBlock>>>(*d_p,*d_w, *d_wmod, *d_dwn1,  *d_wd, order,ordero,dt);
	    //printf("called update\n"); 
    hipDeviceSynchronize();


 return status;


}



