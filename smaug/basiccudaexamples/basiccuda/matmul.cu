
#include <hip/hip_runtime.h>
#include <stdio.h>

// Thread block size
#define BLOCK_SIZE 16
// Forward declaration of the device multiplication function
__global__ void Muld(float*, float*, int, int, float*, float *);




// Host multiplication function
// Compute C = A * B
//   hA is the height of A
//   wA is the width of A
//   wB is the width of B
void Mul(const float* A, const float* B, int hA, int wA, int wB,
         float* C, float *par)
{
    


    int size;
    // Load A and B to the device
    float* Ad;
    float* pard;
    size = hA * wA * sizeof(float);
    hipMalloc((void**)&Ad, size);
    hipMalloc((void **)&pard,2);
    hipMemcpy(Ad, A, size, hipMemcpyHostToDevice);
    float* Bd;
    size = wA * wB * sizeof(float);
    hipMalloc((void**)&Bd, size);
    hipMemcpy(Bd, B, size, hipMemcpyHostToDevice);
    // Allocate C on the device
    float* Cd;
    size = hA * wB * sizeof(float);
    hipMalloc((void**)&Cd, size);
    // Compute the execution configuration assuming
    // the matrix dimensions are multiples of BLOCK_SIZE
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 dimGrid(wB / dimBlock.x, hA / dimBlock.y);
    // Launch the device computation
    Muld<<<dimGrid, dimBlock>>>(Ad, Bd, wA, wB, Cd,pard);
    // Read C from the device
    hipMemcpy(C, Cd, size, hipMemcpyDeviceToHost);
    hipMemcpy(par,pard,2,hipMemcpyDeviceToHost);
    // Free device memory
    hipFree(Ad);
    hipFree(Bd);
    hipFree(Cd);
}

// Device multiplication function called by Mul()
// Compute C = A * B
//   wA is the width of A
//   wB is the width of B
__global__ void Muld(float* A, float* B, int wA, int wB, float* C, float* pard)
{
    // Block index
    int bx = blockIdx.x;
    int by = blockIdx.y;
    // Thread index
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    // Index of the first sub-matrix of A processed by the block
    int aBegin = wA * BLOCK_SIZE * by;
    // Index of the last sub-matrix of A processed by the block
    int aEnd   = aBegin + wA - 1;
    // Step size used to iterate through the sub-matrices of A
    int aStep = BLOCK_SIZE;
    // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;
    // Step size used to iterate through the sub-matrices of B
    int bStep = BLOCK_SIZE * wB;
    // The element of the block sub-matrix that is computed
    // by the thread
    float Csub = 0;
     if(bx<1 ) pard[0]=bx;
     if(tx<1) pard[1]=tx;
    // Loop over all the sub-matrices of A and B required to
    // compute the block sub-matrix
    for (int a = aBegin, b = bBegin;
             a <= aEnd;
             a += aStep, b += bStep) {
        // Shared memory for the sub-matrix of A
        __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
        // Shared memory for the sub-matrix of B
        __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];
        // Load the matrices from global memory to shared memory;
        // each thread loads one element of each matrix
        As[ty][tx] = A[a + wA * ty + tx];
        Bs[ty][tx] = B[b + wB * ty + tx];
        // Synchronize to make sure the matrices are loaded
        __syncthreads();
        // Multiply the two matrices together;
        // each thread computes one element
        // of the block sub-matrix
        for (int k = 0; k < BLOCK_SIZE; ++k)
          Csub += As[ty][k] * Bs[k][tx];
      // Synchronize to make sure that the preceding
      // computation is done before loading two new
      // sub-matrices of A and B in the next iteration
      __syncthreads();
  }
  // Write the block sub-matrix to global memory;
  // each thread writes one element
  int c = wB * BLOCK_SIZE * by + BLOCK_SIZE * bx;
  C[c + wB * ty + tx] = Csub;
}


int main(void)
{
    int hA,wA,wB;
    int size;
    int i;

    hA=20;
    wA=4;
    wB=20;
    // Load A and B to the device
    float* A;
    float *par;
    size = hA * wA * sizeof(float);
    A=(float *)malloc( size);
    for(i=0; i<hA*wA; i++) A[i]=i;
    
    float* B;
    size = wA * wB * sizeof(float);
    B=(float *)malloc( size);
    for(i=0; i<wA*wB; i++) B[i]=2*i;
    // Allocate C on the device
    float* C;
    size = hA * wB * sizeof(float);
    C=(float *)malloc(size);


    par=(float *)malloc(1);




    Mul(A,B,hA,wA,wB,C,par);
    for(i=0; i<hA*wA; i++) printf("%d %f ",i,A[i]);
    printf("\n");
    for(i=0; i<wA*wB; i++) printf("%d %f ",i,B[i]);
    printf("\n");
    for(i=0; i<hA*wB; i++) printf("%d %f ",i,C[i]);
    printf("\n");



    printf("\n%f %f\n",par[0],par[1]);

}

